#include "hip/hip_runtime.h"
#include "CudaKernels.cuh"

__global__ void cudaFindMinMax( CudaImageContainer arrayIn, double* minArrayOut, double* maxArrayOut, size_t n )
{
	extern __shared__ double maxData[];
	extern __shared__ double minData[];

	size_t tid = threadIdx.x;
	size_t i = blockIdx.x*blockDim.x*2 + tid;
	size_t gridSize = blockDim.x*2*gridDim.x;

	while (i<n)
	{
		maxData[tid] = arrayIn[i];
		minData[tid] = arrayIn[i];

		if (i+blockDim.x<n)
		{
			if(maxData[tid]<arrayIn[i+blockDim.x])
				maxData[tid] = arrayIn[i+blockDim.x];

			if(minData[tid]>arrayIn[i+blockDim.x])
				minData[tid] = arrayIn[i+blockDim.x];
		}

		i += gridSize;
	}
	__syncthreads();


	if (blockDim.x >= 2048)
	{
		if (tid < 1024) 
		{
			if(maxData[tid]<maxData[tid + 1024])
				maxData[tid] = maxData[tid + 1024];

			if(minData[tid]>minData[tid + 1024])
				minData[tid] = minData[tid + 1024];
		}
		__syncthreads();
	}
	if (blockDim.x >= 1024)
	{
		if (tid < 512) 
		{
			if(maxData[tid]<maxData[tid + 512])
				maxData[tid] = maxData[tid + 512];

			if(minData[tid]>minData[tid + 512])
				minData[tid] = minData[tid + 512];
		}
		__syncthreads();
	}
	if (blockDim.x >= 512)
	{
		if (tid < 256) 
		{
			if(maxData[tid]<maxData[tid + 256])
				maxData[tid] = maxData[tid + 256];

			if(minData[tid]>minData[tid + 256])
				minData[tid] = minData[tid + 256];
		}
		__syncthreads();
	}
	if (blockDim.x >= 256) {
		if (tid < 128)
		{
			if(maxData[tid]<maxData[tid + 128])
				maxData[tid] = maxData[tid + 128];

			if(minData[tid]>minData[tid + 128])
				minData[tid] = minData[tid + 128];
		}
		__syncthreads(); 
	}
	if (blockDim.x >= 128) 
	{
		if (tid < 64)
		{
			if(maxData[tid]<maxData[tid + 64])
				maxData[tid] = maxData[tid + 64];

			if(minData[tid]>minData[tid + 64])
				minData[tid] = minData[tid + 64];
		}
		__syncthreads(); 
	}

	if (tid < 32) {
		if (blockDim.x >= 64) 
		{
			{
				if(maxData[tid]<maxData[tid + 64])
					maxData[tid] = maxData[tid + 64];

				if(minData[tid]>minData[tid + 64])
					minData[tid] = minData[tid + 64];
			}
			__syncthreads(); 
		}
		if (blockDim.x >= 32)
		{
			if(maxData[tid]<maxData[tid + 16])
				maxData[tid] = maxData[tid + 16];

			if(minData[tid]>minData[tid + 16])
				minData[tid] = minData[tid + 16];
			__syncthreads(); 
		}
		if (blockDim.x >= 16)
		{
			if(maxData[tid]<maxData[tid + 8])
				maxData[tid] = maxData[tid + 8];

			if(minData[tid]>minData[tid + 8])
				minData[tid] = minData[tid + 8];
			__syncthreads(); 
		}
		if (blockDim.x >= 8)
		{
			if(maxData[tid]<maxData[tid + 4])
				maxData[tid] = maxData[tid + 4];

			if(minData[tid]>minData[tid + 4])
				minData[tid] = minData[tid + 4];
			__syncthreads(); 
		}
		if (blockDim.x >= 4)
		{
			if(maxData[tid]<maxData[tid + 2])
				maxData[tid] = maxData[tid + 2];

			if(minData[tid]>minData[tid + 2])
				minData[tid] = minData[tid + 2];
			__syncthreads(); 
		}
		if (blockDim.x >= 2)
		{
			if(maxData[tid]<maxData[tid + 1])
				maxData[tid] = maxData[tid + 1];

			if(minData[tid]>minData[tid + 1])
				minData[tid] = minData[tid + 1];
			__syncthreads(); 
		}
	}

	if (tid==0)
	{
		minArrayOut[blockIdx.x] = minData[0];
		maxArrayOut[blockIdx.x] = maxData[0];
	}
}

