#include "hip/hip_runtime.h"
#include "CudaKernels.cuh"

__global__ void cudaNormalizeHistogram(size_t* histogram, double* normHistogram, Vec<size_t> imageDims)
{
	int x = blockIdx.x;
	normHistogram[x] = (double)(histogram[x]) / (imageDims.x*imageDims.y*imageDims.z);
}

