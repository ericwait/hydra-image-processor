#include "hip/hip_runtime.h"
#include "CudaKernels.cuh"

__global__ void cudaAddTwoImagesWithFactor( CudaImageContainer imageIn1, CudaImageContainer imageIn2, CudaImageContainer imageOut, double factor,
										   DevicePixelType minValue, DevicePixelType maxValue )
{
	DeviceVec<size_t> coordinate;
	coordinate.x = threadIdx.x + blockIdx.x * blockDim.x;
	coordinate.y = threadIdx.y + blockIdx.y * blockDim.y;
	coordinate.z = threadIdx.z + blockIdx.z * blockDim.z;

	if (coordinate<imageIn1.getDeviceDims())
	{
		double subtractor = factor*(double)imageIn2[coordinate];
		DevicePixelType outValue = (double)imageIn1[coordinate] + subtractor;

		imageOut[coordinate] = min(maxValue,max(minValue,outValue));
	}
}

