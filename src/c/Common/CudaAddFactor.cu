#include "hip/hip_runtime.h"
#include "CudaKernels.cuh"

__global__ void cudaAddFactor( CudaImageContainer imageIn1, CudaImageContainer imageOut, double factor, DevicePixelType minValue,
							  DevicePixelType maxValue )
{
	DeviceVec<size_t> coordinate;
	coordinate.x = threadIdx.x + blockIdx.x * blockDim.x;
	coordinate.y = threadIdx.y + blockIdx.y * blockDim.y;
	coordinate.z = threadIdx.z + blockIdx.z * blockDim.z;

	if (coordinate<imageIn1.getDeviceDims())
	{
		double outValue = imageIn1[coordinate] + factor;
		imageOut[coordinate] = (outValue>maxValue) ? (maxValue) : ((outValue<minValue) ? (minValue) : (outValue));
	}
}
