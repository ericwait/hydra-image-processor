#include "hip/hip_runtime.h"
#include "CudaUtilities.cuh"
#include "CudaKernels.cuh"
#include "CudaProcessBuffer.cuh"
#include "CudaDeviceImages.cuh"

//Percent of memory that can be used on the device
const double MAX_MEM_AVAIL = 0.95;

std::vector<ImageChunk> calculateBuffers(Vec<size_t> imageDims, int numBuffersNeeded, size_t memAvailable, const hipDeviceProp_t& prop,
										 Vec<size_t> kernalDims/*=Vec<size_t>(0,0,0)*/)
{
	size_t numVoxels = (size_t)(memAvailable / (sizeof(HostPixelType)*numBuffersNeeded));

	Vec<size_t> overlapVolume;
	overlapVolume.x = kernalDims.x * imageDims.y * imageDims.z;
	overlapVolume.y = imageDims.x * kernalDims.y * imageDims.z;
	overlapVolume.z = imageDims.x * imageDims.y * kernalDims.z;

	Vec<size_t> deviceDims(0,0,0);

	if (overlapVolume.x>overlapVolume.y && overlapVolume.x>overlapVolume.z) // chunking in X is the worst
	{
		deviceDims.x = imageDims.x;
		double leftOver = (double)numVoxels/imageDims.x;
		double squareDim = sqrt(leftOver);

		if (overlapVolume.y<overlapVolume.z) // chunking in Y is second worst
		{
			if (squareDim>imageDims.y)
				deviceDims.y = imageDims.y;
			else 
				deviceDims.y = (size_t)squareDim;

			deviceDims.z = (size_t)(leftOver/deviceDims.y);

			if (deviceDims.z>imageDims.z)
				deviceDims.z = imageDims.z;
		}
		else // chunking in Z is second worst
		{
			if (squareDim>imageDims.z)
				deviceDims.z = imageDims.z;
			else 
				deviceDims.z = (size_t)squareDim;

			deviceDims.y = (size_t)(leftOver/deviceDims.z);

			if (deviceDims.y>imageDims.y)
				deviceDims.y = imageDims.y;
		}
	}
	else if (overlapVolume.y>overlapVolume.z) // chunking in Y is the worst
	{
		deviceDims.y = imageDims.y;
		double leftOver = (double)numVoxels/imageDims.y;
		double squareDim = sqrt(leftOver);

		if (overlapVolume.x<overlapVolume.z)
		{
			if (squareDim>imageDims.x)
				deviceDims.x = imageDims.x;
			else 
				deviceDims.x = (size_t)squareDim;

			deviceDims.z = (size_t)(leftOver/deviceDims.x);

			if (deviceDims.z>imageDims.z)
				deviceDims.z = imageDims.z;
		}
		else
		{
			if (squareDim>imageDims.z)
				deviceDims.z = imageDims.z;
			else 
				deviceDims.z = (size_t)squareDim;

			deviceDims.x = (size_t)(leftOver/deviceDims.z);

			if (deviceDims.x>imageDims.x)
				deviceDims.x = imageDims.x;
		}
	}
	else // chunking in Z is the worst
	{
		deviceDims.z = imageDims.z;
		double leftOver = (double)numVoxels/imageDims.z;
		double squareDim = sqrt(leftOver);

		if (overlapVolume.x<overlapVolume.y)
		{
			if (squareDim>imageDims.x)
				deviceDims.x = imageDims.x;
			else 
				deviceDims.x = (size_t)squareDim;

			deviceDims.y = (size_t)(leftOver/deviceDims.x);

			if (deviceDims.y>imageDims.y)
				deviceDims.y = imageDims.y;
		}
		else
		{
			if (squareDim>imageDims.y)
				deviceDims.y = imageDims.y;
			else 
				deviceDims.y = (size_t)squareDim;

			deviceDims.x = (size_t)(leftOver/deviceDims.z);

			if (deviceDims.x>imageDims.x)
				deviceDims.x = imageDims.x;
		}
	}

	return calculateChunking(imageDims, deviceDims, prop, kernalDims);
}

std::vector<ImageChunk> calculateChunking(Vec<size_t> orgImageDims, Vec<size_t> deviceDims, const hipDeviceProp_t& prop,
										  Vec<size_t> kernalDims/*=Vec<size_t>(0,0,0)*/)
{
	std::vector<ImageChunk> localChunks;
	Vec<size_t> margin((kernalDims + 1)/2); //integer round
	Vec<size_t> chunkDelta(deviceDims-margin*2);
	Vec<size_t> numChunks(1,1,1);

	if (orgImageDims.x>deviceDims.x)
		numChunks.x = (size_t)ceil((double)orgImageDims.x/chunkDelta.x);
	else
		chunkDelta.x = orgImageDims.x;

	if (orgImageDims.y>deviceDims.y)
		numChunks.y = (size_t)ceil((double)orgImageDims.y/chunkDelta.y);
	else
		chunkDelta.y = orgImageDims.y;

	if (orgImageDims.z>deviceDims.z)
		numChunks.z = (size_t)ceil((double)orgImageDims.z/chunkDelta.z);
	else
		chunkDelta.z = orgImageDims.z;

	localChunks.resize(numChunks.product());

	Vec<size_t> curChunk(0,0,0);
	Vec<size_t> imageStart(0,0,0);
	Vec<size_t> chunkROIstart(0,0,0);
	Vec<size_t> imageROIstart(0,0,0);
	Vec<size_t> imageEnd(0,0,0);
	Vec<size_t> chunkROIend(0,0,0);
	Vec<size_t> imageROIend(0,0,0);

	for (curChunk.z=0; curChunk.z<numChunks.z; ++curChunk.z)
	{
		for (curChunk.y=0; curChunk.y<numChunks.y; ++curChunk.y)
		{
			for (curChunk.x=0; curChunk.x<numChunks.x; ++curChunk.x)
			{
				imageROIstart = chunkDelta * curChunk;
				imageROIend = Vec<size_t>::min(imageROIstart + chunkDelta, orgImageDims);
				imageStart = Vec<size_t>(Vec<int>::max(Vec<int>(imageROIstart)-Vec<int>(margin), Vec<int>(0,0,0)));
				imageEnd = Vec<size_t>::min(imageROIend + margin, orgImageDims);
				chunkROIstart = imageROIstart - imageStart;
				chunkROIend = imageROIend - imageStart;

				ImageChunk* curImageBuffer = &localChunks[numChunks.linearAddressAt(curChunk)];

				curImageBuffer->imageStart = imageStart;
				curImageBuffer->chunkROIstart = chunkROIstart;
				curImageBuffer->imageROIstart = imageROIstart;
				curImageBuffer->imageEnd = imageEnd;
				curImageBuffer->chunkROIend = chunkROIend;
				curImageBuffer->imageROIend = imageROIend;

				calcBlockThread(curImageBuffer->getFullChunkSize(),prop,curImageBuffer->blocks,curImageBuffer->threads);
			}

			curChunk.x = 0;
		}

		curChunk.y = 0;
	}

	return localChunks;
}

CudaProcessBuffer::CudaProcessBuffer(int device/*=0*/)
{
	defaults();
	this->device = device;
	deviceSetup();
}

CudaProcessBuffer::~CudaProcessBuffer()
{
	defaults();
}

void CudaProcessBuffer::deviceSetup()
{
	HANDLE_ERROR(hipSetDevice(device));
	HANDLE_ERROR(hipGetDeviceProperties(&deviceProp,device));
}

void CudaProcessBuffer::defaults()
{
	device = 0;
	orgImageDims = Vec<size_t>(0,0,0);
	maxDeviceDims = Vec<size_t>(0,0,0);
}

//////////////////////////////////////////////////////////////////////////
// Helper Functions
//////////////////////////////////////////////////////////////////////////

void CudaProcessBuffer::setMaxDeviceDims(std::vector<ImageChunk> &chunks, Vec<size_t> &maxDeviceDims)
{
	for (std::vector<ImageChunk>::iterator curChunk=chunks.begin(); curChunk!=chunks.end(); ++curChunk)
	{
		Vec<size_t> curDim = curChunk->getFullChunkSize();

		if (curDim.x>maxDeviceDims.x)
			maxDeviceDims.x = curDim.x;

		if (curDim.y>maxDeviceDims.y)
			maxDeviceDims.y = curDim.y;

		if (curDim.z>maxDeviceDims.z)
			maxDeviceDims.z = curDim.z;
	}
}

void runGaussIterations(Vec<int> &gaussIterations, std::vector<ImageChunk>::iterator& curChunk, CudaDeviceImages& deviceImages,
						Vec<size_t> sizeconstKernelDims)
{
	for (int x=0; x<gaussIterations.x; ++x)
	{
		cudaMultAddFilter<<<curChunk->blocks,curChunk->threads>>>(*(deviceImages.getCurBuffer()),*(deviceImages.getNextBuffer()),
			Vec<size_t>(sizeconstKernelDims.x,1,1));
		deviceImages.incrementBuffer();
	}

	for (int y=0; y<gaussIterations.y; ++y)
	{
		cudaMultAddFilter<<<curChunk->blocks,curChunk->threads>>>(*(deviceImages.getCurBuffer()),*(deviceImages.getNextBuffer()),
			Vec<size_t>(1,sizeconstKernelDims.y,1),	sizeconstKernelDims.x);
		deviceImages.incrementBuffer();
	}

	for (int z=0; z<gaussIterations.z; ++z)
	{
		cudaMultAddFilter<<<curChunk->blocks,curChunk->threads>>>(*(deviceImages.getCurBuffer()),*(deviceImages.getNextBuffer()),
			Vec<size_t>(1,1,sizeconstKernelDims.z),	sizeconstKernelDims.y);
		deviceImages.incrementBuffer();
	}
}

void runMedianFilter(hipDeviceProp_t& deviceProp, std::vector<ImageChunk>::iterator curChunk, Vec<size_t> &neighborhood, 
					 CudaDeviceImages& deviceImages)
{
	dim3 blocks(curChunk->blocks);
	dim3 threads(curChunk->threads);
	double threadVolume = threads.x * threads.y * threads.z;
	double newThreadVolume = (double)deviceProp.sharedMemPerBlock/(sizeof(DevicePixelType)*neighborhood.product());

	double alpha = pow(threadVolume/newThreadVolume,1.0/3.0);
	threads.x = (unsigned int)(threads.x / alpha);
	threads.y = (unsigned int)(threads.y / alpha);
	threads.z = (unsigned int)(threads.z / alpha);

	blocks.x = (unsigned int)ceil((double)curChunk->getFullChunkSize().x / threads.x);
	blocks.y = (unsigned int)ceil((double)curChunk->getFullChunkSize().y / threads.y);
	blocks.z = (unsigned int)ceil((double)curChunk->getFullChunkSize().z / threads.z);

	size_t sharedMemorysize = neighborhood.product() * threads.x * threads.y * threads.z;

	cudaMedianFilter<<<blocks,threads,sharedMemorysize>>>(*(deviceImages.getCurBuffer()),*(deviceImages.getNextBuffer()),neighborhood);
	deviceImages.incrementBuffer();
}

DevicePixelType* CudaProcessBuffer::setUpOutIm(Vec<size_t> dims, DevicePixelType** imageOut)
{
	orgImageDims = dims;

	DevicePixelType* imOut;
	if (imageOut==NULL)
		imOut = new DevicePixelType[orgImageDims.product()];
	else
		imOut = *imageOut;

	return imOut;
}

//////////////////////////////////////////////////////////////////////////
//Cuda Operators (Alphabetical order)
//////////////////////////////////////////////////////////////////////////

DevicePixelType* CudaProcessBuffer::addConstant(const DevicePixelType* imageIn, Vec<size_t> dims, double additive,
												DevicePixelType** imageOut/*=NULL*/)
{
	DevicePixelType* imOut = setUpOutIm(dims, imageOut);

	DevicePixelType minVal = std::numeric_limits<DevicePixelType>::min();
	DevicePixelType maxVal = std::numeric_limits<DevicePixelType>::max();

	std::vector<ImageChunk> chunks = calculateBuffers(dims,2,(size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL),deviceProp);

	setMaxDeviceDims(chunks, maxDeviceDims);

	CudaDeviceImages deviceImages(2,maxDeviceDims,device);

	for (std::vector<ImageChunk>::iterator curChunk=chunks.begin(); curChunk!=chunks.end(); ++curChunk)
	{
		curChunk->sendROI(imageIn,dims,deviceImages.getCurBuffer());
		deviceImages.setNextDims(curChunk->getFullChunkSize());

		cudaAddFactor<<<curChunk->blocks,curChunk->threads>>>(*(deviceImages.getCurBuffer()),*(deviceImages.getNextBuffer()),
			additive,minVal,maxVal);

		deviceImages.incrementBuffer();

		curChunk->retriveROI(imOut,dims,deviceImages.getCurBuffer());
	}

	return imOut;
}

DevicePixelType* CudaProcessBuffer::addImageWith(const DevicePixelType* imageIn1, const DevicePixelType* imageIn2, Vec<size_t> dims,
													  double additive, DevicePixelType** imageOut/*=NULL*/)
{
	DevicePixelType* imOut = setUpOutIm(dims, imageOut);

	DevicePixelType minVal = std::numeric_limits<DevicePixelType>::min();
	DevicePixelType maxVal = std::numeric_limits<DevicePixelType>::max();

	std::vector<ImageChunk> chunks = calculateBuffers(dims,3,(size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL),deviceProp);

	setMaxDeviceDims(chunks, maxDeviceDims);

	CudaDeviceImages deviceImages(3,maxDeviceDims,device);

	for (std::vector<ImageChunk>::iterator curChunk=chunks.begin(); curChunk!=chunks.end(); ++curChunk)
	{
		deviceImages.setAllDims(curChunk->getFullChunkSize());
		curChunk->sendROI(imageIn1,dims,deviceImages.getCurBuffer());
		curChunk->sendROI(imageIn2,dims,deviceImages.getNextBuffer());

		cudaAddTwoImagesWithFactor<<<curChunk->blocks,curChunk->threads>>>(*(deviceImages.getCurBuffer()),*(deviceImages.getNextBuffer()),
			*(deviceImages.getThirdBuffer()),additive,minVal,maxVal);

		curChunk->retriveROI(imOut,dims,deviceImages.getThirdBuffer());
	}

	return imOut;
}

DevicePixelType* CudaProcessBuffer::applyPolyTransformation(const DevicePixelType* imageIn, Vec<size_t> dims, double a, double b, double c,
												DevicePixelType minValue, DevicePixelType maxValue, DevicePixelType** imageOut/*=NULL*/)
{
	DevicePixelType* imOut = setUpOutIm(dims, imageOut);

	std::vector<ImageChunk> chunks = calculateBuffers(dims,2,(size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL),deviceProp);
	
	setMaxDeviceDims(chunks, maxDeviceDims);

	CudaDeviceImages deviceImages(2,maxDeviceDims,device);

	for (std::vector<ImageChunk>::iterator curChunk=chunks.begin(); curChunk!=chunks.end(); ++curChunk)
	{
		curChunk->sendROI(imageIn,dims,deviceImages.getCurBuffer());
		deviceImages.setNextDims(curChunk->getFullChunkSize());

		cudaPolyTransferFuncImage<<<curChunk->blocks,curChunk->threads>>>(*(deviceImages.getCurBuffer()),*(deviceImages.getNextBuffer()),
			a,b,c,minValue,maxValue);

		deviceImages.incrementBuffer();

		curChunk->retriveROI(imOut,dims,deviceImages.getCurBuffer());
	}

	return imOut;
}

void CudaProcessBuffer::calculateMinMax(double& minValue, double& maxValue)
{
	throw std::logic_error("The method or operation is not implemented.");
}

DevicePixelType* CudaProcessBuffer::contrastEnhancement(const DevicePixelType* imageIn, Vec<size_t> dims, Vec<float> sigmas,
														Vec<size_t> neighborhood, DevicePixelType** imageOut/*=NULL*/)
{
	DevicePixelType* imOut = setUpOutIm(dims, imageOut);

	DevicePixelType minVal = std::numeric_limits<DevicePixelType>::min();
	DevicePixelType maxVal = std::numeric_limits<DevicePixelType>::max();

	Vec<int> gaussIterations(0,0,0);
	Vec<size_t> sizeconstKernelDims = createGaussianKernel(sigmas,hostKernel,gaussIterations);
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaConstKernel), hostKernel, sizeof(float)*
		(sizeconstKernelDims.x+sizeconstKernelDims.y+sizeconstKernelDims.z)));

	std::vector<ImageChunk> chunks = calculateBuffers(dims,3,(size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL),deviceProp,
		sizeconstKernelDims);

	setMaxDeviceDims(chunks, maxDeviceDims);

	CudaDeviceImages deviceImages(3,maxDeviceDims,device);
 
	for (std::vector<ImageChunk>::iterator curChunk=chunks.begin(); curChunk!=chunks.end(); ++curChunk)
	{
		deviceImages.setAllDims(curChunk->getFullChunkSize());

		curChunk->sendROI(imageIn,dims,deviceImages.getCurBuffer());

		runGaussIterations(gaussIterations, curChunk, deviceImages, sizeconstKernelDims);

		curChunk->sendROI(imageIn,dims,deviceImages.getNextBuffer());

		cudaAddTwoImagesWithFactor<<<curChunk->blocks,curChunk->threads>>>(*(deviceImages.getCurBuffer()),*(deviceImages.getNextBuffer()),
			*(deviceImages.getThirdBuffer()),-1.0,minVal,maxVal);

		deviceImages.setNthBuffCurent(3);

		runMedianFilter(deviceProp, curChunk, neighborhood, deviceImages);

		curChunk->retriveROI(imOut,dims,deviceImages.getCurBuffer());
	}

	return imOut;
}

size_t* CudaProcessBuffer::createHistogram(const DevicePixelType* imageIn, Vec<size_t> dims, int& arraySize)
{
	arraySize = NUM_BINS;
	size_t* hostHist = new size_t[arraySize];

	size_t* deviceHist;
	HANDLE_ERROR(hipMalloc((void**)&deviceHist,sizeof(size_t)*arraySize));
	HANDLE_ERROR(hipMemset(deviceHist,0,sizeof(size_t)*arraySize));

	std::vector<ImageChunk> chunks = calculateBuffers(dims,1,(size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL),deviceProp);
	setMaxDeviceDims(chunks, maxDeviceDims);
	CudaDeviceImages deviceImages(1,maxDeviceDims,device);

	for (std::vector<ImageChunk>::iterator curChunk=chunks.begin(); curChunk!=chunks.end(); ++curChunk)
	{
		curChunk->sendROI(imageIn,dims,deviceImages.getCurBuffer());
		
		cudaHistogramCreate<<<deviceProp.multiProcessorCount*2,arraySize,sizeof(size_t)*arraySize>>>(*(deviceImages.getCurBuffer()),
			deviceHist);
	}
	HANDLE_ERROR(hipMemcpy(hostHist,deviceHist,sizeof(size_t)*arraySize,hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipFree(deviceHist));

	return hostHist;
}

DevicePixelType* CudaProcessBuffer::gaussianFilter(const DevicePixelType* imageIn, Vec<size_t> dims, Vec<float> sigmas,
												   DevicePixelType** imageOut/*=NULL*/)
{
	DevicePixelType* imOut = setUpOutIm(dims, imageOut);

	Vec<int> gaussIterations(0,0,0);
	Vec<size_t> sizeconstKernelDims = createGaussianKernel(sigmas,hostKernel,gaussIterations);
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaConstKernel), hostKernel, sizeof(float)*
		(sizeconstKernelDims.x+sizeconstKernelDims.y+sizeconstKernelDims.z)));

	std::vector<ImageChunk> chunks = calculateBuffers(dims,2,(size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL),deviceProp,
		sizeconstKernelDims);

	setMaxDeviceDims(chunks, maxDeviceDims);

	CudaDeviceImages deviceImages(2,maxDeviceDims,device);

	for (std::vector<ImageChunk>::iterator curChunk=chunks.begin(); curChunk!=chunks.end(); ++curChunk)
	{
		deviceImages.setAllDims(curChunk->getFullChunkSize());

		curChunk->sendROI(imageIn,dims,deviceImages.getCurBuffer());

		runGaussIterations(gaussIterations, curChunk, deviceImages, sizeconstKernelDims);

		curChunk->retriveROI(imOut,dims,deviceImages.getCurBuffer());
	}

	return imOut;
}

void CudaProcessBuffer::mask(const DevicePixelType* imageMask, DevicePixelType threshold/*=1*/)
{
	throw std::logic_error("The method or operation is not implemented.");
}

void CudaProcessBuffer::maxFilter(Vec<size_t> neighborhood, double* kernel/*=NULL*/)
{
	throw std::logic_error("The method or operation is not implemented.");
}

void CudaProcessBuffer::maximumIntensityProjection()
{
	throw std::logic_error("The method or operation is not implemented.");
}

DevicePixelType* CudaProcessBuffer::meanFilter(const DevicePixelType* imageIn, Vec<size_t> dims, Vec<size_t> neighborhood,
											 DevicePixelType** imageOut/*=NULL*/)
{
	DevicePixelType* imOut = setUpOutIm(dims, imageOut);

	std::vector<ImageChunk> chunks = calculateBuffers(dims,2,(size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL),deviceProp,neighborhood);

	setMaxDeviceDims(chunks, maxDeviceDims);

	CudaDeviceImages deviceImages(2,maxDeviceDims,device);

	for (std::vector<ImageChunk>::iterator curChunk=chunks.begin(); curChunk!=chunks.end(); ++curChunk)
	{
		curChunk->sendROI(imageIn,dims,deviceImages.getCurBuffer());
		deviceImages.setNextDims(curChunk->getFullChunkSize());
		
		cudaMeanFilter<<<curChunk->blocks,curChunk->threads>>>(*(deviceImages.getCurBuffer()),*(deviceImages.getNextBuffer()),neighborhood);
		
		curChunk->retriveROI(imOut,dims,deviceImages.getCurBuffer());
	}
	
	return imOut;
}

DevicePixelType* CudaProcessBuffer::medianFilter(const DevicePixelType* imageIn, Vec<size_t> dims, Vec<size_t> neighborhood,
												 DevicePixelType** imageOut/*=NULL*/)
{
	DevicePixelType* imOut = setUpOutIm(dims, imageOut);

	std::vector<ImageChunk> chunks = calculateBuffers(dims,2,(size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL),deviceProp,neighborhood);

	setMaxDeviceDims(chunks, maxDeviceDims);

	CudaDeviceImages deviceImages(2,maxDeviceDims,device);

	for (std::vector<ImageChunk>::iterator curChunk=chunks.begin(); curChunk!=chunks.end(); ++curChunk)
	{
		curChunk->sendROI(imageIn,dims,deviceImages.getCurBuffer());
		deviceImages.setNextDims(curChunk->getFullChunkSize());

		runMedianFilter(deviceProp, curChunk, neighborhood, deviceImages);

		curChunk->retriveROI(imOut,dims,deviceImages.getCurBuffer());
	}

	return imOut;
}

void CudaProcessBuffer::minFilter(Vec<size_t> neighborhood, double* kernel/*=NULL*/)
{
	throw std::logic_error("The method or operation is not implemented.");
}

void CudaProcessBuffer::morphClosure(Vec<size_t> neighborhood, double* kernel/*=NULL*/)
{
	throw std::logic_error("The method or operation is not implemented.");
}

void CudaProcessBuffer::morphOpening(Vec<size_t> neighborhood, double* kernel/*=NULL*/)
{
	throw std::logic_error("The method or operation is not implemented.");
}

void CudaProcessBuffer::multiplyImage(double factor)
{
	throw std::logic_error("The method or operation is not implemented.");
}

void CudaProcessBuffer::multiplyImageWith(const DevicePixelType* image)
{
	throw std::logic_error("The method or operation is not implemented.");
}

double CudaProcessBuffer::normalizedCovariance(DevicePixelType* otherImage)
{
	throw std::logic_error("The method or operation is not implemented.");
	//return 0.0;
}

double* CudaProcessBuffer::normalizeHistogram(const DevicePixelType* imageIn, Vec<size_t> dims, int& arraySize)
{
	arraySize = NUM_BINS;
	double* hostHist = new double[arraySize];

	size_t* deviceHist;
	double* deviceHistNorm;
	HANDLE_ERROR(hipMalloc((void**)&deviceHist,sizeof(size_t)*arraySize));
	HANDLE_ERROR(hipMalloc((void**)&deviceHist,sizeof(double)*arraySize));
	HANDLE_ERROR(hipMemset(deviceHist,0,sizeof(size_t)*arraySize));

	std::vector<ImageChunk> chunks = calculateBuffers(dims,1,(size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL),deviceProp);
	setMaxDeviceDims(chunks, maxDeviceDims);
	CudaDeviceImages deviceImages(1,maxDeviceDims,device);

	for (std::vector<ImageChunk>::iterator curChunk=chunks.begin(); curChunk!=chunks.end(); ++curChunk)
	{
		curChunk->sendROI(imageIn,dims,deviceImages.getCurBuffer());

		cudaHistogramCreate<<<deviceProp.multiProcessorCount*2,arraySize,sizeof(size_t)*arraySize>>>(*(deviceImages.getCurBuffer()),
			deviceHist);
	}

	cudaNormalizeHistogram<<<arraySize,1>>>(deviceHist,deviceHistNorm,dims);

	HANDLE_ERROR(hipMemcpy(hostHist,deviceHist,sizeof(double)*arraySize,hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipFree(deviceHist));

	return hostHist;
}

void CudaProcessBuffer::otsuThresholdFilter(float alpha/*=1.0f*/)
{
	throw std::logic_error("The method or operation is not implemented.");
}


double CudaProcessBuffer::otsuThresholdValue(const DevicePixelType* imageIn, Vec<size_t> dims)
{
	int arraySize;
	double* hist = normalizeHistogram(imageIn,dims,arraySize);

	double thrsh = calcOtsuThreshold(hist,arraySize);

	delete[] hist;

	return thrsh;
}

void CudaProcessBuffer::imagePow(int p)
{
	throw std::logic_error("The method or operation is not implemented.");
}

void CudaProcessBuffer::sumArray(double& sum)
{
	throw std::logic_error("The method or operation is not implemented.");
}

DevicePixelType* CudaProcessBuffer::reduceImage(const DevicePixelType* imageIn, Vec<size_t> dims, Vec<size_t> reductions,
												Vec<size_t>& reducedDims, DevicePixelType** imageOut/*=NULL*/)
{
	orgImageDims = dims;
	reducedDims = orgImageDims / reductions;
	DevicePixelType* reducedImage;
	if (imageOut==NULL)
		reducedImage = new DevicePixelType[reducedDims.product()];
	else
		reducedImage = *imageOut;

	double ratio = (double)reducedDims.product() / dims.product();

	if (ratio==1.0)
	{
		memcpy(reducedImage,imageIn,sizeof(DevicePixelType)*reducedDims.product());
		return reducedImage;
	}

	std::vector<ImageChunk> orgChunks = calculateBuffers(dims,1,(size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL*(1-ratio)),deviceProp,reductions);
	std::vector<ImageChunk> reducedChunks = orgChunks;

	for (std::vector<ImageChunk>::iterator it=reducedChunks.begin(); it!=reducedChunks.end(); ++it)
	{
		it->imageStart = it->imageROIstart/reductions;
		it->chunkROIstart = Vec<size_t>(0,0,0);
		it->imageROIstart = it->imageROIstart/reductions;
		it->imageEnd = it->imageROIend/reductions;
		it->imageROIend = it->imageROIend/reductions;
		it->chunkROIend = it->imageEnd-it->imageStart;

		calcBlockThread(it->getFullChunkSize(),deviceProp,it->blocks,it->threads);
	}

	CudaImageContainerClean* deviceImageIn = new CudaImageContainerClean(orgChunks[0].getFullChunkSize(),device);
	CudaImageContainerClean* deviceImageOut = new CudaImageContainerClean(reducedChunks[0].getFullChunkSize(),device);

	std::vector<ImageChunk>::iterator orgIt = orgChunks.begin();
	std::vector<ImageChunk>::iterator reducedIt = reducedChunks.begin();

	while (orgIt!=orgChunks.end() && reducedIt!=reducedChunks.end())
	{
		orgIt->sendROI(imageIn,dims,deviceImageIn);
		deviceImageOut->setDims(reducedIt->getFullChunkSize());

		dim3 blocks(reducedIt->blocks);
		dim3 threads(reducedIt->threads);
 		double threadVolume = threads.x * threads.y * threads.z;
 		double newThreadVolume = (double)deviceProp.sharedMemPerBlock/(sizeof(DevicePixelType)*reductions.product());
 
 		double alpha = pow(threadVolume/newThreadVolume,1.0/3.0);
		threads.x = (unsigned int)(threads.x / alpha);
		threads.y = (unsigned int)(threads.y / alpha);
		threads.z = (unsigned int)(threads.z / alpha);

		if (threads.x*threads.y*threads.z>(unsigned int)deviceProp.maxThreadsPerBlock)
		{
			unsigned int maxThreads = (unsigned int)pow(deviceProp.maxThreadsPerBlock,1.0/3.0);
			threads.x = maxThreads;
			threads.y = maxThreads;
			threads.z = maxThreads;
		}
 
 		blocks.x = (unsigned int)ceil((double)reducedIt->getFullChunkSize().x / threads.x);
 		blocks.y = (unsigned int)ceil((double)reducedIt->getFullChunkSize().y / threads.y);
 		blocks.z = (unsigned int)ceil((double)reducedIt->getFullChunkSize().z / threads.z);
 
 		size_t sharedMemorysize = reductions.product() * threads.x * threads.y * threads.z;
 
 		cudaMedianImageReduction<<<blocks,threads,sharedMemorysize>>>(*deviceImageIn, *deviceImageOut, reductions);

		//cudaMeanImageReduction<<<blocks,threads>>>(*deviceImageIn,*deviceImageOut,reductions);

		reducedIt->retriveROI(reducedImage,reducedDims,deviceImageOut);
		
		++orgIt;
		++reducedIt;
	}

	delete deviceImageIn;
	delete deviceImageOut;

	hipDeviceReset();

 	return reducedImage;
}

void CudaProcessBuffer::thresholdFilter(double threshold)
{
	throw std::logic_error("The method or operation is not implemented.");
}

void CudaProcessBuffer::unmix(const DevicePixelType* image, Vec<size_t> neighborhood)
{
	throw std::logic_error("The method or operation is not implemented.");
}
