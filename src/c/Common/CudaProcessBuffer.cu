#include "hip/hip_runtime.h"
#include "CudaUtilities.cuh"
#include "CudaKernels.cuh"
#include "CudaProcessBuffer.cuh"
#include "CudaDeviceImages.cuh"
#include "CHelpers.h"

//Percent of memory that can be used on the device
const double MAX_MEM_AVAIL = 0.95;

std::vector<ImageChunk> calculateBuffers(Vec<size_t> imageDims, int numBuffersNeeded, size_t memAvailable, const hipDeviceProp_t& prop,
										 Vec<size_t> kernalDims/*=Vec<size_t>(0,0,0)*/)
{
	size_t numVoxels = (size_t)(memAvailable / (sizeof(HostPixelType)*numBuffersNeeded));

	Vec<size_t> overlapVolume;
	overlapVolume.x = kernalDims.x * imageDims.y * imageDims.z;
	overlapVolume.y = imageDims.x * kernalDims.y * imageDims.z;
	overlapVolume.z = imageDims.x * imageDims.y * kernalDims.z;

	Vec<size_t> deviceDims(0,0,0);

	if (overlapVolume.x>overlapVolume.y && overlapVolume.x>overlapVolume.z) // chunking in X is the worst
	{
		deviceDims.x = imageDims.x;
		double leftOver = (double)numVoxels/imageDims.x;
		double squareDim = sqrt(leftOver);

		if (overlapVolume.y<overlapVolume.z) // chunking in Y is second worst
		{
			if (squareDim>imageDims.y)
				deviceDims.y = imageDims.y;
			else 
				deviceDims.y = (size_t)squareDim;

			deviceDims.z = (size_t)(leftOver/deviceDims.y);

			if (deviceDims.z>imageDims.z)
				deviceDims.z = imageDims.z;
		}
		else // chunking in Z is second worst
		{
			if (squareDim>imageDims.z)
				deviceDims.z = imageDims.z;
			else 
				deviceDims.z = (size_t)squareDim;

			deviceDims.y = (size_t)(leftOver/deviceDims.z);

			if (deviceDims.y>imageDims.y)
				deviceDims.y = imageDims.y;
		}
	}
	else if (overlapVolume.y>overlapVolume.z) // chunking in Y is the worst
	{
		deviceDims.y = imageDims.y;
		double leftOver = (double)numVoxels/imageDims.y;
		double squareDim = sqrt(leftOver);

		if (overlapVolume.x<overlapVolume.z)
		{
			if (squareDim>imageDims.x)
				deviceDims.x = imageDims.x;
			else 
				deviceDims.x = (size_t)squareDim;

			deviceDims.z = (size_t)(leftOver/deviceDims.x);

			if (deviceDims.z>imageDims.z)
				deviceDims.z = imageDims.z;
		}
		else
		{
			if (squareDim>imageDims.z)
				deviceDims.z = imageDims.z;
			else 
				deviceDims.z = (size_t)squareDim;

			deviceDims.x = (size_t)(leftOver/deviceDims.z);

			if (deviceDims.x>imageDims.x)
				deviceDims.x = imageDims.x;
		}
	}
	else // chunking in Z is the worst
	{
		deviceDims.z = imageDims.z;
		double leftOver = (double)numVoxels/imageDims.z;
		double squareDim = sqrt(leftOver);

		if (overlapVolume.x<overlapVolume.y)
		{
			if (squareDim>imageDims.x)
				deviceDims.x = imageDims.x;
			else 
				deviceDims.x = (size_t)squareDim;

			deviceDims.y = (size_t)(leftOver/deviceDims.x);

			if (deviceDims.y>imageDims.y)
				deviceDims.y = imageDims.y;
		}
		else
		{
			if (squareDim>imageDims.y)
				deviceDims.y = imageDims.y;
			else 
				deviceDims.y = (size_t)squareDim;

			deviceDims.x = (size_t)(leftOver/deviceDims.z);

			if (deviceDims.x>imageDims.x)
				deviceDims.x = imageDims.x;
		}
	}

	return calculateChunking(imageDims, deviceDims, prop, kernalDims);
}

std::vector<ImageChunk> calculateChunking(Vec<size_t> orgImageDims, Vec<size_t> deviceDims, const hipDeviceProp_t& prop,
										  Vec<size_t> kernalDims/*=Vec<size_t>(0,0,0)*/)
{
	std::vector<ImageChunk> localChunks;
	Vec<size_t> margin((kernalDims + 1)/2); //integer round
	Vec<size_t> chunkDelta(deviceDims-margin*2);
	Vec<size_t> numChunks(1,1,1);

	if (orgImageDims.x>deviceDims.x)
		numChunks.x = (size_t)ceil((double)orgImageDims.x/chunkDelta.x);
	else
		chunkDelta.x = orgImageDims.x;

	if (orgImageDims.y>deviceDims.y)
		numChunks.y = (size_t)ceil((double)orgImageDims.y/chunkDelta.y);
	else
		chunkDelta.y = orgImageDims.y;

	if (orgImageDims.z>deviceDims.z)
		numChunks.z = (size_t)ceil((double)orgImageDims.z/chunkDelta.z);
	else
		chunkDelta.z = orgImageDims.z;

	localChunks.resize(numChunks.product());

	Vec<size_t> curChunk(0,0,0);
	Vec<size_t> imageStart(0,0,0);
	Vec<size_t> chunkROIstart(0,0,0);
	Vec<size_t> imageROIstart(0,0,0);
	Vec<size_t> imageEnd(0,0,0);
	Vec<size_t> chunkROIend(0,0,0);
	Vec<size_t> imageROIend(0,0,0);

	for (curChunk.z=0; curChunk.z<numChunks.z; ++curChunk.z)
	{
		for (curChunk.y=0; curChunk.y<numChunks.y; ++curChunk.y)
		{
			for (curChunk.x=0; curChunk.x<numChunks.x; ++curChunk.x)
			{
				imageROIstart = chunkDelta * curChunk;
				imageROIend = Vec<size_t>::min(imageROIstart + chunkDelta, orgImageDims);
				imageStart = Vec<size_t>(Vec<int>::max(Vec<int>(imageROIstart)-Vec<int>(margin), Vec<int>(0,0,0)));
				imageEnd = Vec<size_t>::min(imageROIend + margin, orgImageDims);
				chunkROIstart = imageROIstart - imageStart;
				chunkROIend = imageROIend - imageStart;

				ImageChunk* curImageBuffer = &localChunks[numChunks.linearAddressAt(curChunk)];

				curImageBuffer->imageStart = imageStart;
				curImageBuffer->chunkROIstart = chunkROIstart;
				curImageBuffer->imageROIstart = imageROIstart;
				curImageBuffer->imageEnd = imageEnd;
				curImageBuffer->chunkROIend = chunkROIend;
				curImageBuffer->imageROIend = imageROIend;

				calcBlockThread(curImageBuffer->getFullChunkSize(),prop,curImageBuffer->blocks,curImageBuffer->threads);
			}

			curChunk.x = 0;
		}

		curChunk.y = 0;
	}

	return localChunks;
}

CudaProcessBuffer::CudaProcessBuffer(int device/*=0*/)
{
	defaults();
	this->device = device;
	deviceSetup();
}

CudaProcessBuffer::~CudaProcessBuffer()
{
	defaults();
}

void CudaProcessBuffer::deviceSetup()
{
	HANDLE_ERROR(hipSetDevice(device));
	HANDLE_ERROR(hipGetDeviceProperties(&deviceProp,device));
}

void CudaProcessBuffer::defaults()
{
	device = 0;
	orgImageDims = Vec<size_t>(0,0,0);
	maxDeviceDims = Vec<size_t>(0,0,0);
}

//////////////////////////////////////////////////////////////////////////
// Helper Functions
//////////////////////////////////////////////////////////////////////////

void CudaProcessBuffer::setMaxDeviceDims(std::vector<ImageChunk> &chunks, Vec<size_t> &maxDeviceDims)
{
	maxDeviceDims = Vec<size_t>(0,0,0);

	for (std::vector<ImageChunk>::iterator curChunk=chunks.begin(); curChunk!=chunks.end(); ++curChunk)
	{
		Vec<size_t> curDim = curChunk->getFullChunkSize();

		if (curDim.x>maxDeviceDims.x)
			maxDeviceDims.x = curDim.x;

		if (curDim.y>maxDeviceDims.y)
			maxDeviceDims.y = curDim.y;

		if (curDim.z>maxDeviceDims.z)
			maxDeviceDims.z = curDim.z;
	}
}

void runGaussIterations(Vec<int> &gaussIterations, std::vector<ImageChunk>::iterator& curChunk, CudaDeviceImages& deviceImages,
						Vec<size_t> sizeconstKernelDims)
{
	for (int x=0; x<gaussIterations.x; ++x)
	{
		cudaMultAddFilter<<<curChunk->blocks,curChunk->threads>>>(*(deviceImages.getCurBuffer()),*(deviceImages.getNextBuffer()),
			Vec<size_t>(sizeconstKernelDims.x,1,1));
		DEBUG_KERNEL_CHECK();
		deviceImages.incrementBuffer();
	}

	for (int y=0; y<gaussIterations.y; ++y)
	{
		cudaMultAddFilter<<<curChunk->blocks,curChunk->threads>>>(*(deviceImages.getCurBuffer()),*(deviceImages.getNextBuffer()),
			Vec<size_t>(1,sizeconstKernelDims.y,1),	sizeconstKernelDims.x);
		DEBUG_KERNEL_CHECK();
		deviceImages.incrementBuffer();
	}

	for (int z=0; z<gaussIterations.z; ++z)
	{
		cudaMultAddFilter<<<curChunk->blocks,curChunk->threads>>>(*(deviceImages.getCurBuffer()),*(deviceImages.getNextBuffer()),
			Vec<size_t>(1,1,sizeconstKernelDims.z),	sizeconstKernelDims.y);
		DEBUG_KERNEL_CHECK();
		deviceImages.incrementBuffer();
	}
}

void runMedianFilter(hipDeviceProp_t& deviceProp, std::vector<ImageChunk>::iterator curChunk, Vec<size_t> &neighborhood, 
					 CudaDeviceImages& deviceImages)
{
	dim3 blocks(curChunk->blocks);
	dim3 threads(curChunk->threads);
	double threadVolume = threads.x * threads.y * threads.z;
	double newThreadVolume = (double)deviceProp.sharedMemPerBlock/(sizeof(DevicePixelType)*neighborhood.product());

	if (newThreadVolume<threadVolume)
	{
		double alpha = pow(threadVolume/newThreadVolume,1.0/3.0);
		threads.x = (unsigned int)(threads.x / alpha);
		threads.y = (unsigned int)(threads.y / alpha);
		threads.z = (unsigned int)(threads.z / alpha);
		threads.x = (threads.x>0) ? (threads.x) : (1);
		threads.y = (threads.y>0) ? (threads.y) : (1);
		threads.z = (threads.z>0) ? (threads.z) : (1);

		blocks.x = (unsigned int)ceil((double)curChunk->getFullChunkSize().x / threads.x);
		blocks.y = (unsigned int)ceil((double)curChunk->getFullChunkSize().y / threads.y);
		blocks.z = (unsigned int)ceil((double)curChunk->getFullChunkSize().z / threads.z);
	}

	size_t sharedMemorysize = neighborhood.product()*sizeof(DevicePixelType) * threads.x * threads.y * threads.z;

	cudaMedianFilter<<<blocks,threads,sharedMemorysize>>>(*(deviceImages.getCurBuffer()),*(deviceImages.getNextBuffer()),neighborhood);
	DEBUG_KERNEL_CHECK();
	deviceImages.incrementBuffer();
}

DevicePixelType* CudaProcessBuffer::setUpOutIm(Vec<size_t> dims, DevicePixelType** imageOut)
{
	orgImageDims = dims;

	DevicePixelType* imOut;
	if (imageOut==NULL)
		imOut = new DevicePixelType[orgImageDims.product()];
	else
		imOut = *imageOut;

	return imOut;
}

//////////////////////////////////////////////////////////////////////////
//Cuda Operators (Alphabetical order)
//////////////////////////////////////////////////////////////////////////

DevicePixelType* CudaProcessBuffer::addConstant(const DevicePixelType* imageIn, Vec<size_t> dims, double additive,
												DevicePixelType** imageOut/*=NULL*/)
{
	DevicePixelType* imOut = setUpOutIm(dims, imageOut);

	DevicePixelType minVal = std::numeric_limits<DevicePixelType>::lowest();
	DevicePixelType maxVal = std::numeric_limits<DevicePixelType>::max();

	std::vector<ImageChunk> chunks = calculateBuffers(dims,2,(size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL),deviceProp);

	setMaxDeviceDims(chunks, maxDeviceDims);

	CudaDeviceImages deviceImages(2,maxDeviceDims,device);

	for (std::vector<ImageChunk>::iterator curChunk=chunks.begin(); curChunk!=chunks.end(); ++curChunk)
	{
		curChunk->sendROI(imageIn,dims,deviceImages.getCurBuffer());
		deviceImages.setNextDims(curChunk->getFullChunkSize());

		cudaAddFactor<<<curChunk->blocks,curChunk->threads>>>(*(deviceImages.getCurBuffer()),*(deviceImages.getNextBuffer()),
			additive,minVal,maxVal);
		DEBUG_KERNEL_CHECK();

		deviceImages.incrementBuffer();

		curChunk->retriveROI(imOut,dims,deviceImages.getCurBuffer());
	}

	return imOut;
}

DevicePixelType* CudaProcessBuffer::addImageWith(const DevicePixelType* imageIn1, const DevicePixelType* imageIn2, Vec<size_t> dims,
													  double additive, DevicePixelType** imageOut/*=NULL*/)
{
	DevicePixelType* imOut = setUpOutIm(dims, imageOut);

	DevicePixelType minVal = std::numeric_limits<DevicePixelType>::lowest();
	DevicePixelType maxVal = std::numeric_limits<DevicePixelType>::max();

	std::vector<ImageChunk> chunks = calculateBuffers(dims,3,(size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL),deviceProp);

	setMaxDeviceDims(chunks, maxDeviceDims);

	CudaDeviceImages deviceImages(3,maxDeviceDims,device);

	for (std::vector<ImageChunk>::iterator curChunk=chunks.begin(); curChunk!=chunks.end(); ++curChunk)
	{
		deviceImages.setAllDims(curChunk->getFullChunkSize());
		curChunk->sendROI(imageIn1,dims,deviceImages.getCurBuffer());
		curChunk->sendROI(imageIn2,dims,deviceImages.getNextBuffer());

		cudaAddTwoImagesWithFactor<<<curChunk->blocks,curChunk->threads>>>(*(deviceImages.getCurBuffer()),*(deviceImages.getNextBuffer()),
			*(deviceImages.getThirdBuffer()),additive,minVal,maxVal);
		DEBUG_KERNEL_CHECK();

		curChunk->retriveROI(imOut,dims,deviceImages.getThirdBuffer());
	}

	return imOut;
}

DevicePixelType* CudaProcessBuffer::applyPolyTransformation(const DevicePixelType* imageIn, Vec<size_t> dims, double a, double b, double c,
												DevicePixelType minValue, DevicePixelType maxValue, DevicePixelType** imageOut/*=NULL*/)
{
	DevicePixelType* imOut = setUpOutIm(dims, imageOut);

	std::vector<ImageChunk> chunks = calculateBuffers(dims,2,(size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL),deviceProp);
	
	setMaxDeviceDims(chunks, maxDeviceDims);

	CudaDeviceImages deviceImages(2,maxDeviceDims,device);

	for (std::vector<ImageChunk>::iterator curChunk=chunks.begin(); curChunk!=chunks.end(); ++curChunk)
	{
		curChunk->sendROI(imageIn,dims,deviceImages.getCurBuffer());
		deviceImages.setNextDims(curChunk->getFullChunkSize());

		cudaPolyTransferFuncImage<<<curChunk->blocks,curChunk->threads>>>(*(deviceImages.getCurBuffer()),*(deviceImages.getNextBuffer()),
			a,b,c,minValue,maxValue);
		DEBUG_KERNEL_CHECK();

		deviceImages.incrementBuffer();

		curChunk->retriveROI(imOut,dims,deviceImages.getCurBuffer());
	}

	return imOut;
}

DevicePixelType* CudaProcessBuffer::contrastEnhancement(const DevicePixelType* imageIn, Vec<size_t> dims, Vec<float> sigmas,
														Vec<size_t> neighborhood, DevicePixelType** imageOut/*=NULL*/)
{
	DevicePixelType* imOut = setUpOutIm(dims, imageOut);

	DevicePixelType minVal = std::numeric_limits<DevicePixelType>::lowest();
	DevicePixelType maxVal = std::numeric_limits<DevicePixelType>::max();

	neighborhood = neighborhood.clamp(Vec<size_t>(1,1,1),dims);

	Vec<int> gaussIterations(0,0,0);
	Vec<size_t> sizeconstKernelDims = createGaussianKernel(sigmas,hostKernel,gaussIterations);
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaConstKernel), hostKernel, sizeof(float)*
		(sizeconstKernelDims.x+sizeconstKernelDims.y+sizeconstKernelDims.z)));

	std::vector<ImageChunk> chunks = calculateBuffers(dims,3,(size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL),deviceProp,
		sizeconstKernelDims);

	setMaxDeviceDims(chunks, maxDeviceDims);

	CudaDeviceImages deviceImages(3,maxDeviceDims,device);
 
	for (std::vector<ImageChunk>::iterator curChunk=chunks.begin(); curChunk!=chunks.end(); ++curChunk)
	{
		deviceImages.setAllDims(curChunk->getFullChunkSize());

		curChunk->sendROI(imageIn,dims,deviceImages.getCurBuffer());

		runGaussIterations(gaussIterations, curChunk, deviceImages, sizeconstKernelDims);

		curChunk->sendROI(imageIn,dims,deviceImages.getNextBuffer());

		cudaAddTwoImagesWithFactor<<<curChunk->blocks,curChunk->threads>>>(*(deviceImages.getCurBuffer()),*(deviceImages.getNextBuffer()),
			*(deviceImages.getThirdBuffer()),-1.0,minVal,maxVal);
		DEBUG_KERNEL_CHECK();

		deviceImages.setNthBuffCurent(3);

		runMedianFilter(deviceProp, curChunk, neighborhood, deviceImages);

		curChunk->retriveROI(imOut,dims,deviceImages.getCurBuffer());
	}

	return imOut;
}

size_t* CudaProcessBuffer::createHistogram(const DevicePixelType* imageIn, Vec<size_t> dims, int& arraySize)
{
	arraySize = NUM_BINS;
	size_t* hostHist = new size_t[arraySize];

	size_t* deviceHist;
	HANDLE_ERROR(hipMalloc((void**)&deviceHist,sizeof(size_t)*arraySize));
	HANDLE_ERROR(hipMemset(deviceHist,0,sizeof(size_t)*arraySize));

	std::vector<ImageChunk> chunks = calculateBuffers(dims,1,(size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL),deviceProp);
	setMaxDeviceDims(chunks, maxDeviceDims);
	CudaDeviceImages deviceImages(1,maxDeviceDims,device);

	for (std::vector<ImageChunk>::iterator curChunk=chunks.begin(); curChunk!=chunks.end(); ++curChunk)
	{
		curChunk->sendROI(imageIn,dims,deviceImages.getCurBuffer());
		
		cudaHistogramCreate<<<deviceProp.multiProcessorCount*2,arraySize,sizeof(size_t)*arraySize>>>(*(deviceImages.getCurBuffer()),
			deviceHist);
		DEBUG_KERNEL_CHECK();
	}
	HANDLE_ERROR(hipMemcpy(hostHist,deviceHist,sizeof(size_t)*arraySize,hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipFree(deviceHist));

	return hostHist;
}

DevicePixelType* CudaProcessBuffer::gaussianFilter(const DevicePixelType* imageIn, Vec<size_t> dims, Vec<float> sigmas,
												   DevicePixelType** imageOut/*=NULL*/)
{
	DevicePixelType* imOut = setUpOutIm(dims, imageOut);

	Vec<int> gaussIterations(0,0,0);
	sigmas.x = (dims.x==1) ? (0) : (sigmas.x);
	sigmas.y = (dims.y==1) ? (0) : (sigmas.y);
	sigmas.z = (dims.z==1) ? (0) : (sigmas.z);

	Vec<size_t> sizeconstKernelDims = createGaussianKernel(sigmas,hostKernel,gaussIterations);
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaConstKernel), hostKernel, sizeof(float)*
		(sizeconstKernelDims.x+sizeconstKernelDims.y+sizeconstKernelDims.z)));

	std::vector<ImageChunk> chunks = calculateBuffers(dims,2,(size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL),deviceProp,
		sizeconstKernelDims);

	setMaxDeviceDims(chunks, maxDeviceDims);

	CudaDeviceImages deviceImages(2,maxDeviceDims,device);

	for (std::vector<ImageChunk>::iterator curChunk=chunks.begin(); curChunk!=chunks.end(); ++curChunk)
	{
		deviceImages.setAllDims(curChunk->getFullChunkSize());

		curChunk->sendROI(imageIn,dims,deviceImages.getCurBuffer());

		runGaussIterations(gaussIterations, curChunk, deviceImages, sizeconstKernelDims);

		curChunk->retriveROI(imOut,dims,deviceImages.getCurBuffer());
	}

	return imOut;
}

DevicePixelType* CudaProcessBuffer::maxFilter(const DevicePixelType* imageIn, Vec<size_t> dims, Vec<size_t> kernalDims, float* kernel/*=NULL*/,
						   DevicePixelType** imageOut/*=NULL*/)
{
	DevicePixelType* imOut = setUpOutIm(dims, imageOut);

	DevicePixelType minVal = std::numeric_limits<DevicePixelType>::lowest();
	DevicePixelType maxVal = std::numeric_limits<DevicePixelType>::max();

	if (kernel==NULL)
	{
		kernalDims = kernalDims.clamp(Vec<size_t>(1,1,1),dims);
		float* ones = new float[kernalDims.product()];
		memset(ones,1,kernalDims.product());
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaConstKernel), ones, sizeof(float)*kernalDims.product()));
		delete[] ones;
	} 
	else
	{
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaConstKernel), kernel, sizeof(float)*kernalDims.product()));
	}

	std::vector<ImageChunk> chunks = calculateBuffers(dims,2,(size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL),deviceProp,kernalDims);

	setMaxDeviceDims(chunks, maxDeviceDims);

	CudaDeviceImages deviceImages(2,maxDeviceDims,device);

	for (std::vector<ImageChunk>::iterator curChunk=chunks.begin(); curChunk!=chunks.end(); ++curChunk)
	{
		curChunk->sendROI(imageIn,dims,deviceImages.getCurBuffer());
		deviceImages.setNextDims(curChunk->getFullChunkSize());

		cudaMaxFilter<<<curChunk->blocks,curChunk->threads>>>(*(deviceImages.getCurBuffer()),*(deviceImages.getNextBuffer()),kernalDims,
			minVal,maxVal);
		DEBUG_KERNEL_CHECK();

		deviceImages.incrementBuffer();

		curChunk->retriveROI(imOut,dims,deviceImages.getCurBuffer());
	}

	return imOut;
}

DevicePixelType* CudaProcessBuffer::meanFilter(const DevicePixelType* imageIn, Vec<size_t> dims, Vec<size_t> neighborhood,
											 DevicePixelType** imageOut/*=NULL*/)
{
	DevicePixelType* imOut = setUpOutIm(dims, imageOut);

	neighborhood = neighborhood.clamp(Vec<size_t>(1,1,1),dims);

	std::vector<ImageChunk> chunks = calculateBuffers(dims,2,(size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL),deviceProp,neighborhood);

	setMaxDeviceDims(chunks, maxDeviceDims);

	CudaDeviceImages deviceImages(2,maxDeviceDims,device);

	for (std::vector<ImageChunk>::iterator curChunk=chunks.begin(); curChunk!=chunks.end(); ++curChunk)
	{
		curChunk->sendROI(imageIn,dims,deviceImages.getCurBuffer());
		deviceImages.setNextDims(curChunk->getFullChunkSize());
		
		cudaMeanFilter<<<curChunk->blocks,curChunk->threads>>>(*(deviceImages.getCurBuffer()),*(deviceImages.getNextBuffer()),neighborhood);
		DEBUG_KERNEL_CHECK();

		deviceImages.incrementBuffer();
		
		curChunk->retriveROI(imOut,dims,deviceImages.getCurBuffer());
	}
	
	return imOut;
}

DevicePixelType* CudaProcessBuffer::medianFilter(const DevicePixelType* imageIn, Vec<size_t> dims, Vec<size_t> neighborhood,
												 DevicePixelType** imageOut/*=NULL*/)
{
	DevicePixelType* imOut = setUpOutIm(dims, imageOut);

	neighborhood = neighborhood.clamp(Vec<size_t>(1,1,1),dims);

	std::vector<ImageChunk> chunks = calculateBuffers(dims,2,(size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL),deviceProp,neighborhood);

	setMaxDeviceDims(chunks, maxDeviceDims);

	CudaDeviceImages deviceImages(2,maxDeviceDims,device);

	for (std::vector<ImageChunk>::iterator curChunk=chunks.begin(); curChunk!=chunks.end(); ++curChunk)
	{
		curChunk->sendROI(imageIn,dims,deviceImages.getCurBuffer());
		deviceImages.setNextDims(curChunk->getFullChunkSize());

		runMedianFilter(deviceProp, curChunk, neighborhood, deviceImages);

		curChunk->retriveROI(imOut,dims,deviceImages.getCurBuffer());
	}

	return imOut;
}

DevicePixelType* CudaProcessBuffer::minFilter(const DevicePixelType* imageIn, Vec<size_t> dims, Vec<size_t> kernalDims, float* kernel/*=NULL*/,
											  DevicePixelType** imageOut/*=NULL*/)
{
	DevicePixelType* imOut = setUpOutIm(dims, imageOut);

	DevicePixelType minVal = std::numeric_limits<DevicePixelType>::lowest();
	DevicePixelType maxVal = std::numeric_limits<DevicePixelType>::max();

	if (kernel==NULL)
	{
		kernalDims = kernalDims.clamp(Vec<size_t>(1,1,1),dims);
		float* ones = new float[kernalDims.product()];
		memset(ones,1,kernalDims.product());
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaConstKernel), ones, sizeof(float)*kernalDims.product()));
		delete[] ones;
	} 
	else
	{
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaConstKernel), kernel, sizeof(float)*kernalDims.product()));
	}

	std::vector<ImageChunk> chunks = calculateBuffers(dims,2,(size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL),deviceProp,kernalDims);

	setMaxDeviceDims(chunks, maxDeviceDims);

	CudaDeviceImages deviceImages(2,maxDeviceDims,device);

	for (std::vector<ImageChunk>::iterator curChunk=chunks.begin(); curChunk!=chunks.end(); ++curChunk)
	{
		curChunk->sendROI(imageIn,dims,deviceImages.getCurBuffer());
		deviceImages.setNextDims(curChunk->getFullChunkSize());

		cudaMinFilter<<<curChunk->blocks,curChunk->threads>>>(*(deviceImages.getCurBuffer()),*(deviceImages.getNextBuffer()),kernalDims,
			minVal,maxVal);
		DEBUG_KERNEL_CHECK();

		deviceImages.incrementBuffer();

		curChunk->retriveROI(imOut,dims,deviceImages.getCurBuffer());
	}

	return imOut;
}

DevicePixelType* CudaProcessBuffer::multiplyImage(const DevicePixelType* imageIn, Vec<size_t> dims, double multiplier, 
												  DevicePixelType** imageOut/*=NULL*/)
{
	DevicePixelType* imOut = setUpOutIm(dims, imageOut);

	DevicePixelType minVal = std::numeric_limits<DevicePixelType>::lowest();
	DevicePixelType maxVal = std::numeric_limits<DevicePixelType>::max();

	std::vector<ImageChunk> chunks = calculateBuffers(dims,2,(size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL),deviceProp);

	setMaxDeviceDims(chunks, maxDeviceDims);

	CudaDeviceImages deviceImages(2,maxDeviceDims,device);

	for (std::vector<ImageChunk>::iterator curChunk=chunks.begin(); curChunk!=chunks.end(); ++curChunk)
	{
		curChunk->sendROI(imageIn,dims,deviceImages.getCurBuffer());
		deviceImages.setNextDims(curChunk->getFullChunkSize());

		cudaMultiplyImage<<<curChunk->blocks,curChunk->threads>>>(*(deviceImages.getCurBuffer()),*(deviceImages.getNextBuffer()),
			multiplier,minVal,maxVal);
		DEBUG_KERNEL_CHECK();

		deviceImages.incrementBuffer();

		curChunk->retriveROI(imOut,dims,deviceImages.getCurBuffer());
	}

	return imOut;
}

DevicePixelType* CudaProcessBuffer::multiplyImageWith(const DevicePixelType* imageIn1, const DevicePixelType* imageIn2, Vec<size_t> dims,
													  double factor, DevicePixelType** imageOut/*=NULL*/)
{
	DevicePixelType* imOut = setUpOutIm(dims, imageOut);

	DevicePixelType minVal = std::numeric_limits<DevicePixelType>::lowest();
	DevicePixelType maxVal = std::numeric_limits<DevicePixelType>::max();

	std::vector<ImageChunk> chunks = calculateBuffers(dims,3,(size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL),deviceProp);

	setMaxDeviceDims(chunks, maxDeviceDims);

	CudaDeviceImages deviceImages(3,maxDeviceDims,device);

	for (std::vector<ImageChunk>::iterator curChunk=chunks.begin(); curChunk!=chunks.end(); ++curChunk)
	{
		deviceImages.setAllDims(curChunk->getFullChunkSize());
		curChunk->sendROI(imageIn1,dims,deviceImages.getCurBuffer());
		curChunk->sendROI(imageIn2,dims,deviceImages.getNextBuffer());

		cudaMultiplyTwoImages<<<curChunk->blocks,curChunk->threads>>>(*(deviceImages.getCurBuffer()),*(deviceImages.getNextBuffer()),
			*(deviceImages.getThirdBuffer()),factor,minVal,maxVal);
		DEBUG_KERNEL_CHECK();

		curChunk->retriveROI(imOut,dims,deviceImages.getThirdBuffer());
	}

	return imOut;
}

double CudaProcessBuffer::normalizedCovariance(const DevicePixelType* imageIn1, const DevicePixelType* imageIn2, Vec<size_t> dims)
{
// 	double im1Mean = sumArray(imageIn1,dims.product()) / dims.product();
// 	double im2Mean = sumArray(imageIn2,dims.product()) / dims.product();
// 
// 	DevicePixelType* im1Sub = addConstant(imageIn1,dims,-1.0*im1Mean);
// 	DevicePixelType* im2Sub = addConstant(imageIn2,dims,-1.0*im2Mean);
// 
// 	DevicePixelType* im1P = imagePow(im1Sub,dims,2.0);
// 	DevicePixelType* im2P = imagePow(im2Sub,dims,2.0);
// 
// 	double sigma1 = sqrt(sumArray(im1P,dims.product())/dims.product());
// 	double sigma2 = sqrt(sumArray(im2P,dims.product())/dims.product());
// 
// 	DevicePixelType* imMul = multiplyImageWith(im1Sub,im2Sub,dims,1.0);
// 	double numarator = sumArray(imMul,dims.product());
// 
// 	double coVar = numarator/(dims.product()*sigma1*sigma2);
// 
// 	delete[] im1Sub;
// 	delete[] im2Sub;
// 	delete[] im1P;
// 	delete[] im2P;
// 	delete[] imMul;
// 
// 	return coVar;

	return 0.0;
}

double* CudaProcessBuffer::normalizeHistogram(const DevicePixelType* imageIn, Vec<size_t> dims, int& arraySize)
{
	arraySize = NUM_BINS;
	double* hostHistNorm = new double[arraySize];

	size_t* deviceHist;
	double* deviceHistNorm;
	
	checkFreeMemory(sizeof(size_t)*arraySize+sizeof(double)*arraySize,device,true);

	HANDLE_ERROR(hipMalloc((void**)&deviceHist,sizeof(size_t)*arraySize));
	HANDLE_ERROR(hipMalloc((void**)&deviceHistNorm,sizeof(double)*arraySize));
	HANDLE_ERROR(hipMemset(deviceHist,0,sizeof(size_t)*arraySize));

	std::vector<ImageChunk> chunks = calculateBuffers(dims,1,(size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL),deviceProp);
	setMaxDeviceDims(chunks, maxDeviceDims);
	CudaDeviceImages deviceImages(1,maxDeviceDims,device);

	for (std::vector<ImageChunk>::iterator curChunk=chunks.begin(); curChunk!=chunks.end(); ++curChunk)
	{
		curChunk->sendROI(imageIn,dims,deviceImages.getCurBuffer());

		cudaHistogramCreate<<<deviceProp.multiProcessorCount*2,arraySize,sizeof(size_t)*arraySize>>>(*(deviceImages.getCurBuffer()),
			deviceHist);
		DEBUG_KERNEL_CHECK();
	}

	cudaNormalizeHistogram<<<arraySize,1>>>(deviceHist,deviceHistNorm,dims);
	DEBUG_KERNEL_CHECK();

	HANDLE_ERROR(hipMemcpy(hostHistNorm,deviceHistNorm,sizeof(double)*arraySize,hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipFree(deviceHist));

	return hostHistNorm;
}

DevicePixelType* CudaProcessBuffer::otsuThresholdFilter(const DevicePixelType* imageIn, Vec<size_t> dims, double alpha/*=1.0*/,
														DevicePixelType** imageOut/*=NULL*/)
{
	double thresh = otsuThresholdValue(imageIn,dims);
	thresh *= alpha;

	return thresholdFilter(imageIn,dims,(DevicePixelType)thresh,imageOut);
}

double CudaProcessBuffer::otsuThresholdValue(const DevicePixelType* imageIn, Vec<size_t> dims)
{
	int arraySize;
	double* hist = normalizeHistogram(imageIn,dims,arraySize);

	double thrsh = calcOtsuThreshold(hist,arraySize);

	delete[] hist;

	return thrsh;
}

DevicePixelType* CudaProcessBuffer::imagePow(const DevicePixelType* imageIn, Vec<size_t> dims, double power, DevicePixelType** imageOut/*=NULL*/)
{
	DevicePixelType* imOut = setUpOutIm(dims, imageOut);

	DevicePixelType minVal = std::numeric_limits<DevicePixelType>::lowest();
	DevicePixelType maxVal = std::numeric_limits<DevicePixelType>::max();

	std::vector<ImageChunk> chunks = calculateBuffers(dims,2,(size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL),deviceProp);

	setMaxDeviceDims(chunks, maxDeviceDims);

	CudaDeviceImages deviceImages(2,maxDeviceDims,device);

	for (std::vector<ImageChunk>::iterator curChunk=chunks.begin(); curChunk!=chunks.end(); ++curChunk)
	{
		curChunk->sendROI(imageIn,dims,deviceImages.getCurBuffer());
		deviceImages.setNextDims(curChunk->getFullChunkSize());

		cudaPow<<<curChunk->blocks,curChunk->threads>>>(*(deviceImages.getCurBuffer()),*(deviceImages.getNextBuffer()),
			power,minVal,maxVal);
		DEBUG_KERNEL_CHECK();

		deviceImages.incrementBuffer();

		curChunk->retriveROI(imOut,dims,deviceImages.getCurBuffer());
	}

	return imOut;
}

double CudaProcessBuffer::sumArray(const DevicePixelType* imageIn, size_t n)
{
	double sum = 0.0;
	double* deviceSum;
	double* hostSum;
	DevicePixelType* deviceImage;

	unsigned int blocks = deviceProp.multiProcessorCount;
	unsigned int threads = deviceProp.maxThreadsPerBlock;

	Vec<size_t> maxDeviceDims(1,1,1);

	maxDeviceDims.x = (n < (double)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL)/sizeof(DevicePixelType)) ? (n) :
		((size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL/sizeof(DevicePixelType)));

	checkFreeMemory(sizeof(DevicePixelType)*maxDeviceDims.x+sizeof(double)*blocks,device,true);
	HANDLE_ERROR(hipMalloc((void**)&deviceImage,sizeof(DevicePixelType)*maxDeviceDims.x));
	HANDLE_ERROR(hipMalloc((void**)&deviceSum,sizeof(double)*blocks));
	hostSum = new double[blocks];

	for (int i=0; i<ceil((double)n/maxDeviceDims.x); ++i)
	{
		const DevicePixelType* imStart = imageIn + i*maxDeviceDims.x;
		size_t numValues = ((i+1)*maxDeviceDims.x < n) ? (maxDeviceDims.x) : (n-i*maxDeviceDims.x);

		HANDLE_ERROR(hipMemcpy(deviceImage,imStart,sizeof(DevicePixelType)*numValues,hipMemcpyHostToDevice));

		cudaSumArray<<<blocks,threads,sizeof(double)*threads>>>(deviceImage,deviceSum,numValues);
		DEBUG_KERNEL_CHECK();

		HANDLE_ERROR(hipMemcpy(hostSum,deviceSum,sizeof(double)*blocks,hipMemcpyDeviceToHost));

		for (unsigned int i=0; i<blocks; ++i)
		{
			sum += hostSum[i];
		}
	}

	HANDLE_ERROR(hipFree(deviceSum));
	HANDLE_ERROR(hipFree(deviceImage));

	delete[] hostSum;

	return sum;
}

DevicePixelType* CudaProcessBuffer::reduceImage(const DevicePixelType* imageIn, Vec<size_t> dims, Vec<size_t> reductions,
												Vec<size_t>& reducedDims, DevicePixelType** imageOut/*=NULL*/)
{
	reductions = reductions.clamp(Vec<size_t>(1,1,1),dims);
	orgImageDims = dims;
	reducedDims = orgImageDims / reductions;
	DevicePixelType* reducedImage;
	if (imageOut==NULL)
		reducedImage = new DevicePixelType[reducedDims.product()];
	else
		reducedImage = *imageOut;

	double ratio = (double)reducedDims.product() / dims.product();

	if (ratio==1.0)
	{
		memcpy(reducedImage,imageIn,sizeof(DevicePixelType)*reducedDims.product());
		return reducedImage;
	}

	std::vector<ImageChunk> orgChunks = calculateBuffers(dims,1,(size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL*(1-ratio)),deviceProp,reductions);
	std::vector<ImageChunk> reducedChunks = orgChunks;

	for (std::vector<ImageChunk>::iterator it=reducedChunks.begin(); it!=reducedChunks.end(); ++it)
	{
		it->imageStart = it->imageROIstart/reductions;
		it->chunkROIstart = Vec<size_t>(0,0,0);
		it->imageROIstart = it->imageROIstart/reductions;
		it->imageEnd = it->imageROIend/reductions;
		it->imageROIend = it->imageROIend/reductions;
		it->chunkROIend = it->imageEnd-it->imageStart;

		calcBlockThread(it->getFullChunkSize(),deviceProp,it->blocks,it->threads);
	}

	CudaImageContainerClean* deviceImageIn = new CudaImageContainerClean(orgChunks[0].getFullChunkSize(),device);
	CudaImageContainerClean* deviceImageOut = new CudaImageContainerClean(reducedChunks[0].getFullChunkSize(),device);

	std::vector<ImageChunk>::iterator orgIt = orgChunks.begin();
	std::vector<ImageChunk>::iterator reducedIt = reducedChunks.begin();

	while (orgIt!=orgChunks.end() && reducedIt!=reducedChunks.end())
	{
		orgIt->sendROI(imageIn,dims,deviceImageIn);
		deviceImageOut->setDims(reducedIt->getFullChunkSize());

		dim3 blocks(reducedIt->blocks);
		dim3 threads(reducedIt->threads);
 		double threadVolume = threads.x * threads.y * threads.z;
 		double newThreadVolume = (double)deviceProp.sharedMemPerBlock/(sizeof(DevicePixelType)*reductions.product());
 
		if (newThreadVolume<threadVolume)
		{
			double alpha = pow(threadVolume/newThreadVolume,1.0/3.0);
			threads.x = (unsigned int)(threads.x / alpha);
			threads.y = (unsigned int)(threads.y / alpha);
			threads.z = (unsigned int)(threads.z / alpha);
			threads.x = (threads.x>0) ? (threads.x) : (1);
			threads.y = (threads.y>0) ? (threads.y) : (1);
			threads.z = (threads.z>0) ? (threads.z) : (1);

			if (threads.x*threads.y*threads.z>(unsigned int)deviceProp.maxThreadsPerBlock)
			{
				unsigned int maxThreads = (unsigned int)pow(deviceProp.maxThreadsPerBlock,1.0/3.0);
				threads.x = maxThreads;
				threads.y = maxThreads;
				threads.z = maxThreads;
			}


			blocks.x = (unsigned int)ceil((double)reducedIt->getFullChunkSize().x / threads.x);
			blocks.y = (unsigned int)ceil((double)reducedIt->getFullChunkSize().y / threads.y);
			blocks.z = (unsigned int)ceil((double)reducedIt->getFullChunkSize().z / threads.z);
		}
 
 		size_t sharedMemorysize = reductions.product()*sizeof(DevicePixelType) * threads.x * threads.y * threads.z;
 
 		cudaMedianImageReduction<<<blocks,threads,sharedMemorysize>>>(*deviceImageIn, *deviceImageOut, reductions);
		DEBUG_KERNEL_CHECK();

		reducedIt->retriveROI(reducedImage,reducedDims,deviceImageOut);
		
		++orgIt;
		++reducedIt;
	}

	delete deviceImageIn;
	delete deviceImageOut;

	hipDeviceReset();

 	return reducedImage;
}

DevicePixelType* CudaProcessBuffer::thresholdFilter(const DevicePixelType* imageIn, Vec<size_t> dims, DevicePixelType thresh,
													DevicePixelType** imageOut/*=NULL*/)
{
	DevicePixelType* imOut = setUpOutIm(dims, imageOut);

	DevicePixelType minVal = std::numeric_limits<DevicePixelType>::lowest();
	DevicePixelType maxVal = std::numeric_limits<DevicePixelType>::max();

	std::vector<ImageChunk> chunks = calculateBuffers(dims,2,(size_t)(deviceProp.totalGlobalMem*MAX_MEM_AVAIL),deviceProp);

	setMaxDeviceDims(chunks, maxDeviceDims);

	CudaDeviceImages deviceImages(2,maxDeviceDims,device);

	for (std::vector<ImageChunk>::iterator curChunk=chunks.begin(); curChunk!=chunks.end(); ++curChunk)
	{
		curChunk->sendROI(imageIn,dims,deviceImages.getCurBuffer());
		deviceImages.setNextDims(curChunk->getFullChunkSize());

		cudaThresholdImage<<<curChunk->blocks,curChunk->threads>>>(*(deviceImages.getCurBuffer()),*(deviceImages.getNextBuffer()),
			thresh,minVal,maxVal);
		DEBUG_KERNEL_CHECK();

		deviceImages.incrementBuffer();

		curChunk->retriveROI(imOut,dims,deviceImages.getCurBuffer());
	}

	return imOut;
}

void CudaProcessBuffer::unmix(const DevicePixelType* image, Vec<size_t> neighborhood)
{
	//neighborhood = neighborhood.clamp(Vec<size_t>(1,1,1),dims);
	throw std::logic_error("The method or operation is not implemented.");
}
