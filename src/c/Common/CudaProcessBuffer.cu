#include "hip/hip_runtime.h"
#include "CudaProcessBuffer.cuh"
#include "CudaUtilities.cuh"
#include "CudaStorageBuffer.cuh"
#include "CudaKernels.cuh"

CudaProcessBuffer::CudaProcessBuffer(int device/*=0*/)
{
	defaults();
	hostImageBuffers = NULL;
	deviceImageBuffers = NULL;
	this->device = device;
	deviceSetup();
}

// CudaProcessBuffer::CudaProcessBuffer(HostPixelType* imageIn, Vec<size_t> dims, int device/*=0*/)
// {
// 	defaults();
// 
// 	orgImageDims = dims;
// 	this->device = device;
// 
// 	calculateChunking();
// 	createBuffers();
// 
// 	loadImage(imageIn);
// }

CudaProcessBuffer::~CudaProcessBuffer()
{
	clearBuffers();
	defaults();
}

void CudaProcessBuffer::calculateChunking()
{
	size_t numVoxels = (size_t)(((double)deviceProp.totalGlobalMem/sizeof(HostPixelType))*0.4);

	if (orgImageDims.z==1)
	{
		if (orgImageDims.y==1)
		{
			if (orgImageDims.x<numVoxels)
			{
				chunkDims.x = orgImageDims.x;
				chunkDims.y = 1;
				chunkDims.z = 1;

				numChunks.x = 1;
				numChunks.y = 1;
				numChunks.z = 1;
			} 
			else
			{
				chunkDims.x = numVoxels;
				chunkDims.y = 1;
				chunkDims.z = 1;

				numChunks.x = (int)ceil((float)orgImageDims.x/numVoxels);
				numChunks.y = 1;
				numChunks.z = 1;
			}
		}
		else
		{
			if (orgImageDims.x*orgImageDims.y<numVoxels)
			{
				chunkDims.x = orgImageDims.x;
				chunkDims.y = orgImageDims.y;
				chunkDims.z = 1;

				numChunks.x = 1;
				numChunks.y = 1;
				numChunks.z = 1;
			} 
			else
			{
				int dim = (int)sqrt((double)numVoxels);

				chunkDims.x = dim;
				chunkDims.y = dim;
				chunkDims.z = 1;

				numChunks.x = (int)ceil((float)orgImageDims.x/dim);
				numChunks.y = (int)ceil((float)orgImageDims.y/dim);
				numChunks.z = 1;
			}
		}
	}
	else
	{
		if(orgImageDims.product()<numVoxels)
		{
			chunkDims.x = orgImageDims.x;
			chunkDims.y = orgImageDims.y;
			chunkDims.z = orgImageDims.z;

			numChunks.x = 1;
			numChunks.y = 1;
			numChunks.z = 1;
		}
		else
		{
			Vec<size_t> dims;
			size_t dim = (size_t)pow((float)numVoxels,1/3.0f);
			if (dim>orgImageDims.z)
			{
				dim = (int)sqrt((double)numVoxels/orgImageDims.z);
				dims.z = orgImageDims.z;
				dims.x = dim;
				dims.y = dim;
			}

			float extra = (float)(numVoxels-dims.x*dims.y*dims.z)/(dims.x*dims.y);

			chunkDims.x = dims.x + (int)extra;
			chunkDims.y = dims.y;
			chunkDims.z = dims.z;

			numChunks.x = (size_t)ceil((float)orgImageDims.x/chunkDims.x);
			numChunks.y = (size_t)ceil((float)orgImageDims.y/chunkDims.y);
			numChunks.z = (size_t)ceil((float)orgImageDims.z/chunkDims.z);
		}
	}
}

void CudaProcessBuffer::deviceSetup()
{
	HANDLE_ERROR(hipSetDevice(device));
	HANDLE_ERROR(hipGetDeviceProperties(&deviceProp,device));
	calculateChunking();
	updateBlockThread();
}

void CudaProcessBuffer::updateBlockThread()
{
	calcBlockThread(deviceDims,deviceProp,blocks,threads);
}

void CudaProcessBuffer::defaults()
{
	bufferSize = 0;
	device = 0;
	blocks = dim3(0,0,0);
	threads = dim3(0,0,0);
	orgImageDims = Vec<size_t>(0,0,0);
	chunkDims = Vec<size_t>(0,0,0);
	numChunks = Vec<size_t>(0,0,0);
	numDeviceBuffers = 0;
	deviceDims = Vec<size_t>(0,0,0);
}

void CudaProcessBuffer::createBuffers()
{
	hostImageBuffers = new ImageContainer*[numChunks.product()];
	Vec<size_t> curChunk(0,0,0);
	Vec<size_t> startIdx(0,0,0);
	Vec<size_t> curChunkDim;
// 	for (; curChunk.z<numChunks.z; ++curChunk.z)
// 	{
// 		if (curChunk.z*chunkDims.z+chunkDims.z<orgImageDims.z)
// 		{
// 			curChunkDim.z = chunkDims.z;
// 		}
// 		else
// 		{
// 			curChunkDim.z = orgImageDims.z - curChunk.z*chunkDims.z;
// 		}
// 
// 		for (curChunk.y=0; curChunk.y<numChunks.y; ++curChunk.y)
// 		{
// 			if (curChunk.y*chunkDims.y+chunkDims.y<orgImageDims.y)
// 			{
// 				curChunkDim.y = chunkDims.y;
// 			}
// 			else
// 			{
// 				curChunkDim.y = orgImageDims.y - curChunk.y*chunkDims.y;
// 			}
// 
// 			for (curChunk.x=0; curChunk.x<numChunks.x; ++curChunk.x)
// 			{
// 				if (curChunk.x*chunkDims.x+chunkDims.x<orgImageDims.x)
// 				{
// 					curChunkDim.x = chunkDims.x;
// 				}
// 				else
// 				{
// 					curChunkDim.x = orgImageDims.x - curChunk.x*chunkDims.x;
// 				}
// 
// 				hostImageBuffers[numChunks.linearAddressAt(curChunk)] = new ImageContainer(curChunkDim);
// 			}
// 		}
// 	}

// 	for (int i=0; i<NUM_DEVICE_BUFFERS; ++i)
// 		deviceImageBuffers[i] = new CudaImageContainerClean(chunkDims,device);
}

void CudaProcessBuffer::clearBuffers()
{
	if (hostImageBuffers!=NULL)
	{
		for (int i=0; i<numChunks.product(); ++i)
		{
			if (hostImageBuffers[i]!=NULL)
			{
				delete hostImageBuffers[i];
				hostImageBuffers[i] = NULL;
			}
		}

		delete[] hostImageBuffers;
		hostImageBuffers = NULL;
	}

	if (deviceImageBuffers!=NULL)
	{
		for (int i=0; i<numDeviceBuffers; ++i)
		{
			if (deviceImageBuffers[i]!=NULL)
			{
				delete deviceImageBuffers[i];
				deviceImageBuffers[i] = NULL;
			}
		}
		delete[] deviceImageBuffers;
		deviceImageBuffers = NULL;
	}
}

void CudaProcessBuffer::loadImage(HostPixelType* imageIn)
{
	if (numChunks.product() == 1)
	{
		hostImageBuffers[0]->setImagePointer(imageIn,orgImageDims);
		return;
	}
// 
// 	Vec<size_t> curChunk(0,0,0);
// 	Vec<size_t> curChunkDim;
// 	for (; curChunk.z<numChunks.z; ++curChunk.z)
// 	{
// 		if (curChunk.z*chunkDims.z+chunkDims.z<orgImageDims.z)
// 		{
// 			curChunkDim.z = chunkDims.z;
// 		}
// 		else
// 		{
// 			curChunkDim.z = orgImageDims.z - curChunk.z*chunkDims.z;
// 		}
// 
// 		for (curChunk.y=0; curChunk.y<numChunks.y; ++curChunk.y)
// 		{
// 			if (curChunk.y*chunkDims.y+chunkDims.y<orgImageDims.y)
// 			{
// 				curChunkDim.y = chunkDims.y;
// 			}
// 			else
// 			{
// 				curChunkDim.y = orgImageDims.y - curChunk.y*chunkDims.y;
// 			}
// 
// 			for (curChunk.x=0; curChunk.x<numChunks.x; ++curChunk.x)
// 			{
// 				if (curChunk.x*chunkDims.x+chunkDims.x<orgImageDims.x)
// 				{
// 					curChunkDim.x = chunkDims.x;
// 				}
// 				else
// 				{
// 					curChunkDim.x = orgImageDims.x - curChunk.x*chunkDims.x;
// 				}
// 
// 				HostPixelType* im = hostImageBuffers[numChunks.linearAddressAt(curChunk)]->getMemoryPointer();
// 				Vec<size_t> startIdx(chunkDims.x*curChunk.x,chunkDims.y*curChunk.y,chunkDims.z*curChunk.z);
// 				Vec<size_t> curIdx(startIdx);
// 
// 				for (; curIdx.z<startIdx.z+chunkDims.z && curIdx.z<curChunkDim.z; ++curIdx.z)
// 				{
// 					for(curIdx.y=startIdx.y; curIdx.y<startIdx.y+chunkDims.y && curIdx.y<curChunkDim.y; ++curIdx.y)
// 					{
// 						memcpy(im+chunkDims.linearAddressAt(curIdx-startIdx),imageIn+orgImageDims.linearAddressAt(curIdx),
// 							sizeof(HostPixelType)*curChunkDim.x);
// 					}
// 				}
// 			}
// 		}
// 	}
}

void CudaProcessBuffer::addConstant(double additive)
{

}

void CudaProcessBuffer::addImageWith(const DevicePixelType* image, double factor)
{

}

void CudaProcessBuffer::applyPolyTransformation(double a, double b, double c, DevicePixelType minValue, DevicePixelType maxValue)
{

}

void CudaProcessBuffer::calculateMinMax(double& minValue, double& maxValue)
{

}

void CudaProcessBuffer::contrastEnhancement(Vec<float> sigmas, Vec<size_t> medianNeighborhood)
{

}

void CudaProcessBuffer::createHistogram()
{

}

void CudaProcessBuffer::gaussianFilter(Vec<float> sigmas)
{

}

void CudaProcessBuffer::mask(const DevicePixelType* imageMask, DevicePixelType threshold/*=1*/)
{

}

void CudaProcessBuffer::maxFilter(Vec<size_t> neighborhood, double* kernel/*=NULL*/)
{

}

void CudaProcessBuffer::maximumIntensityProjection()
{

}

DevicePixelType* CudaProcessBuffer::meanFilter(DevicePixelType* imageIn, Vec<size_t> dims, Vec<size_t> neighborhood,
											 DevicePixelType** imageOut/*=NULL*/)
{
	DevicePixelType* meanImage;
	
	createDeviceBuffers(dims, 2);
	
	if (dims==deviceDims)
		deviceImageBuffers[0]->loadImage(imageIn,dims);
	else
		;

	if (imageOut==NULL)
		meanImage = new DevicePixelType[deviceDims.product()];
	else
		meanImage = *imageOut;

	cudaMeanFilter<<<blocks,threads>>>(*(deviceImageBuffers[0]),*(deviceImageBuffers[1]),neighborhood);

	HANDLE_ERROR(hipMemcpy(meanImage,deviceImageBuffers[1]->getDeviceImagePointer(),sizeof(DevicePixelType)*deviceDims.product(),hipMemcpyDeviceToHost));

	return meanImage;
}

void CudaProcessBuffer::medianFilter(Vec<size_t> neighborhood)
{

}

void CudaProcessBuffer::minFilter(Vec<size_t> neighborhood, double* kernel/*=NULL*/)
{

}

void CudaProcessBuffer::morphClosure(Vec<size_t> neighborhood, double* kernel/*=NULL*/)
{

}

void CudaProcessBuffer::morphOpening(Vec<size_t> neighborhood, double* kernel/*=NULL*/)
{

}

void CudaProcessBuffer::multiplyImage(double factor)
{

}

void CudaProcessBuffer::multiplyImageWith(const DevicePixelType* image)
{

}

double CudaProcessBuffer::normalizedCovariance(DevicePixelType* otherImage)
{
	return 0.0;
}

void CudaProcessBuffer::normalizeHistogram()
{

}

void CudaProcessBuffer::otsuThresholdFilter(float alpha/*=1.0f*/)
{

}

void CudaProcessBuffer::imagePow(int p)
{

}

void CudaProcessBuffer::sumArray(double& sum)
{

}

void CudaProcessBuffer::reduceImage(Vec<double> reductions)
{

}

void CudaProcessBuffer::thresholdFilter(double threshold)
{

}

void CudaProcessBuffer::unmix(const DevicePixelType* image, Vec<size_t> neighborhood)
{

}

void CudaProcessBuffer::createDeviceBuffers(Vec<size_t> dims, int numBuffersNeeded)
{
	deviceImageBuffers = new CudaImageContainerClean*[numBuffersNeeded];
	numDeviceBuffers = numBuffersNeeded;

	size_t numVoxels = (size_t)((double)deviceProp.totalGlobalMem*0.9/(sizeof(HostPixelType)*numBuffersNeeded));
	deviceDims = Vec<size_t>(0,0,dims.z);
	double leftOver = (double)numVoxels/dims.z;

	double squareDim = sqrt(leftOver);
	if (squareDim>dims.y)
	{
		deviceDims.y = dims.y;
		deviceDims.x = (size_t)(leftOver/dims.y);
		if (deviceDims.x>dims.x)
			deviceDims.x = dims.x;
	}
	else 
	{
		deviceDims.x = (size_t)squareDim;
		deviceDims.y = (size_t)squareDim;
	}

	for (int i=0; i<numBuffersNeeded; ++i)
	{
		deviceImageBuffers[i] = new CudaImageContainerClean(deviceDims,device);
	}

	updateBlockThread();
}

	// 	ImagePixelType otsuThresholdValue()
	// 	{
	// 		int temp;
	// 		return calcOtsuThreshold(retrieveNormalizedHistogram(temp),NUM_BINS);
	// 	}
	// 
	// 	ImagePixelType* retrieveImage(ImagePixelType* imageOut=NULL)
	// 	{
	// 		if (currentBuffer<0 || currentBuffer>NUM_BUFFERS)
	// 		{
	// 			return NULL;
	// 		}
	// 		if (imageOut==NULL)
	// 			imageOut = new ImagePixelType[imageDims.product()];
	// 
	// 		const DevicePixelType* deviceImage = getCurrentBuffer()->getConstImagePointer();
	// 
	// 		HANDLE_ERROR(hipMemcpy(imageOut,deviceImage,sizeof(ImagePixelType)*imageDims.product(),hipMemcpyDeviceToHost));
	// 		return imageOut;
	// 	}
	// 
	// 	void retrieveImage(ImageContainer* imageOut)
	// 	{
	// 		if (currentBuffer<0 || currentBuffer>NUM_BUFFERS)
	// 		{
	// 			return;
	// 		}
	// 
	// 		HANDLE_ERROR(hipMemcpy(imageOut->getMemoryPointer(),getCurrentBuffer(),sizeof(ImagePixelType)*imageDims.product(),
	// 			hipMemcpyDeviceToHost));
	// 	}
	// 
	// 	/*
	// 	*	Returns a host pointer to the histogram data
	// 	*	This is destroyed when this' destructor is called
	// 	*	Will call the needed histogram creation methods if not all ready
	// 	*/
	// 	size_t* retrieveHistogram(int& returnSize)
	// 	{
	// 		if (!isCurrentNormHistogramHost)
	// 		{
	// 			createHistogram();
	// 
	// 			HANDLE_ERROR(hipMemcpy(histogramHost,histogramDevice,sizeof(size_t)*NUM_BINS,hipMemcpyDeviceToHost));
	// 			isCurrentHistogramHost = true;
	// 		}
	// 
	// 		returnSize = NUM_BINS;
	// 
	// 		return histogramHost;
	// 	}
	// 
	// 	/*
	// 	*	Returns a host pointer to the normalized histogram data
	// 	*	This is destroyed when this' destructor is called
	// 	*	Will call the needed histogram creation methods if not all ready
	// 	*/
	// 	double* retrieveNormalizedHistogram(int& returnSize)
	// 	{
	// 		if (!isCurrentNormHistogramHost)
	// 		{
	// 			normalizeHistogram();
	// 
	// 			HANDLE_ERROR(hipMemcpy(normalizedHistogramHost,normalizedHistogramDevice,sizeof(double)*NUM_BINS,hipMemcpyDeviceToHost));
	// 			isCurrentNormHistogramHost = true;
	// 		}
	// 
	// 		returnSize = NUM_BINS;
	// 
	// 		return normalizedHistogramHost;
	// 	}
	// 
	// 	ImagePixelType* retrieveReducedImage(Vec<size_t>& reducedDims)
	// 	{
	// 		reducedDims = this->reducedDims;
	// 
	// 		if (reducedImageDevice!=NULL)
	// 		{
	// 			HANDLE_ERROR(hipMemcpy(reducedImageHost,reducedImageDevice,sizeof(ImagePixelType)*reducedDims.product(),hipMemcpyDeviceToHost));
	// 		}
	// 
	// 		return reducedImageHost;
	// 	}
	// 
	// 	Vec<size_t> getDimension() const {return imageDims;}
	// 	int getDevice() const {return device;}
	// 	size_t getBufferSize() {return bufferSize;}
	// 
	// 	/*
	// 	*	This will replace this' cuda image buffer with the region of interest
	// 	*	from the passed in buffer.
	// 	*	****ENSURE that this' original size is big enough to accommodates the
	// 	*	the new buffer size.  Does not do error checking thus far.
	// 	*/
	// 	void copyROI(const CudaProcessBuffer<ImagePixelType>* image, Vec<size_t> starts, Vec<size_t> sizes)
	// 	{
	// 		if (sizes.product()>bufferSize || this->device!=image->getDevice())
	// 		{
	// 			clean();
	// 			this->device = image->getDevice();
	// 			imageDims = sizes;
	// 			deviceSetup();
	// 			memoryAllocation();
	// 		}
	// 
	// 		imageDims = sizes;
	// 		currentBuffer = 0;
	// 		image->getRoi(getCurrentBuffer(),starts,sizes);
	// 		updateBlockThread();
	// 	}
	// 
	// 	void copyROI(const CudaStorageBuffer<ImagePixelType>* imageIn, Vec<size_t> starts, Vec<size_t> sizes)
	// 	{
	// 		if ((size_t)sizes.product()>bufferSize || this->device!=imageIn->getDevice())
	// 		{
	// 			clean();
	// 			this->device = imageIn->getDevice();
	// 			imageDims = sizes;
	// 			deviceSetup();
	// 			memoryAllocation();
	// 		}
	// 
	// 		imageDims = sizes;
	// 		currentBuffer = 0;
	// 		imageIn->getRoi(getCurrentBuffer(),starts,sizes);
	// 		updateBlockThread();	
	// 	}
	// 
	// 	void copyImage(const CudaProcessBuffer<ImagePixelType>* bufferIn)
	// 	{
	// 		if (bufferIn->getDimension().product()>bufferSize)
	// 		{
	// 			clean();
	// 			this->device = device;
	// 			imageDims = bufferIn->getDimension();
	// 			deviceSetup();
	// 			memoryAllocation();
	// 		}
	// 
	// 		imageDims = bufferIn->getDimension();
	// 		device = bufferIn->getDevice();
	// 		updateBlockThread();
	// 
	// 		currentBuffer = 0;
	// 		HANDLE_ERROR(hipMemcpy(getCurrentBuffer(),bufferIn->getCudaBuffer(),sizeof(ImagePixelType)*imageDims.product(),
	// 			hipMemcpyDeviceToDevice));
	// 	}
	// 
	// 	const CudaImageContainer* getCudaBuffer() const
	// 	{
	// 		return getCurrentBuffer();
	// 	}
	// 
	// 	size_t getMemoryUsed() {return memoryUsage;}
	// 	size_t getGlobalMemoryAvailable() {return deviceProp.totalGlobalMem;}
	// 


//void memoryAllocation();
// 	{
// 		assert(sizeof(ImagePixelType)*imageDims.product()*NUM_BUFFERS < deviceProp.totalGlobalMem*.8);
// 
// 		for (int i=0; i<NUM_BUFFERS; ++i)
// 		{
// 			imageBuffers[i] = new CudaImageContainerClean(imageDims,device);
// 		}
// 
// 		currentBuffer = -1;
// 		bufferSize = imageDims.product();
// 
// 		updateBlockThread();
// 
// 		sizeSum = sumBlocks.x;
// 		HANDLE_ERROR(hipMalloc((void**)&deviceSum,sizeof(double)*sumBlocks.x));
// 		memoryUsage += sizeof(double)*sumBlocks.x;
// 		hostSum = new double[sumBlocks.x];
// 
// 		HANDLE_ERROR(hipMalloc((void**)&minValuesDevice,sizeof(double)*sumBlocks.x));
// 		memoryUsage += sizeof(double)*sumBlocks.x;
// 
// 		histogramHost = new size_t[NUM_BINS];
// 		HANDLE_ERROR(hipMalloc((void**)&histogramDevice,NUM_BINS*sizeof(size_t)));
// 		memoryUsage += NUM_BINS*sizeof(size_t);
// 
// 		normalizedHistogramHost = new double[NUM_BINS];
// 		HANDLE_ERROR(hipMalloc((void**)&normalizedHistogramDevice,NUM_BINS*sizeof(double)));
// 		memoryUsage += NUM_BINS*sizeof(double);
// 
// 		minPixel = std::numeric_limits<ImagePixelType>::min();
// 		maxPixel = std::numeric_limits<ImagePixelType>::max();
// 	}
// 
// 	void setStatus( Vec<size_t> dims )
// 	{
// 		if (dims.product()>bufferSize)
// 		{
// 			int device = this->device;
// 			clean();
// 			this->device = device;
// 			imageDims = dims;
// 			deviceSetup();
// 			memoryAllocation();
// 		}
// 		else
// 		{
// 			isCurrentHistogramHost = false;
// 			isCurrentHistogramDevice = false;
// 			isCurrentNormHistogramHost = false;
// 			isCurrentNormHistogramDevice = false;
// 		}
// 
// 		imageDims = dims;
// 		currentBuffer = 0;
// 		reservedBuffer = -1;
// 	}
// 
// 	void getRoi(ImagePixelType* roi, Vec<size_t> starts, Vec<size_t> sizes) const
// 	{
// #if CUDA_CALLS_ON
// 		cudaGetROI<<<blocks,threads>>>(*getCurrentBuffer(),roi,starts,sizes);
// #endif
// 	}
// 
// 	void copy(const CudaProcessBuffer<ImagePixelType>* bufferIn)
// 	{
// 		defaults();
// 
// 		imageDims = bufferIn->getDimension();
// 		device = bufferIn->getDevice();
// 
// 		deviceSetup();
// 		memoryAllocation();
// 
// 		currentBuffer = 0;
// 		ImagePixelType* inImage = bufferIn->getCurrentBuffer();
// 
// 		if (inImage!=NULL)
// 			HANDLE_ERROR(hipMemcpy(imageBuffers[currentBuffer],inImage,sizeof(ImagePixelType)*imageDims.product(),hipMemcpyDeviceToDevice));
// 
// 		if (bufferIn->reducedImageHost!=NULL)
// 			memcpy(reducedImageHost,bufferIn->reducedImageHost,sizeof(ImagePixelType)*reducedDims.product());
// 
// 		if (bufferIn->reducedImageDevice!=NULL)
// 			HANDLE_ERROR(hipMemcpy(reducedImageDevice,bufferIn->reducedImageDevice,sizeof(ImagePixelType)*reducedDims.product(),
// 			hipMemcpyDeviceToDevice));
// 
// 		if (bufferIn->histogramHost!=NULL)
// 			memcpy(histogramHost,bufferIn->histogramHost,sizeof(size_t)*imageDims.product());
// 
// 		if (bufferIn->histogramDevice!=NULL)
// 			HANDLE_ERROR(hipMemcpy(histogramDevice,bufferIn->histogramDevice,sizeof(size_t)*NUM_BINS,hipMemcpyDeviceToDevice));
// 
// 		if (bufferIn->normalizedHistogramHost!=NULL)
// 			memcpy(normalizedHistogramHost,bufferIn->normalizedHistogramHost,sizeof(double)*imageDims.product());
// 
// 		if (bufferIn->normalizedHistogramDevice!=NULL)
// 			HANDLE_ERROR(hipMemcpy(normalizedHistogramDevice,bufferIn->normalizedHistogramDevice,sizeof(double)*NUM_BINS,
// 			hipMemcpyDeviceToDevice));
// 	}
// 
// 	void constKernelOnes()
// 	{
// 		memset(hostKernel,1,sizeof(float)*MAX_KERNEL_DIM*MAX_KERNEL_DIM*MAX_KERNEL_DIM);
// 		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaConstKernel),hostKernel,sizeof(float)*MAX_KERNEL_DIM*MAX_KERNEL_DIM*MAX_KERNEL_DIM));
// 	}
// 
// 	void constKernelZeros()
// 	{
// 		memset(hostKernel,1,sizeof(float)*MAX_KERNEL_DIM*MAX_KERNEL_DIM*MAX_KERNEL_DIM);
// 		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaConstKernel),hostKernel,sizeof(float)*MAX_KERNEL_DIM*MAX_KERNEL_DIM*MAX_KERNEL_DIM));
// 	}
// 
// 	void setConstKernel(double* kernel, Vec<size_t> kernelDims)
// 	{
// 		memset(hostKernel,0,sizeof(float)*MAX_KERNEL_DIM*MAX_KERNEL_DIM*MAX_KERNEL_DIM);
// 
// 		Vec<size_t> coordinate(0,0,0);
// 		for (; coordinate.x<kernelDims.x; ++coordinate.x)
// 		{
// 			coordinate.y = 0;
// 			for (; coordinate.y<kernelDims.y; ++coordinate.y)
// 			{
// 				coordinate.z = 0;
// 				for (; coordinate.z<kernelDims.z; ++coordinate.z)
// 				{
// 					hostKernel[kernelDims.linearAddressAt(coordinate)] = (float)kernel[kernelDims.linearAddressAt(coordinate)];
// 				}
// 			}
// 		}
// 		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaConstKernel),hostKernel,sizeof(float)*kernelDims.product()));
// 	}
// 
// 	void defaults()
// 	{
// 		imageDims = UNSET;
// 		reducedDims = UNSET;
// 		constKernelDims = UNSET;
// 		gausKernelSigmas  = Vec<float>(0.0f,0.0f,0.0f);
// 		device = -1;
// 		currentBuffer = -1;
// 		bufferSize = 0;
// 		for (int i=0; i<NUM_BUFFERS; ++i)
// 		{
// 			imageBuffers[i] = NULL;
// 		}
// 
// 		reducedImageHost = NULL;
// 		reducedImageDevice = NULL;
// 		histogramHost = NULL;
// 		histogramDevice = NULL;
// 		normalizedHistogramHost = NULL;
// 		normalizedHistogramDevice = NULL;
// 		isCurrentHistogramHost = false;
// 		isCurrentHistogramDevice = false;
// 		isCurrentNormHistogramHost = false;
// 		isCurrentNormHistogramDevice = false;
// 		deviceSum = NULL;
// 		minValuesDevice = NULL;
// 		hostSum = NULL;
// 		gaussIterations = Vec<int>(0,0,0);
// 		reservedBuffer = -1;
// 		memoryUsage = 0;
// 	}
// 
// 	void clean() 
// 	{
// 		for (int i=0; i<NUM_BUFFERS && imageBuffers!=NULL; ++i)
// 		{
// 			if (imageBuffers[i]!=NULL)
// 				delete imageBuffers[i];
// 		}
// 
// 		if (reducedImageHost!=NULL)
// 			delete reducedImageHost;
// 
// 		if (reducedImageDevice!=NULL)
// 			delete reducedImageDevice;
// 
// 		if (histogramHost!=NULL)
// 			delete[] histogramHost;
// 
// 		if (histogramDevice!=NULL)
// 			HANDLE_ERROR(hipFree(histogramDevice));
// 
// 		if (normalizedHistogramHost!=NULL)
// 			delete[] normalizedHistogramHost;
// 
// 		if (normalizedHistogramDevice!=NULL)
// 			HANDLE_ERROR(hipFree(normalizedHistogramDevice));
// 
// 		if (deviceSum!=NULL)
// 			HANDLE_ERROR(hipFree(deviceSum));
// 
// 		if (hostSum!=NULL)
// 			delete[] hostSum;
// 
// 		if (minValuesDevice!=NULL)
// 			HANDLE_ERROR(hipFree(minValuesDevice));
// 
// 		memset(hostKernel,0,sizeof(float)*MAX_KERNEL_DIM*MAX_KERNEL_DIM*MAX_KERNEL_DIM);
// 
// 		defaults();
// 	}
// 
// 	CudaImageContainer* getCurrentBuffer() const 
// 	{
// 		if (currentBuffer<0 || currentBuffer>NUM_BUFFERS)
// 			return NULL;
// 
// 		return imageBuffers[currentBuffer];
// 	}
// 
// 	CudaImageContainer* getNextBuffer()
// 	{
// 		return imageBuffers[getNextBufferNum()];
// 	}
// 
// 	int getNextBufferNum()
// 	{
// 		int nextIndex = currentBuffer;
// 		do 
// 		{
// 			++nextIndex;
// 			if (nextIndex>=NUM_BUFFERS)
// 				nextIndex = 0;
// 		} while (nextIndex==reservedBuffer);
// 		return nextIndex;
// 	}
// 
// 	CudaImageContainer* getReservedBuffer()
// 	{
// 		if (reservedBuffer<0)
// 			return NULL;
// 
// 		return imageBuffers[reservedBuffer];
// 	}
// 
// 	void reserveCurrentBuffer()
// 	{
// 		reservedBuffer = currentBuffer;
// 	}
// 
// 	void releaseReservedBuffer()
// 	{
// 		reservedBuffer = -1;
// 	}
// 
// 	void incrementBufferNumber()
// 	{
// 		hipDeviceSynchronize();
// #ifdef _DEBUG
// 		gpuErrchk( hipPeekAtLastError() );
// #endif // _DEBUG
// 
// 		currentBuffer = getNextBufferNum();
// 	}