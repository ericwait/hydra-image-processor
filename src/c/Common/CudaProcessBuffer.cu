#include "hip/hip_runtime.h"
#include "CudaUtilities.cuh"
#include "Vec.h"
#include "CudaProcessBuffer.cuh"
#include "CudaDeviceImages.cuh"
#include "CHelpers.h"
#include "CudaImageReduction.cuh"
#include "CudaMeanFilter.cuh"

CudaProcessBuffer::CudaProcessBuffer(int device/*=0*/)
{
	defaults();
	this->device = device;
	deviceSetup();
}

CudaProcessBuffer::~CudaProcessBuffer()
{
	defaults();
}

void CudaProcessBuffer::deviceSetup()
{
	HANDLE_ERROR(hipSetDevice(device));
	HANDLE_ERROR(hipGetDeviceProperties(&deviceProp,device));
}

void CudaProcessBuffer::defaults()
{
	device = 0;
	orgImageDims = Vec<size_t>(0,0,0);
	maxDeviceDims = Vec<size_t>(0,0,0);
}

//////////////////////////////////////////////////////////////////////////
//Cuda Operators (Alphabetical order)
//////////////////////////////////////////////////////////////////////////


double CudaProcessBuffer::normalizedCovariance(const DevicePixelType* imageIn1, const DevicePixelType* imageIn2, Vec<size_t> dims)
{
// 	double im1Mean = sumArray(imageIn1,dims.product()) / dims.product();
// 	double im2Mean = sumArray(imageIn2,dims.product()) / dims.product();
// 
// 	DevicePixelType* im1Sub = addConstant(imageIn1,dims,-1.0*im1Mean);
// 	DevicePixelType* im2Sub = addConstant(imageIn2,dims,-1.0*im2Mean);
// 
// 	DevicePixelType* im1P = imagePow(im1Sub,dims,2.0);
// 	DevicePixelType* im2P = imagePow(im2Sub,dims,2.0);
// 
// 	double sigma1 = sqrt(sumArray(im1P,dims.product())/dims.product());
// 	double sigma2 = sqrt(sumArray(im2P,dims.product())/dims.product());
// 
// 	DevicePixelType* imMul = multiplyImageWith(im1Sub,im2Sub,dims,1.0);
// 	double numarator = sumArray(imMul,dims.product());
// 
// 	double coVar = numarator/(dims.product()*sigma1*sigma2);
// 
// 	delete[] im1Sub;
// 	delete[] im2Sub;
// 	delete[] im1P;
// 	delete[] im2P;
// 	delete[] imMul;
// 
// 	return coVar;

	return 0.0;
}

