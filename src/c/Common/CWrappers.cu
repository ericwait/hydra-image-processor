#include "hip/hip_runtime.h"
#include "CWrappers.cuh"
#include "CudaAdd.cuh"
#include "CudaHistogram.cuh"
#include "CudaGaussianFilter.cuh"
#include "CudaGetMinMax.cuh"
#include "CudaMaxFilter.cuh"
#include "CudaMedianFilter.cuh"
#include "CudaMinFilter.cuh"
#include "CudaMultiplyImage.cuh"
#include "CudaPow.cuh"
#include "CudaSum.cuh"

unsigned char* cAddConstant(const unsigned char* imageIn, Vec<size_t> dims, double additive, unsigned char** imageOut/*=NULL*/,
							int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

unsigned int* cAddConstant(const unsigned int* imageIn, Vec<size_t> dims, double additive, unsigned int** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

int* cAddConstant(const int* imageIn, Vec<size_t> dims, double additive, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

float* cAddConstant(const float* imageIn, Vec<size_t> dims, double additive, float** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

double* cAddConstant(const double* imageIn, Vec<size_t> dims, double additive, double** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

unsigned char* cAddImageWith(const unsigned char* imageIn1, const unsigned char* imageIn2, Vec<size_t> dims, double additive,
							 unsigned char** imageOut/*=NULL*/, int device/*=0*/)
{
	return addImageWith(imageIn1,imageIn2,dims,additive,imageOut,device);
}

unsigned int* cAddImageWith(const unsigned int* imageIn1, const unsigned int* imageIn2, Vec<size_t> dims, double additive,
							unsigned int** imageOut/*=NULL*/, int device/*=0*/)
{
	return addImageWith(imageIn1,imageIn2,dims,additive,imageOut,device);
}

int* cAddImageWith(const int* imageIn1, const int* imageIn2, Vec<size_t> dims, double additive, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return addImageWith(imageIn1,imageIn2,dims,additive,imageOut,device);
}

float* cAddImageWith(const float* imageIn1, const float* imageIn2, Vec<size_t> dims, double additive, float** imageOut/*=NULL*/,
					 int device/*=0*/)
{
	return addImageWith(imageIn1,imageIn2,dims,additive,imageOut,device);
}

double* cAddImageWith(const double* imageIn1, const double* imageIn2, Vec<size_t> dims, double additive, double** imageOut/*=NULL*/,
					  int device/*=0*/)
{
	return addImageWith(imageIn1,imageIn2,dims,additive,imageOut,device);
}

size_t* cHistogram(const unsigned char* imageIn, Vec<size_t> dims, unsigned int arraySize,
				   unsigned char minVal/*=std::numeric_limits<unsigned char>::lowest()*/,
				   unsigned char maxVal/*=std::numeric_limits<unsigned char>::max()*/, int device/*=0*/)
{
	return calculateHistogram(imageIn,dims,arraySize,minVal,maxVal,device);
}

size_t* cHistogram(const unsigned int* imageIn, Vec<size_t> dims, unsigned int arraySize,
				   unsigned int minVal/*=std::numeric_limits<unsigned int>::lowest()*/,
				   unsigned int maxVal/*=std::numeric_limits<unsigned int>::max()*/, int device/*=0*/)
{
	return calculateHistogram(imageIn,dims,arraySize,minVal,maxVal,device);
}

size_t* cHistogram(const int* imageIn, Vec<size_t> dims, unsigned int arraySize, int minVal/*=std::numeric_limits<int>::lowest()*/,
				   int maxVal/*=std::numeric_limits<int>::max()*/, int device/*=0*/)
{
	return calculateHistogram(imageIn,dims,arraySize,minVal,maxVal,device);
}

size_t* cHistogram(const float* imageIn, Vec<size_t> dims, unsigned int arraySize, float minVal/*=std::numeric_limits<float>::lowest()*/,
				   float maxVal/*=std::numeric_limits<float>::max()*/, int device/*=0*/)
{
	return calculateHistogram(imageIn,dims,arraySize,minVal,maxVal,device);
}

size_t* cHistogram(const double* imageIn, Vec<size_t> dims, unsigned int arraySize, double minVal/*=std::numeric_limits<double>::lowest()*/,
				   double maxVal/*=std::numeric_limits<double>::max()*/, int device/*=0*/)
{
	return calculateHistogram(imageIn,dims,arraySize,minVal,maxVal,device);
}

unsigned char* cGaussianFilter(const unsigned char* imageIn, Vec<size_t> dims, Vec<float> sigmas, unsigned char** imageOut/*=NULL*/,
							   int device/*=0*/)
{
	return gaussianFilter(imageIn,dims,sigmas,imageOut,device);
}

unsigned int* cGaussianFilter(const unsigned int* imageIn, Vec<size_t> dims, Vec<float> sigmas, unsigned int** imageOut/*=NULL*/,
							  int device/*=0*/)
{
	return gaussianFilter(imageIn,dims,sigmas,imageOut,device);
}

int* cGaussianFilter(const int* imageIn, Vec<size_t> dims, Vec<float> sigmas, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return gaussianFilter(imageIn,dims,sigmas,imageOut,device);
}

float* cGaussianFilter(const float* imageIn, Vec<size_t> dims, Vec<float> sigmas, float** imageOut/*=NULL*/, int device/*=0*/)
{
	return gaussianFilter(imageIn,dims,sigmas,imageOut,device);
}

double* cGaussianFilter(const double* imageIn, Vec<size_t> dims, Vec<float> sigmas, double** imageOut/*=NULL*/, int device/*=0*/)
{
	return gaussianFilter(imageIn,dims,sigmas,imageOut,device);
}

void cGetMinMax(const unsigned char* imageIn, Vec<size_t> dims, unsigned char& minVal, unsigned char& maxVal, int device/*=0*/)
{
	getMinMax(imageIn,dims,minVal,maxVal,device);
}

void cGetMinMax(const unsigned int* imageIn, Vec<size_t> dims, unsigned int& minVal, unsigned int& maxVal, int device/*=0*/)
{
	getMinMax(imageIn,dims,minVal,maxVal,device);
}

void cGetMinMax(const int* imageIn, Vec<size_t> dims, int& minVal, int& maxVal, int device/*=0*/)
{
	getMinMax(imageIn,dims,minVal,maxVal,device);
}

void cGetMinMax(const float* imageIn, Vec<size_t> dims, float& minVal, float& maxVal, int device/*=0*/)
{
	getMinMax(imageIn,dims,minVal,maxVal,device);
}

void cGetMinMax(const double* imageIn, Vec<size_t> dims, double& minVal, double& maxVal, int device/*=0*/)
{
	getMinMax(imageIn,dims,minVal,maxVal,device);
}

unsigned char* cImagePow(const unsigned char* imageIn, Vec<size_t> dims, double additive, unsigned char** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

unsigned int* cImagePow(const unsigned int* imageIn, Vec<size_t> dims, double power, unsigned int** imageOut/*=NULL*/, int device/*=0*/)
{
	return imagePow(imageIn,dims,power,imageOut,device);
}

int* cImagePow(const int* imageIn, Vec<size_t> dims, double power, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return imagePow(imageIn,dims,power,imageOut,device);
}

float* cImagePow(const float* imageIn, Vec<size_t> dims, double power, float** imageOut/*=NULL*/, int device/*=0*/)
{
	return imagePow(imageIn,dims,power,imageOut,device);
}

double* cImagePow(const double* imageIn, Vec<size_t> dims, double power, double** imageOut/*=NULL*/, int device/*=0*/)
{
	return imagePow(imageIn,dims,power,imageOut,device);
}

unsigned char* cMedianFilter(const unsigned char* imageIn, Vec<size_t> dims, Vec<size_t> neighborhood, unsigned char** imageOut/*=NULL*/,
							 int device/*=0*/)
{
	return medianFilter(imageIn,dims,neighborhood,imageOut,device);
}

unsigned int* cMedianFilter(const unsigned int* imageIn, Vec<size_t> dims, Vec<size_t> neighborhood, unsigned int** imageOut/*=NULL*/,
							int device/*=0*/)
{
	return medianFilter(imageIn,dims,neighborhood,imageOut,device);
}

unsigned char* cMaxFilter(const unsigned char* imageIn, Vec<size_t> dims, Vec<size_t> kernelDims, float* kernel/*=NULL*/,
						  unsigned char** imageOut/*=NULL*/, int device/*=0*/)
{
	return maxFilter(imageIn,dims,kernelDims,kernel,imageOut,device);
}

unsigned int* cMaxFilter(const unsigned int* imageIn, Vec<size_t> dims, Vec<size_t> kernelDims, float* kernel/*=NULL*/,
						 unsigned int** imageOut/*=NULL*/, int device/*=0*/)
{
	return maxFilter(imageIn,dims,kernelDims,kernel,imageOut,device);
}

int* cMaxFilter(const int* imageIn, Vec<size_t> dims, Vec<size_t> kernelDims, float* kernel/*=NULL*/, int** imageOut/*=NULL*/,
				int device/*=0*/)
{
	return maxFilter(imageIn,dims,kernelDims,kernel,imageOut,device);
}

float* cMaxFilter(const float* imageIn, Vec<size_t> dims, Vec<size_t> kernelDims, float* kernel/*=NULL*/, float** imageOut/*=NULL*/,
				  int device/*=0*/)
{
	return maxFilter(imageIn,dims,kernelDims,kernel,imageOut,device);
}

double* cMaxFilter(const double* imageIn, Vec<size_t> dims, Vec<size_t> kernelDims, float* kernel/*=NULL*/, double** imageOut/*=NULL*/,
				   int device/*=0*/)
{
	return maxFilter(imageIn,dims,kernelDims,kernel,imageOut,device);
}

int* cMedianFilter(const int* imageIn, Vec<size_t> dims, Vec<size_t> neighborhood, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return medianFilter(imageIn,dims,neighborhood,imageOut,device);
}

float* cMedianFilter(const float* imageIn, Vec<size_t> dims, Vec<size_t> neighborhood, float** imageOut/*=NULL*/, int device/*=0*/)
{
	return medianFilter(imageIn,dims,neighborhood,imageOut,device);
}

double* cMedianFilter(const double* imageIn, Vec<size_t> dims, Vec<size_t> neighborhood, double** imageOut/*=NULL*/, int device/*=0*/)
{
	return medianFilter(imageIn,dims,neighborhood,imageOut,device);
}

unsigned char* cMinFilter(const unsigned char* imageIn, Vec<size_t> dims, Vec<size_t> kernelDims, float* kernel/*=NULL*/,
						  unsigned char** imageOut/*=NULL*/, int device/*=0*/)
{
	return minFilter(imageIn,dims,kernelDims,kernel,imageOut,device);
}

unsigned int* cMinFilter(const unsigned int* imageIn, Vec<size_t> dims, Vec<size_t> kernelDims, float* kernel/*=NULL*/,
						 unsigned int** imageOut/*=NULL*/, int device/*=0*/)
{
	return minFilter(imageIn,dims,kernelDims,kernel,imageOut,device);
}

int* cMinFilter(const int* imageIn, Vec<size_t> dims, Vec<size_t> kernelDims, float* kernel/*=NULL*/, int** imageOut/*=NULL*/,
				int device/*=0*/)
{
	return minFilter(imageIn,dims,kernelDims,kernel,imageOut,device);
}

float* cMinFilter(const float* imageIn, Vec<size_t> dims, Vec<size_t> kernelDims, float* kernel/*=NULL*/, float** imageOut/*=NULL*/, int device/*=0*/)
{
	return minFilter(imageIn,dims,kernelDims,kernel,imageOut,device);
}

double* cMinFilter(const double* imageIn, Vec<size_t> dims, Vec<size_t> kernelDims, float* kernel/*=NULL*/, double** imageOut/*=NULL*/, int device/*=0*/)
{
	return minFilter(imageIn,dims,kernelDims,kernel,imageOut,device);
}

unsigned char* cMultiplyImage(const unsigned char* imageIn, Vec<size_t> dims, double multiplier, unsigned char** imageOut/*=NULL*/, int device/*=0*/)
{
	return multiplyImage(imageIn,dims,multiplier,imageOut,device);
}

unsigned int* cMultiplyImage(const unsigned int* imageIn, Vec<size_t> dims, double multiplier, unsigned int** imageOut/*=NULL*/, int device/*=0*/)
{
	return multiplyImage(imageIn,dims,multiplier,imageOut,device);
}

int* cMultiplyImage(const int* imageIn, Vec<size_t> dims, double multiplier, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return multiplyImage(imageIn,dims,multiplier,imageOut,device);
}

float* cMultiplyImage(const float* imageIn, Vec<size_t> dims, double multiplier, float** imageOut/*=NULL*/, int device/*=0*/)
{
	return multiplyImage(imageIn,dims,multiplier,imageOut,device);
}

double* cMultiplyImage(const double* imageIn, Vec<size_t> dims, double multiplier, double** imageOut/*=NULL*/, int device/*=0*/)
{
	return multiplyImage(imageIn,dims,multiplier,imageOut,device);
}

unsigned char* cMultiplyImageWith(const unsigned char* imageIn1, const unsigned char* imageIn2, Vec<size_t> dims, double factor,
								  unsigned char** imageOut/*=NULL*/, int device/*=0*/)
{
	return multiplyImageWith(imageIn1,imageIn2,dims,factor,imageOut,device);
}

unsigned int* cMultiplyImageWith(const unsigned int* imageIn1, const unsigned int* imageIn2, Vec<size_t> dims, double factor,
								 unsigned int** imageOut/*=NULL*/, int device/*=0*/)
{
	return multiplyImageWith(imageIn1,imageIn2,dims,factor,imageOut,device);
}

int* cMultiplyImageWith(const int* imageIn1, const int* imageIn2, Vec<size_t> dims, double factor, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return multiplyImageWith(imageIn1,imageIn2,dims,factor,imageOut,device);
}

float* cMultiplyImageWith(const float* imageIn1, const float* imageIn2, Vec<size_t> dims, double factor, float** imageOut/*=NULL*/,
						  int device/*=0*/)
{
	return multiplyImageWith(imageIn1,imageIn2,dims,factor,imageOut,device);
}

double* cMultiplyImageWith(const double* imageIn1, const double* imageIn2, Vec<size_t> dims, double factor, double** imageOut/*=NULL*/,
						   int device/*=0*/)
{
	return multiplyImageWith(imageIn1,imageIn2,dims,factor,imageOut,device);
}

double* cNormalizeHistogram(const unsigned char* imageIn, Vec<size_t> dims, unsigned int arraySize,
							unsigned char minVal/*=std::numeric_limits<unsigned char>::lowest()*/,
							unsigned char maxVal/*=std::numeric_limits<unsigned char>::max()*/, int device/*=0*/)
{
	return normalizeHistogram(imageIn,dims,arraySize,minVal,maxVal,device);
}

double* cNormalizeHistogram(const unsigned int* imageIn, Vec<size_t> dims, unsigned int arraySize,
							unsigned int minVal/*=std::numeric_limits<unsigned int>::lowest()*/,
							unsigned int maxVal/*=std::numeric_limits<unsigned int>::max()*/, int device/*=0*/)
{
	return normalizeHistogram(imageIn,dims,arraySize,minVal,maxVal,device);
}

double* cNormalizeHistogram(const int* imageIn, Vec<size_t> dims, unsigned int arraySize, int minVal/*=std::numeric_limits<int>::lowest()*/,
							int maxVal/*=std::numeric_limits<int>::max()*/, int device/*=0*/)
{
	return normalizeHistogram(imageIn,dims,arraySize,minVal,maxVal,device);
}

double* cNormalizeHistogram(const float* imageIn, Vec<size_t> dims, unsigned int arraySize,
							float minVal/*=std::numeric_limits<float>::lowest()*/, float maxVal/*=std::numeric_limits<float>::max()*/,
							int device/*=0*/)
{
	return normalizeHistogram(imageIn,dims,arraySize,minVal,maxVal,device);
}

double* cNormalizeHistogram(const double* imageIn, Vec<size_t> dims, unsigned int arraySize,
							double minVal/*=std::numeric_limits<double>::lowest()*/, double maxVal/*=std::numeric_limits<double>::max()*/,
							int device/*=0*/)
{
	return normalizeHistogram(imageIn,dims,arraySize,minVal,maxVal,device);
}

double cSumArray(const unsigned char* imageIn, size_t n, int device/*=0*/)
{
	return sumArray(imageIn,n,device);
}

double cSumArray(const unsigned int* imageIn, size_t n, int device/*=0*/)
{
	return sumArray(imageIn,n,device);
}

double cSumArray(const int* imageIn, size_t n, int device/*=0*/)
{
	return sumArray(imageIn,n,device);
}

double cSumArray(const float* imageIn, size_t n, int device/*=0*/)
{
	return sumArray(imageIn,n,device);
}

double cSumArray(const double* imageIn, size_t n, int device/*=0*/)
{
	return sumArray(imageIn,n,device);
}
