#include "CWrappers.cuh"
#include "CudaAdd.cuh"
#include "CudaPow.cuh"
#include "CudaSum.cuh"

unsigned char* cAddConstant(const unsigned char* imageIn, Vec<size_t> dims, double additive, unsigned char** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

unsigned int* cAddConstant(const unsigned int* imageIn, Vec<size_t> dims, double additive, unsigned int** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

int* cAddConstant(const int* imageIn, Vec<size_t> dims, double additive, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

float* cAddConstant(const float* imageIn, Vec<size_t> dims, double additive, float** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

double* cAddConstant(const double* imageIn, Vec<size_t> dims, double additive, double** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

unsigned char* cImagePow(const unsigned char* imageIn, Vec<size_t> dims, double additive, unsigned char** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

unsigned int* cImagePow(const unsigned int* imageIn, Vec<size_t> dims, double power, unsigned int** imageOut/*=NULL*/, int device/*=0*/)
{
	return imagePow(imageIn,dims,power,imageOut,device);
}

int* cImagePow(const int* imageIn, Vec<size_t> dims, double power, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return imagePow(imageIn,dims,power,imageOut,device);
}

float* cImagePow(const float* imageIn, Vec<size_t> dims, double power, float** imageOut/*=NULL*/, int device/*=0*/)
{
	return imagePow(imageIn,dims,power,imageOut,device);
}

double* cImagePow(const double* imageIn, Vec<size_t> dims, double power, double** imageOut/*=NULL*/, int device/*=0*/)
{
	return imagePow(imageIn,dims,power,imageOut,device);
}

double cSumArray(const unsigned char* imageIn, size_t n, int device/*=0*/)
{
	return sumArray(imageIn,n,device);
}

double cSumArray(const unsigned int* imageIn, size_t n, int device/*=0*/)
{
	return sumArray(imageIn,n,device);
}

double cSumArray(const int* imageIn, size_t n, int device/*=0*/)
{
	return sumArray(imageIn,n,device);
}

double cSumArray(const float* imageIn, size_t n, int device/*=0*/)
{
	return sumArray(imageIn,n,device);
}

double cSumArray(const double* imageIn, size_t n, int device/*=0*/)
{
	return sumArray(imageIn,n,device);
}
