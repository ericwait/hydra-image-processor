#include "CWrappers.cuh"
#include "CudaAdd.cuh"
#include "CudaGaussianFilter.cuh"
#include "CudaMedianFilter.cuh"
#include "CudaMultiplyImage.cuh"
#include "CudaPow.cuh"
#include "CudaSum.cuh"

unsigned char* cAddConstant(const unsigned char* imageIn, Vec<size_t> dims, double additive, unsigned char** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

unsigned int* cAddConstant(const unsigned int* imageIn, Vec<size_t> dims, double additive, unsigned int** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

int* cAddConstant(const int* imageIn, Vec<size_t> dims, double additive, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

float* cAddConstant(const float* imageIn, Vec<size_t> dims, double additive, float** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

double* cAddConstant(const double* imageIn, Vec<size_t> dims, double additive, double** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

unsigned char* cAddImageWith(const unsigned char* imageIn1, const unsigned char* imageIn2, Vec<size_t> dims, double additive,
							 unsigned char** imageOut/*=NULL*/, int device/*=0*/)
{
	return addImageWith(imageIn1,imageIn2,dims,additive,imageOut,device);
}

unsigned int* cAddImageWith(const unsigned int* imageIn1, const unsigned int* imageIn2, Vec<size_t> dims, double additive,
							unsigned int** imageOut/*=NULL*/, int device/*=0*/)
{
	return addImageWith(imageIn1,imageIn2,dims,additive,imageOut,device);
}

int* cAddImageWith(const int* imageIn1, const int* imageIn2, Vec<size_t> dims, double additive, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return addImageWith(imageIn1,imageIn2,dims,additive,imageOut,device);
}

float* cAddImageWith(const float* imageIn1, const float* imageIn2, Vec<size_t> dims, double additive, float** imageOut/*=NULL*/,
					 int device/*=0*/)
{
	return addImageWith(imageIn1,imageIn2,dims,additive,imageOut,device);
}

double* cAddImageWith(const double* imageIn1, const double* imageIn2, Vec<size_t> dims, double additive, double** imageOut/*=NULL*/,
					  int device/*=0*/)
{
	return addImageWith(imageIn1,imageIn2,dims,additive,imageOut,device);
}

unsigned char* cGaussianFilter(const unsigned char* imageIn, Vec<size_t> dims, Vec<float> sigmas, unsigned char** imageOut/*=NULL*/, int device/*=0*/)
{
	return gaussianFilter(imageIn,dims,sigmas,imageOut,device);
}

unsigned int* cGaussianFilter(const unsigned int* imageIn, Vec<size_t> dims, Vec<float> sigmas, unsigned int** imageOut/*=NULL*/, int device/*=0*/)
{
	return gaussianFilter(imageIn,dims,sigmas,imageOut,device);
}

int* cGaussianFilter(const int* imageIn, Vec<size_t> dims, Vec<float> sigmas, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return gaussianFilter(imageIn,dims,sigmas,imageOut,device);
}

float* cGaussianFilter(const float* imageIn, Vec<size_t> dims, Vec<float> sigmas, float** imageOut/*=NULL*/, int device/*=0*/)
{
	return gaussianFilter(imageIn,dims,sigmas,imageOut,device);
}

double* cGaussianFilter(const double* imageIn, Vec<size_t> dims, Vec<float> sigmas, double** imageOut/*=NULL*/, int device/*=0*/)
{
	return gaussianFilter(imageIn,dims,sigmas,imageOut,device);
}

unsigned char* cImagePow(const unsigned char* imageIn, Vec<size_t> dims, double additive, unsigned char** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

unsigned int* cImagePow(const unsigned int* imageIn, Vec<size_t> dims, double power, unsigned int** imageOut/*=NULL*/, int device/*=0*/)
{
	return imagePow(imageIn,dims,power,imageOut,device);
}

int* cImagePow(const int* imageIn, Vec<size_t> dims, double power, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return imagePow(imageIn,dims,power,imageOut,device);
}

float* cImagePow(const float* imageIn, Vec<size_t> dims, double power, float** imageOut/*=NULL*/, int device/*=0*/)
{
	return imagePow(imageIn,dims,power,imageOut,device);
}

double* cImagePow(const double* imageIn, Vec<size_t> dims, double power, double** imageOut/*=NULL*/, int device/*=0*/)
{
	return imagePow(imageIn,dims,power,imageOut,device);
}

unsigned char* cMedianFilter(const unsigned char* imageIn, Vec<size_t> dims, Vec<size_t> neighborhood, unsigned char** imageOut/*=NULL*/,
							 int device/*=0*/)
{
	return medianFilter(imageIn,dims,neighborhood,imageOut,device);
}

unsigned int* cMedianFilter(const unsigned int* imageIn, Vec<size_t> dims, Vec<size_t> neighborhood, unsigned int** imageOut/*=NULL*/,
							int device/*=0*/)
{
	return medianFilter(imageIn,dims,neighborhood,imageOut,device);
}

int* cMedianFilter(const int* imageIn, Vec<size_t> dims, Vec<size_t> neighborhood, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return medianFilter(imageIn,dims,neighborhood,imageOut,device);
}

float* cMedianFilter(const float* imageIn, Vec<size_t> dims, Vec<size_t> neighborhood, float** imageOut/*=NULL*/, int device/*=0*/)
{
	return medianFilter(imageIn,dims,neighborhood,imageOut,device);
}

double* cMedianFilter(const double* imageIn, Vec<size_t> dims, Vec<size_t> neighborhood, double** imageOut/*=NULL*/, int device/*=0*/)
{
	return medianFilter(imageIn,dims,neighborhood,imageOut,device);
}

unsigned char* cMultiplyImage(const unsigned char* imageIn, Vec<size_t> dims, double multiplier, unsigned char** imageOut/*=NULL*/, int device/*=0*/)
{
	return multiplyImage(imageIn,dims,multiplier,imageOut,device);
}

unsigned int* cMultiplyImage(const unsigned int* imageIn, Vec<size_t> dims, double multiplier, unsigned int** imageOut/*=NULL*/, int device/*=0*/)
{
	return multiplyImage(imageIn,dims,multiplier,imageOut,device);
}

int* cMultiplyImage(const int* imageIn, Vec<size_t> dims, double multiplier, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return multiplyImage(imageIn,dims,multiplier,imageOut,device);
}

float* cMultiplyImage(const float* imageIn, Vec<size_t> dims, double multiplier, float** imageOut/*=NULL*/, int device/*=0*/)
{
	return multiplyImage(imageIn,dims,multiplier,imageOut,device);
}

double* cMultiplyImage(const double* imageIn, Vec<size_t> dims, double multiplier, double** imageOut/*=NULL*/, int device/*=0*/)
{
	return multiplyImage(imageIn,dims,multiplier,imageOut,device);
}

double cSumArray(const unsigned char* imageIn, size_t n, int device/*=0*/)
{
	return sumArray(imageIn,n,device);
}

double cSumArray(const unsigned int* imageIn, size_t n, int device/*=0*/)
{
	return sumArray(imageIn,n,device);
}

double cSumArray(const int* imageIn, size_t n, int device/*=0*/)
{
	return sumArray(imageIn,n,device);
}

double cSumArray(const float* imageIn, size_t n, int device/*=0*/)
{
	return sumArray(imageIn,n,device);
}

double cSumArray(const double* imageIn, size_t n, int device/*=0*/)
{
	return sumArray(imageIn,n,device);
}
