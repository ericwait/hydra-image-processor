#include "hip/hip_runtime.h"
#include "CWrappers.cuh"
#include "CudaAdd.cuh"
#include "CudaHistogram.cuh"
#include "CudaGaussianFilter.cuh"
#include "CudaGetMinMax.cuh"
#include "CudaMaxFilter.cuh"
#include "CudaMeanFilter.cuh"
#include "CudaMedianFilter.cuh"
#include "CudaMinFilter.cuh"
#include "CudaMultiplyImage.cuh"
#include "CudaPolyTransferFunc.cuh"
#include "CudaPow.cuh"
#include "CudaSum.cuh"
#include "CudaThreshold.cuh"

unsigned char* cAddConstant(const unsigned char* imageIn, Vec<size_t> dims, double additive, unsigned char** imageOut/*=NULL*/,
							int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

unsigned int* cAddConstant(const unsigned int* imageIn, Vec<size_t> dims, double additive, unsigned int** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

int* cAddConstant(const int* imageIn, Vec<size_t> dims, double additive, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

float* cAddConstant(const float* imageIn, Vec<size_t> dims, double additive, float** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

double* cAddConstant(const double* imageIn, Vec<size_t> dims, double additive, double** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}


unsigned char* cApplyPolyTransferFunction(const unsigned char* imageIn, Vec<size_t> dims, double a, double b, double c,
										  unsigned char minValue/*=std::numeric_limits<PixelType>::lowest()*/,
										  unsigned char maxValue/*=std::numeric_limits<PixelType>::max()*/,
										  unsigned char** imageOut/*=NULL*/, int device/*=0*/)
{
	return applyPolyTransferFunction(imageIn,dims,a,b,c,minValue,maxValue,imageOut,device);
}

unsigned int* cApplyPolyTransferFunction(const unsigned int* imageIn, Vec<size_t> dims, double a, double b, double c,
										 unsigned int minValue/*=std::numeric_limits<PixelType>::lowest()*/,
										 unsigned int maxValue/*=std::numeric_limits<PixelType>::max()*/,
										 unsigned int** imageOut/*=NULL*/, int device/*=0*/)
{
	return applyPolyTransferFunction(imageIn,dims,a,b,c,minValue,maxValue,imageOut,device);
}

int* cApplyPolyTransferFunction(const int* imageIn, Vec<size_t> dims, double a, double b, double c,
								int minValue/*=std::numeric_limits<PixelType>::lowest()*/,
								int maxValue/*=std::numeric_limits<PixelType>::max()*/, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return applyPolyTransferFunction(imageIn,dims,a,b,c,minValue,maxValue,imageOut,device);
}

float* cApplyPolyTransferFunction(const float* imageIn, Vec<size_t> dims, double a, double b, double c,
								  float minValue/*=std::numeric_limits<PixelType>::lowest()*/,
								  float maxValue/*=std::numeric_limits<PixelType>::max()*/, float** imageOut/*=NULL*/, int device/*=0*/)
{
	return applyPolyTransferFunction(imageIn,dims,a,b,c,minValue,maxValue,imageOut,device);
}

double* cApplyPolyTransferFunction(const double* imageIn, Vec<size_t> dims, double a, double b, double c,
								   double minValue/*=std::numeric_limits<PixelType>::lowest()*/,
								   double maxValue/*=std::numeric_limits<PixelType>::max()*/,double** imageOut/*=NULL*/, int device/*=0*/)
{
	return applyPolyTransferFunction(imageIn,dims,a,b,c,minValue,maxValue,imageOut,device);
}


unsigned char* cAddImageWith(const unsigned char* imageIn1, const unsigned char* imageIn2, Vec<size_t> dims, double additive,
							 unsigned char** imageOut/*=NULL*/, int device/*=0*/)
{
	return addImageWith(imageIn1,imageIn2,dims,additive,imageOut,device);
}

unsigned int* cAddImageWith(const unsigned int* imageIn1, const unsigned int* imageIn2, Vec<size_t> dims, double additive,
							unsigned int** imageOut/*=NULL*/, int device/*=0*/)
{
	return addImageWith(imageIn1,imageIn2,dims,additive,imageOut,device);
}

int* cAddImageWith(const int* imageIn1, const int* imageIn2, Vec<size_t> dims, double additive, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return addImageWith(imageIn1,imageIn2,dims,additive,imageOut,device);
}

float* cAddImageWith(const float* imageIn1, const float* imageIn2, Vec<size_t> dims, double additive, float** imageOut/*=NULL*/,
					 int device/*=0*/)
{
	return addImageWith(imageIn1,imageIn2,dims,additive,imageOut,device);
}

double* cAddImageWith(const double* imageIn1, const double* imageIn2, Vec<size_t> dims, double additive, double** imageOut/*=NULL*/,
					  int device/*=0*/)
{
	return addImageWith(imageIn1,imageIn2,dims,additive,imageOut,device);
}


size_t* cHistogram(const unsigned char* imageIn, Vec<size_t> dims, unsigned int arraySize,
				   unsigned char minVal/*=std::numeric_limits<unsigned char>::lowest()*/,
				   unsigned char maxVal/*=std::numeric_limits<unsigned char>::max()*/, int device/*=0*/)
{
	return calculateHistogram(imageIn,dims,arraySize,minVal,maxVal,device);
}

size_t* cHistogram(const unsigned int* imageIn, Vec<size_t> dims, unsigned int arraySize,
				   unsigned int minVal/*=std::numeric_limits<unsigned int>::lowest()*/,
				   unsigned int maxVal/*=std::numeric_limits<unsigned int>::max()*/, int device/*=0*/)
{
	return calculateHistogram(imageIn,dims,arraySize,minVal,maxVal,device);
}

size_t* cHistogram(const int* imageIn, Vec<size_t> dims, unsigned int arraySize, int minVal/*=std::numeric_limits<int>::lowest()*/,
				   int maxVal/*=std::numeric_limits<int>::max()*/, int device/*=0*/)
{
	return calculateHistogram(imageIn,dims,arraySize,minVal,maxVal,device);
}

size_t* cHistogram(const float* imageIn, Vec<size_t> dims, unsigned int arraySize, float minVal/*=std::numeric_limits<float>::lowest()*/,
				   float maxVal/*=std::numeric_limits<float>::max()*/, int device/*=0*/)
{
	return calculateHistogram(imageIn,dims,arraySize,minVal,maxVal,device);
}

size_t* cHistogram(const double* imageIn, Vec<size_t> dims, unsigned int arraySize, double minVal/*=std::numeric_limits<double>::lowest()*/,
				   double maxVal/*=std::numeric_limits<double>::max()*/, int device/*=0*/)
{
	return calculateHistogram(imageIn,dims,arraySize,minVal,maxVal,device);
}


unsigned char* cGaussianFilter(const unsigned char* imageIn, Vec<size_t> dims, Vec<float> sigmas, unsigned char** imageOut/*=NULL*/,
							   int device/*=0*/)
{
	return gaussianFilter(imageIn,dims,sigmas,imageOut,device);
}

unsigned int* cGaussianFilter(const unsigned int* imageIn, Vec<size_t> dims, Vec<float> sigmas, unsigned int** imageOut/*=NULL*/,
							  int device/*=0*/)
{
	return gaussianFilter(imageIn,dims,sigmas,imageOut,device);
}

int* cGaussianFilter(const int* imageIn, Vec<size_t> dims, Vec<float> sigmas, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return gaussianFilter(imageIn,dims,sigmas,imageOut,device);
}

float* cGaussianFilter(const float* imageIn, Vec<size_t> dims, Vec<float> sigmas, float** imageOut/*=NULL*/, int device/*=0*/)
{
	return gaussianFilter(imageIn,dims,sigmas,imageOut,device);
}

double* cGaussianFilter(const double* imageIn, Vec<size_t> dims, Vec<float> sigmas, double** imageOut/*=NULL*/, int device/*=0*/)
{
	return gaussianFilter(imageIn,dims,sigmas,imageOut,device);
}


void cGetMinMax(const unsigned char* imageIn, Vec<size_t> dims, unsigned char& minVal, unsigned char& maxVal, int device/*=0*/)
{
	getMinMax(imageIn,dims,minVal,maxVal,device);
}

void cGetMinMax(const unsigned int* imageIn, Vec<size_t> dims, unsigned int& minVal, unsigned int& maxVal, int device/*=0*/)
{
	getMinMax(imageIn,dims,minVal,maxVal,device);
}

void cGetMinMax(const int* imageIn, Vec<size_t> dims, int& minVal, int& maxVal, int device/*=0*/)
{
	getMinMax(imageIn,dims,minVal,maxVal,device);
}

void cGetMinMax(const float* imageIn, Vec<size_t> dims, float& minVal, float& maxVal, int device/*=0*/)
{
	getMinMax(imageIn,dims,minVal,maxVal,device);
}

void cGetMinMax(const double* imageIn, Vec<size_t> dims, double& minVal, double& maxVal, int device/*=0*/)
{
	getMinMax(imageIn,dims,minVal,maxVal,device);
}


unsigned char* cImagePow(const unsigned char* imageIn, Vec<size_t> dims, double additive, unsigned char** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

unsigned int* cImagePow(const unsigned int* imageIn, Vec<size_t> dims, double power, unsigned int** imageOut/*=NULL*/, int device/*=0*/)
{
	return imagePow(imageIn,dims,power,imageOut,device);
}

int* cImagePow(const int* imageIn, Vec<size_t> dims, double power, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return imagePow(imageIn,dims,power,imageOut,device);
}

float* cImagePow(const float* imageIn, Vec<size_t> dims, double power, float** imageOut/*=NULL*/, int device/*=0*/)
{
	return imagePow(imageIn,dims,power,imageOut,device);
}

double* cImagePow(const double* imageIn, Vec<size_t> dims, double power, double** imageOut/*=NULL*/, int device/*=0*/)
{
	return imagePow(imageIn,dims,power,imageOut,device);
}


unsigned char* cMedianFilter(const unsigned char* imageIn, Vec<size_t> dims, Vec<size_t> neighborhood, unsigned char** imageOut/*=NULL*/,
							 int device/*=0*/)
{
	return medianFilter(imageIn,dims,neighborhood,imageOut,device);
}

unsigned int* cMedianFilter(const unsigned int* imageIn, Vec<size_t> dims, Vec<size_t> neighborhood, unsigned int** imageOut/*=NULL*/,
							int device/*=0*/)
{
	return medianFilter(imageIn,dims,neighborhood,imageOut,device);
}

unsigned char* cMaxFilter(const unsigned char* imageIn, Vec<size_t> dims, Vec<size_t> kernelDims, float* kernel/*=NULL*/,
						  unsigned char** imageOut/*=NULL*/, int device/*=0*/)
{
	return maxFilter(imageIn,dims,kernelDims,kernel,imageOut,device);
}


unsigned int* cMaxFilter(const unsigned int* imageIn, Vec<size_t> dims, Vec<size_t> kernelDims, float* kernel/*=NULL*/,
						 unsigned int** imageOut/*=NULL*/, int device/*=0*/)
{
	return maxFilter(imageIn,dims,kernelDims,kernel,imageOut,device);
}

int* cMaxFilter(const int* imageIn, Vec<size_t> dims, Vec<size_t> kernelDims, float* kernel/*=NULL*/, int** imageOut/*=NULL*/,
				int device/*=0*/)
{
	return maxFilter(imageIn,dims,kernelDims,kernel,imageOut,device);
}

float* cMaxFilter(const float* imageIn, Vec<size_t> dims, Vec<size_t> kernelDims, float* kernel/*=NULL*/, float** imageOut/*=NULL*/,
				  int device/*=0*/)
{
	return maxFilter(imageIn,dims,kernelDims,kernel,imageOut,device);
}

double* cMaxFilter(const double* imageIn, Vec<size_t> dims, Vec<size_t> kernelDims, float* kernel/*=NULL*/, double** imageOut/*=NULL*/,
				   int device/*=0*/)
{
	return maxFilter(imageIn,dims,kernelDims,kernel,imageOut,device);
}


unsigned char* cMeanFilter(const unsigned char* imageIn, Vec<size_t> dims, Vec<size_t> neighborhood, unsigned char** imageOut/*=NULL*/,
						   int device/*=0*/)
{
	return meanFilter(imageIn,dims,neighborhood,imageOut,device);
}

unsigned int* cMeanFilter(const unsigned int* imageIn, Vec<size_t> dims, Vec<size_t> neighborhood, unsigned int** imageOut/*=NULL*/,
						  int device/*=0*/)
{
	return meanFilter(imageIn,dims,neighborhood,imageOut,device);
}

int* cMeanFilter(const int* imageIn, Vec<size_t> dims, Vec<size_t> neighborhood, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return meanFilter(imageIn,dims,neighborhood,imageOut,device);
}

float* cMeanFilter(const float* imageIn, Vec<size_t> dims, Vec<size_t> neighborhood, float** imageOut/*=NULL*/, int device/*=0*/)
{
	return meanFilter(imageIn,dims,neighborhood,imageOut,device);
}

double* cMeanFilter(const double* imageIn, Vec<size_t> dims, Vec<size_t> neighborhood, double** imageOut/*=NULL*/, int device/*=0*/)
{
	return meanFilter(imageIn,dims,neighborhood,imageOut,device);
}


int* cMedianFilter(const int* imageIn, Vec<size_t> dims, Vec<size_t> neighborhood, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return medianFilter(imageIn,dims,neighborhood,imageOut,device);
}

float* cMedianFilter(const float* imageIn, Vec<size_t> dims, Vec<size_t> neighborhood, float** imageOut/*=NULL*/, int device/*=0*/)
{
	return medianFilter(imageIn,dims,neighborhood,imageOut,device);
}

double* cMedianFilter(const double* imageIn, Vec<size_t> dims, Vec<size_t> neighborhood, double** imageOut/*=NULL*/, int device/*=0*/)
{
	return medianFilter(imageIn,dims,neighborhood,imageOut,device);
}


unsigned char* cMinFilter(const unsigned char* imageIn, Vec<size_t> dims, Vec<size_t> kernelDims, float* kernel/*=NULL*/,
						  unsigned char** imageOut/*=NULL*/, int device/*=0*/)
{
	return minFilter(imageIn,dims,kernelDims,kernel,imageOut,device);
}

unsigned int* cMinFilter(const unsigned int* imageIn, Vec<size_t> dims, Vec<size_t> kernelDims, float* kernel/*=NULL*/,
						 unsigned int** imageOut/*=NULL*/, int device/*=0*/)
{
	return minFilter(imageIn,dims,kernelDims,kernel,imageOut,device);
}

int* cMinFilter(const int* imageIn, Vec<size_t> dims, Vec<size_t> kernelDims, float* kernel/*=NULL*/, int** imageOut/*=NULL*/,
				int device/*=0*/)
{
	return minFilter(imageIn,dims,kernelDims,kernel,imageOut,device);
}

float* cMinFilter(const float* imageIn, Vec<size_t> dims, Vec<size_t> kernelDims, float* kernel/*=NULL*/, float** imageOut/*=NULL*/, int device/*=0*/)
{
	return minFilter(imageIn,dims,kernelDims,kernel,imageOut,device);
}

double* cMinFilter(const double* imageIn, Vec<size_t> dims, Vec<size_t> kernelDims, float* kernel/*=NULL*/, double** imageOut/*=NULL*/, int device/*=0*/)
{
	return minFilter(imageIn,dims,kernelDims,kernel,imageOut,device);
}


unsigned char* cMultiplyImage(const unsigned char* imageIn, Vec<size_t> dims, double multiplier, unsigned char** imageOut/*=NULL*/, int device/*=0*/)
{
	return multiplyImage(imageIn,dims,multiplier,imageOut,device);
}

unsigned int* cMultiplyImage(const unsigned int* imageIn, Vec<size_t> dims, double multiplier, unsigned int** imageOut/*=NULL*/, int device/*=0*/)
{
	return multiplyImage(imageIn,dims,multiplier,imageOut,device);
}

int* cMultiplyImage(const int* imageIn, Vec<size_t> dims, double multiplier, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return multiplyImage(imageIn,dims,multiplier,imageOut,device);
}

float* cMultiplyImage(const float* imageIn, Vec<size_t> dims, double multiplier, float** imageOut/*=NULL*/, int device/*=0*/)
{
	return multiplyImage(imageIn,dims,multiplier,imageOut,device);
}

double* cMultiplyImage(const double* imageIn, Vec<size_t> dims, double multiplier, double** imageOut/*=NULL*/, int device/*=0*/)
{
	return multiplyImage(imageIn,dims,multiplier,imageOut,device);
}


unsigned char* cMultiplyImageWith(const unsigned char* imageIn1, const unsigned char* imageIn2, Vec<size_t> dims, double factor,
								  unsigned char** imageOut/*=NULL*/, int device/*=0*/)
{
	return multiplyImageWith(imageIn1,imageIn2,dims,factor,imageOut,device);
}

unsigned int* cMultiplyImageWith(const unsigned int* imageIn1, const unsigned int* imageIn2, Vec<size_t> dims, double factor,
								 unsigned int** imageOut/*=NULL*/, int device/*=0*/)
{
	return multiplyImageWith(imageIn1,imageIn2,dims,factor,imageOut,device);
}

int* cMultiplyImageWith(const int* imageIn1, const int* imageIn2, Vec<size_t> dims, double factor, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return multiplyImageWith(imageIn1,imageIn2,dims,factor,imageOut,device);
}

float* cMultiplyImageWith(const float* imageIn1, const float* imageIn2, Vec<size_t> dims, double factor, float** imageOut/*=NULL*/,
						  int device/*=0*/)
{
	return multiplyImageWith(imageIn1,imageIn2,dims,factor,imageOut,device);
}

double* cMultiplyImageWith(const double* imageIn1, const double* imageIn2, Vec<size_t> dims, double factor, double** imageOut/*=NULL*/,
						   int device/*=0*/)
{
	return multiplyImageWith(imageIn1,imageIn2,dims,factor,imageOut,device);
}


double* cNormalizeHistogram(const unsigned char* imageIn, Vec<size_t> dims, unsigned int arraySize,
							unsigned char minVal/*=std::numeric_limits<unsigned char>::lowest()*/,
							unsigned char maxVal/*=std::numeric_limits<unsigned char>::max()*/, int device/*=0*/)
{
	return normalizeHistogram(imageIn,dims,arraySize,minVal,maxVal,device);
}

double* cNormalizeHistogram(const unsigned int* imageIn, Vec<size_t> dims, unsigned int arraySize,
							unsigned int minVal/*=std::numeric_limits<unsigned int>::lowest()*/,
							unsigned int maxVal/*=std::numeric_limits<unsigned int>::max()*/, int device/*=0*/)
{
	return normalizeHistogram(imageIn,dims,arraySize,minVal,maxVal,device);
}

double* cNormalizeHistogram(const int* imageIn, Vec<size_t> dims, unsigned int arraySize, int minVal/*=std::numeric_limits<int>::lowest()*/,
							int maxVal/*=std::numeric_limits<int>::max()*/, int device/*=0*/)
{
	return normalizeHistogram(imageIn,dims,arraySize,minVal,maxVal,device);
}

double* cNormalizeHistogram(const float* imageIn, Vec<size_t> dims, unsigned int arraySize,
							float minVal/*=std::numeric_limits<float>::lowest()*/, float maxVal/*=std::numeric_limits<float>::max()*/,
							int device/*=0*/)
{
	return normalizeHistogram(imageIn,dims,arraySize,minVal,maxVal,device);
}

double* cNormalizeHistogram(const double* imageIn, Vec<size_t> dims, unsigned int arraySize,
							double minVal/*=std::numeric_limits<double>::lowest()*/, double maxVal/*=std::numeric_limits<double>::max()*/,
							int device/*=0*/)
{
	return normalizeHistogram(imageIn,dims,arraySize,minVal,maxVal,device);
}


unsigned char* cOtsuThresholdFilter(const unsigned char* imageIn, Vec<size_t> dims, double alpha/*=1.0*/, unsigned char** imageOut/*=NULL*/,
									int device/*=0*/)
{
	return otsuThresholdFilter(imageIn,dims,alpha,imageOut,device);
}

unsigned int* cOtsuThresholdFilter(const unsigned int* imageIn, Vec<size_t> dims, double alpha/*=1.0*/, unsigned int** imageOut/*=NULL*/,
								   int device/*=0*/)
{
	return otsuThresholdFilter(imageIn,dims,alpha,imageOut,device);
}

int* cOtsuThresholdFilter(const int* imageIn, Vec<size_t> dims, double alpha/*=1.0*/, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return otsuThresholdFilter(imageIn,dims,alpha,imageOut,device);
}

float* cOtsuThresholdFilter(const float* imageIn, Vec<size_t> dims, double alpha/*=1.0*/, float** imageOut/*=NULL*/, int device/*=0*/)
{
	return otsuThresholdFilter(imageIn,dims,alpha,imageOut,device);
}

double* cOtsuThresholdFilter(const double* imageIn, Vec<size_t> dims, double alpha/*=1.0*/, double** imageOut/*=NULL*/, int device/*=0*/)
{
	return otsuThresholdFilter(imageIn,dims,alpha,imageOut,device);
}


unsigned char cOtsuThresholdValue(const unsigned char* imageIn, Vec<size_t> dims, int device/*=0*/)
{
	return otsuThresholdValue(imageIn,dims,device);
}

unsigned int cOtsuThresholdValue(const unsigned int* imageIn, Vec<size_t> dims, int device/*=0*/)
{
	return otsuThresholdValue(imageIn,dims,device);
}

int cOtsuThresholdValue(const int* imageIn, Vec<size_t> dims, int device/*=0*/)
{
	return otsuThresholdValue(imageIn,dims,device);
}

float cOtsuThresholdValue(const float* imageIn, Vec<size_t> dims, int device/*=0*/)
{
	return otsuThresholdValue(imageIn,dims,device);
}

double cOtsuThresholdValue(const double* imageIn, Vec<size_t> dims, int device/*=0*/)
{
	return otsuThresholdValue(imageIn,dims,device);
}


double cSumArray(const unsigned char* imageIn, size_t n, int device/*=0*/)
{
	return sumArray(imageIn,n,device);
}

double cSumArray(const unsigned int* imageIn, size_t n, int device/*=0*/)
{
	return sumArray(imageIn,n,device);
}

double cSumArray(const int* imageIn, size_t n, int device/*=0*/)
{
	return sumArray(imageIn,n,device);
}

double cSumArray(const float* imageIn, size_t n, int device/*=0*/)
{
	return sumArray(imageIn,n,device);
}

double cSumArray(const double* imageIn, size_t n, int device/*=0*/)
{
	return sumArray(imageIn,n,device);
}


unsigned char* cThresholdFilter(const unsigned char* imageIn, Vec<size_t> dims, unsigned char thresh, unsigned char** imageOut/*=NULL*/,
							   int device/*=0*/)
{
	return thresholdFilter(imageIn,dims,thresh,imageOut,device);
}

unsigned int* cThresholdFilter(const unsigned int* imageIn, Vec<size_t> dims, unsigned int thresh, unsigned int** imageOut/*=NULL*/,
								int device/*=0*/)
{
	return thresholdFilter(imageIn,dims,thresh,imageOut,device);
}

int* cThresholdFilter(const int* imageIn, Vec<size_t> dims, int thresh, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return thresholdFilter(imageIn,dims,thresh,imageOut,device);
}

float* cThresholdFilter(const float* imageIn, Vec<size_t> dims, float thresh, float** imageOut/*=NULL*/, int device/*=0*/)
{
	return thresholdFilter(imageIn,dims,thresh,imageOut,device);
}

double* cThresholdFilter(const double* imageIn, Vec<size_t> dims, double thresh, double** imageOut/*=NULL*/, int device/*=0*/)
{
	return thresholdFilter(imageIn,dims,thresh,imageOut,device);
}
