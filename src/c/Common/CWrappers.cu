#include "CWrappers.cuh"
#include "CudaAdd.cuh"
#include "CudaPow.cuh"
#include "CudaSum.cuh"

unsigned char* cAddConstant(const unsigned char* imageIn, Vec<size_t> dims, double additive, unsigned char** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

unsigned int* cAddConstant(const unsigned int* imageIn, Vec<size_t> dims, double additive, unsigned int** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

int* cAddConstant(const int* imageIn, Vec<size_t> dims, double additive, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

float* cAddConstant(const float* imageIn, Vec<size_t> dims, double additive, float** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

double* cAddConstant(const double* imageIn, Vec<size_t> dims, double additive, double** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

unsigned char* cAddImageWith(const unsigned char* imageIn1, const unsigned char* imageIn2, Vec<size_t> dims, double additive,
							 unsigned char** imageOut/*=NULL*/, int device/*=0*/)
{
	return addImageWith(imageIn1,imageIn2,dims,additive,imageOut,device);
}

unsigned int* cAddImageWith(const unsigned int* imageIn1, const unsigned int* imageIn2, Vec<size_t> dims, double additive,
							unsigned int** imageOut/*=NULL*/, int device/*=0*/)
{
	return addImageWith(imageIn1,imageIn2,dims,additive,imageOut,device);
}

int* cAddImageWith(const int* imageIn1, const int* imageIn2, Vec<size_t> dims, double additive, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return addImageWith(imageIn1,imageIn2,dims,additive,imageOut,device);
}

float* cAddImageWith(const float* imageIn1, const float* imageIn2, Vec<size_t> dims, double additive, float** imageOut/*=NULL*/,
					 int device/*=0*/)
{
	return addImageWith(imageIn1,imageIn2,dims,additive,imageOut,device);
}

double* cAddImageWith(const double* imageIn1, const double* imageIn2, Vec<size_t> dims, double additive, double** imageOut/*=NULL*/,
					  int device/*=0*/)
{
	return addImageWith(imageIn1,imageIn2,dims,additive,imageOut,device);
}

unsigned char* cImagePow(const unsigned char* imageIn, Vec<size_t> dims, double additive, unsigned char** imageOut/*=NULL*/, int device/*=0*/)
{
	return addConstant(imageIn,dims,additive,imageOut,device);
}

unsigned int* cImagePow(const unsigned int* imageIn, Vec<size_t> dims, double power, unsigned int** imageOut/*=NULL*/, int device/*=0*/)
{
	return imagePow(imageIn,dims,power,imageOut,device);
}

int* cImagePow(const int* imageIn, Vec<size_t> dims, double power, int** imageOut/*=NULL*/, int device/*=0*/)
{
	return imagePow(imageIn,dims,power,imageOut,device);
}

float* cImagePow(const float* imageIn, Vec<size_t> dims, double power, float** imageOut/*=NULL*/, int device/*=0*/)
{
	return imagePow(imageIn,dims,power,imageOut,device);
}

double* cImagePow(const double* imageIn, Vec<size_t> dims, double power, double** imageOut/*=NULL*/, int device/*=0*/)
{
	return imagePow(imageIn,dims,power,imageOut,device);
}

double cSumArray(const unsigned char* imageIn, size_t n, int device/*=0*/)
{
	return sumArray(imageIn,n,device);
}

double cSumArray(const unsigned int* imageIn, size_t n, int device/*=0*/)
{
	return sumArray(imageIn,n,device);
}

double cSumArray(const int* imageIn, size_t n, int device/*=0*/)
{
	return sumArray(imageIn,n,device);
}

double cSumArray(const float* imageIn, size_t n, int device/*=0*/)
{
	return sumArray(imageIn,n,device);
}

double cSumArray(const double* imageIn, size_t n, int device/*=0*/)
{
	return sumArray(imageIn,n,device);
}


