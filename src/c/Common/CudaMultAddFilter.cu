#include "hip/hip_runtime.h"
#include "CudaKernels.cuh"

__global__ void cudaMultAddFilter( CudaImageContainer* imageIn, CudaImageContainer* imageOut, Vec<size_t> hostKernelDims, size_t kernelOffset/*=0*/ )
{
	DeviceVec<size_t> coordinate;
	coordinate.x = threadIdx.x + blockIdx.x * blockDim.x;
	coordinate.y = threadIdx.y + blockIdx.y * blockDim.y;
	coordinate.z = threadIdx.z + blockIdx.z * blockDim.z;

	if (coordinate<imageIn->getDeviceDims())
	{
		double val = 0;
		double kernFactor = 0;

		DeviceVec<size_t> kernelDims = hostKernelDims;
		DeviceVec<size_t> kernelMidIdx;
		DeviceVec<size_t> curCoordIm; 
		DeviceVec<size_t> curCoordKrn;

		kernelMidIdx.x = kernelDims.x/2;
		kernelMidIdx.y = kernelDims.y/2;
		kernelMidIdx.z = kernelDims.z/2;

		//find if the kernel will go off the edge of the image
		curCoordIm.z = (size_t) max(0,(int)coordinate.z-(int)kernelMidIdx.z);
		curCoordKrn.z = ((int)coordinate.z-(int)kernelMidIdx.z>=0) ? (0) : (kernelMidIdx.z-coordinate.z);
		for (; curCoordIm.z<imageIn->getDepth() && curCoordKrn.z<kernelDims.z; ++curCoordIm.z, ++curCoordKrn.z)
		{
			curCoordIm.y = (size_t)max(0,(int)coordinate.y-(int)kernelMidIdx.y);
			curCoordKrn.y = ((int)coordinate.y-(int)kernelMidIdx.y>=0) ? (0) : (kernelMidIdx.y-coordinate.y);
			for (; curCoordIm.y<imageIn->getHeight() && curCoordKrn.y<kernelDims.y; ++curCoordIm.y, ++curCoordKrn.y)
			{
				curCoordIm.x = (size_t)max(0,(int)coordinate.x-(int)kernelMidIdx.x);
				curCoordKrn.x = ((int)coordinate.x-(int)kernelMidIdx.x>=0) ? (0) : (kernelMidIdx.x-coordinate.x);		
				for (; curCoordIm.x<imageIn->getWidth() && curCoordKrn.x<kernelDims.x; ++curCoordIm.x, ++curCoordKrn.x)
				{
					size_t kernIdx = kernelDims.linearAddressAt(curCoordKrn)+kernelOffset;
					kernFactor += cudaConstKernel[kernIdx];
					val += (*imageIn)[curCoordIm] * cudaConstKernel[kernIdx];
				}
			}
		}

		(*imageOut)[coordinate] = val/kernFactor;
	}
}

