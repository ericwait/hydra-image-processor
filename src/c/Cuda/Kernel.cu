#include "Kernel.cuh"
#include "CudaUtilities.h"

#ifndef CUDA_CONST_KERNEL
#define CUDA_CONST_KERNEL
__constant__ float cudaConstKernel[CONST_KERNEL_NUM_EL];
#endif

__host__ Kernel::Kernel(Vec<size_t> dimensions, float* values)
{
	load(dimensions, values);
}

__device__ Kernel::Kernel(const Kernel& other)
{
	dims = other.dims;
	kernel = other.kernel;
	cleanUpHost = other.cleanUpHost;
	cudaKernel = other.cudaKernel;
	cleanUpDevice = other.cleanUpDevice;
}


__host__ Kernel::Kernel(ImageContainer<float> kernelIn)
{
	load(kernelIn.getSpatialDims(), kernelIn.getPtr());
}


__host__ void Kernel::load(Vec<size_t> dimensions, float* values)
{
	init();

	dims = dimensions;

	if (values == NULL)
	{
		setOnes();
	}
	else
	{
		kernel = values;
	}

	if (dimensions.product() < CONST_KERNEL_NUM_EL)
	{
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaConstKernel), kernel, sizeof(float)*dims.product()));
		cudaKernel = cudaConstKernel;
	}
	else
	{
		HANDLE_ERROR(hipMalloc(&cudaKernel, sizeof(float)*dims.product()));
		cleanUpDevice = true;
	}
}

__host__ void Kernel::clean()
{
	if (cleanUpHost)
	{
		delete[] kernel;
		cleanUpHost = false;
	}

	if (cleanUpDevice)
	{
		hipFree(cudaKernel);
		cleanUpDevice = false;
	}

	init();
}


__device__ float Kernel::operator[](size_t idx)
{
	return cudaKernel[idx];
}


__device__ float Kernel::operator()(Vec<size_t> coordinate)
{
	return cudaKernel[dims.linearAddressAt(coordinate)];
}


__host__ void Kernel::init()
{
	dims = Vec<size_t>(0);
	kernel = NULL;
	cleanUpHost = false;
	cudaKernel = NULL;
	cleanUpDevice = false;
}


__host__ void Kernel::setOnes()
{
	kernel = new float[dims.product()];
	for (int i = 0; i < dims.product(); ++i)
		kernel[i] = 1.0f;

	cleanUpHost = true;
}


