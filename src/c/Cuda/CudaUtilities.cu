#include "hip/hip_runtime.h"
#include "CudaUtilities.cuh"

void calcBlockThread(const Vec<size_t>& dims, const hipDeviceProp_t &prop, dim3 &blocks, dim3 &threads,
					 size_t maxThreads/*=std::numeric_limits<size_t>::max()*/)
{
	size_t mxThreads = MIN(prop.maxThreadsPerBlock,maxThreads);
	if (dims.z <= 1)
	{
		if (dims.y <= 1)
		{
			if (dims.x < mxThreads)
			{
				threads.x = (unsigned int)dims.x;
				threads.y = 1;
				threads.z = 1;
			} 
			else
			{
				threads.x = (unsigned int)mxThreads;
				threads.y = 1;
				threads.z = 1;
			}
		}
		else
		{
			if (dims.x*dims.y < mxThreads)
			{
				threads.x = (unsigned int)dims.x;
				threads.y = (unsigned int)dims.y;
				threads.z = 1;
			} 
			else
			{
				int dim = (unsigned int)sqrt((double)mxThreads);

				threads.x = dim;
				threads.y = dim;
				threads.z = 1;
			}
		}
	}
	else
	{
		if(dims.x*dims.y*dims.z < mxThreads)
		{
			threads.x = (unsigned int)dims.x;
			threads.y = (unsigned int)dims.y;
			threads.z = (unsigned int)dims.z;
		}
		else
		{
			unsigned long index;
			_BitScanReverse(&index,unsigned long(mxThreads));

			int dim = index/3;
			threads.x = 1 << MAX(dim,(int)index - 2*dim);
			threads.y = 1 << dim;
			threads.z = 1 << MIN(dim,(int)index - 2*dim);
		}
	}

	blocks.x = (unsigned int)ceil((float)dims.x/threads.x);
	blocks.y = (unsigned int)ceil((float)dims.y/threads.y);
	blocks.z = (unsigned int)ceil((float)dims.z/threads.z);
}

Vec<size_t> createGaussianKernel(Vec<float> sigma, float** kernelOut, Vec<int>& iterations)
{
	Vec<size_t> kernelDims(1,1,1);
	iterations = Vec<int>(1,1,1);

	if ((sigma.x+sigma.y+sigma.z)*3>MAX_KERNEL_DIM*MAX_KERNEL_DIM*MAX_KERNEL_DIM)
	{
		iterations.x = (int)MAX(1.0f,ceil(9.0f*SQR(sigma.x)/SQR(MAX_KERNEL_DIM)));
		iterations.y = (int)MAX(1.0f,ceil(9.0f*SQR(sigma.y)/SQR(MAX_KERNEL_DIM)));
		iterations.z = (int)MAX(1.0f,ceil(9.0f*SQR(sigma.z)/SQR(MAX_KERNEL_DIM)));

		//TODO: Optimize iterations per dim
		sigma.x = sigma.x/sqrt((float)iterations.x);
		sigma.y = sigma.y/sqrt((float)iterations.y);
		sigma.z = sigma.z/sqrt((float)iterations.z);
	}

	kernelDims.x = (size_t)MAX(1.0f,(3*sigma.x));
	kernelDims.y = (size_t)MAX(1.0f,(3*sigma.y));
	kernelDims.z = (size_t)MAX(1.0f,(3*sigma.z));

	kernelDims.x = (kernelDims.x%2==0) ? (kernelDims.x+1) : (kernelDims.x);
	kernelDims.y = (kernelDims.y%2==0) ? (kernelDims.y+1) : (kernelDims.y);
	kernelDims.z = (kernelDims.z%2==0) ? (kernelDims.z+1) : (kernelDims.z);

	Vec<size_t> mid;
	mid.x = kernelDims.x/2;
	mid.y = kernelDims.y/2;
	mid.z = kernelDims.z/2;

	*kernelOut = new float[kernelDims.sum()];
	float* kernel = *kernelOut;

	float total = 0.0;
	if (sigma.x==0)
	{
		kernel[0] = 1.0f;
	}
	else
	{
		for (size_t x=0; x<kernelDims.x ; ++x)
			total += kernel[x] =  exp(-(int)(SQR(mid.x-x)) / (2*SQR(sigma.x)));
		for (size_t x=0; x<kernelDims.x ; ++x)
			kernel[x] /= total;
	}

	total = 0.0;
	if (sigma.y==0)
	{
		kernel[kernelDims.x] = 1;
	}
	else
	{
		for (size_t y=0; y<kernelDims.y ; ++y)
			total += kernel[y+kernelDims.x] = exp(-(int)(SQR(mid.y-y)) / (2*SQR(sigma.y)));
		for (size_t y=0; y < kernelDims.y ; ++y)
			kernel[y+kernelDims.x] /= total;
	}

	total = 0.0;
	if (sigma.z==0)
	{
		kernel[kernelDims.x+kernelDims.y] = 1;
	}
	else
	{
		for (size_t z=0; z<kernelDims.z ; ++z)
			total += kernel[z+kernelDims.x+kernelDims.y] = exp(-(int)(SQR(mid.z-z)) / (2*SQR(sigma.z)));
		for (size_t z=0; z < kernelDims.z ; ++z)
			kernel[z+kernelDims.x+kernelDims.y] /= total;
	}

	return kernelDims;
}

Vec<size_t> createLoGKernel(Vec<float> sigma, float** kernelOut, Vec<int>& iterations)
{
	const double PI = std::atan(1.0)*4;

	Vec<size_t> kernelDims(1, 1, 1);
	iterations = Vec<int>(1, 1, 1);

	if((sigma.x+sigma.y+sigma.z)*10>MAX_KERNEL_DIM*MAX_KERNEL_DIM*MAX_KERNEL_DIM)
	{
		iterations.x = (int)MAX(1.0f, ceil(100.0f*SQR(sigma.x)/SQR(MAX_KERNEL_DIM)));
		iterations.y = (int)MAX(1.0f, ceil(100.0f*SQR(sigma.y)/SQR(MAX_KERNEL_DIM)));
		iterations.z = (int)MAX(1.0f, ceil(100.0f*SQR(sigma.z)/SQR(MAX_KERNEL_DIM)));

		//TODO: Optimize iterations per dim
		sigma.x = sigma.x/sqrt((float)iterations.x);
		sigma.y = sigma.y/sqrt((float)iterations.y);
		sigma.z = sigma.z/sqrt((float)iterations.z);
	}

	kernelDims.x = (size_t)MAX(1.0f, (10*sigma.x));
	kernelDims.y = (size_t)MAX(1.0f, (10*sigma.y));
	kernelDims.z = (size_t)MAX(1.0f, (10*sigma.z));

	kernelDims.x = (kernelDims.x%2==0) ? (kernelDims.x+1) : (kernelDims.x);
	kernelDims.y = (kernelDims.y%2==0) ? (kernelDims.y+1) : (kernelDims.y);
	kernelDims.z = (kernelDims.z%2==0) ? (kernelDims.z+1) : (kernelDims.z);

	Vec<float> mid;
	mid.x = kernelDims.x/2;
	mid.y = kernelDims.y/2;
	mid.z = kernelDims.z/2;

	*kernelOut = new float[kernelDims.sum()];
	float* kernel = *kernelOut;

	double piPow = 2.0;
	double sigmaDem = 0.0;
	double sigmaSub = 0.0;
	int numDim = 0;

	if(sigma.x!=0)
	{
		++numDim;
		sigmaSub = sigmaSub-1.0/sigma.x;
		if(sigma.y!=0)
		{
			++numDim;
			sigmaSub = sigmaSub-1.0/sigma.y;
			if(sigma.z!=0)
			{
				++numDim;
				sigmaSub = sigmaSub-1.0/sigma.z;
				sigmaDem = sigma.product();
				piPow = 4.0;
			}
			else
			{
				sigmaDem = SQR(sigma.x) * SQR(sigma.y);
			}
		}
		else
		{
			if(sigma.z!=0)
			{
				++numDim;
				sigmaSub = sigmaSub-1.0/sigma.z;
				sigmaDem = SQR(sigma.x) * SQR(sigma.z);
			}
			else
			{
				sigmaDem = SQR(sigma.x);
			}
		}
	}
	else
	{
		if(sigma.y!=0)
		{
			++numDim;
			sigmaSub = sigmaSub-1.0/sigma.y;
			if(sigma.z!=0)
			{
				++numDim;
				sigmaSub = sigmaSub-1.0/sigma.z;
				sigmaDem = SQR(sigma.y) * SQR(sigma.z);
			}
			else
			{
				sigmaDem = SQR(sigma.y);
			}
		} 
		else
		{
			if(sigma.z!=0)
			{
				++numDim;
				sigmaSub = sigmaSub-1.0/sigma.z;
				sigmaDem = SQR(sigma.z);
			} 
			else
			{
				std::runtime_error("One dimension has to have a non-zero sigma!");
			}
		}
	}

	double sigmaGPwr = (numDim==3) ? (4) : (2);
	double sigmaEPwr = (numDim==3) ? (2) : (0);
	Vec<double> sigmaG = sigma.pwr(sigmaGPwr);
	Vec<double> sigmaE = Vec<double>(sigma).pwr(sigmaEPwr)*2.0;
	double dem = pow((2*PI), piPow) * sigmaDem;

	for(int i = 0; i<3; ++i)
	{
		size_t indStride = 0;
		for(int j = 0; j<i; ++j)
		{
			indStride += kernelDims.e[j];
		}

		if(sigma.e[i]==0)
		{
			kernel[indStride] = 1.0f;
		} else
		{
			for(int j = 0; j<kernelDims.e[i]; ++j)
			{
				double jSqr = SQR(j-mid.e[i]);// make this a coordinate based on a zero mean
				double kernelVal = (jSqr/sigmaG.e[i]);
				kernelVal += sigmaSub;
				kernelVal *= exp(-(jSqr/(sigmaE.e[i])));
				kernelVal /= dem;

				kernel[j+indStride] = (float)kernelVal;
			}
		}
	}

	return kernelDims;
}

Vec<size_t> createGaussianKernelFull(Vec<float> sigma, float** kernelOut, Vec<size_t> maxKernelSize)
{
    Vec<size_t> kernelDims = Vec<size_t>(sigma.clamp(Vec<float>(1.0f), std::numeric_limits<float>::max()));

    for(float numStd = 3.0f; numStd>1.0f; numStd -= 0.2f)
    {
        if(sigma.product()*numStd<maxKernelSize.product())
        {
            kernelDims = sigma*numStd+0.9999f;
            break;
        }
    }

    kernelDims = kernelDims.clamp(Vec<size_t>(1), Vec<size_t>(std::numeric_limits<size_t>::max()));
    sigma = sigma.clamp(Vec<float>(0.1f), Vec<float>(std::numeric_limits<float>::max()));

    if(kernelDims.product()>MAX_KERNEL_DIM*MAX_KERNEL_DIM*MAX_KERNEL_DIM)
    {
        kernelDims = Vec<size_t>(MAX_KERNEL_DIM, MAX_KERNEL_DIM, MAX_KERNEL_DIM);
    }

    Vec<float> center = (kernelDims-1.0f)/2.0f;

    *kernelOut = new float[kernelDims.product()];
    float* kernel = *kernelOut;

    float total = 0.0f;
    Vec<float> pos(0, 0, 0);
    
    Vec<float> denominator = SQR(sigma)*2;
    for(pos.z = 0; pos.z<kernelDims.z; ++pos.z)
    {
        for(pos.y = 0; pos.y<kernelDims.y; ++pos.y)
        {
            for(pos.x = 0; pos.x<kernelDims.x; ++pos.x)
            {
                Vec<float> mahal = SQR(center-pos)/denominator;
                kernel[kernelDims.linearAddressAt(pos)] = exp(-(mahal.sum()));
                total += kernel[kernelDims.linearAddressAt(pos)];
            }
        }
    }

    for(int i = 0; i<kernelDims.product(); ++i)
        kernel[i] /= total;

    return kernelDims;
}

size_t memoryAvailable(int device, size_t* totalOut/*=NULL*/)
{
	HANDLE_ERROR(hipSetDevice(device));
	size_t free, total;
	HANDLE_ERROR(hipMemGetInfo(&free,&total));

	if (totalOut!=NULL)
		*totalOut = total;

	return free;
}

bool checkFreeMemory(size_t needed, int device, bool throws/*=false*/)
{
	size_t free = memoryAvailable(device);
	if (needed>free)
	{
		if (throws)
		{
			char buff[255];
			sprintf_s(buff,"Out of CUDA Memory!\nNeed: %zu\nHave: %zu\n",needed,free);
			throw std::runtime_error(buff);
		}
		return false;
	}
	return true;
}
