#include "CWrappers.h"
#include "CudaDeviceCount.cuh"
#include "CudaDeviceStats.h"
#include "CudaMemoryStats.cuh"

#include "CudaClosure.cuh"
#include "CudaElementWiseDifference.cuh"
#include "CudaGaussian.cuh"
#include "CudaGetMinMax.cuh"
#include "CudaHighPassFilter.cuh"
#include "CudaLoG.cuh"
#include "CudaMaxFilter.cuh"
#include "CudaMedianFilter.cuh"
#include "CudaMeanFilter.cuh"
#include "CudaMinFilter.cuh"
#include "CudaMultiplySum.cuh"
#include "CudaOpener.cuh"
#include "CudaStdFilter.cuh"


void clearDevice()
{
	hipDeviceReset();
}

 int deviceCount()
 {
	 return cDeviceCount();
 }

 int deviceStats(DevStats** stats)
 {
	 return cDeviceStats(stats);
 }

 int memoryStats(size_t** stats)
 {
	 return cMemoryStats(stats);
 }

 /// Example wrapper code
 //void fooFilter(const ImageContainer<bool> imageIn, ImageContainer<bool>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 //{
	// cFooFilter(imageIn, imageOut, kernel, numIterations, device);
 //}


 //void fooFilter(const ImageContainer<char> imageIn, ImageContainer<char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 //{
	// cFooFilter(imageIn, imageOut, kernel, numIterations, device);
 //}


 //void fooFilter(const ImageContainer<unsigned char> imageIn, ImageContainer<unsigned char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 //{
	// cFooFilter(imageIn, imageOut, kernel, numIterations, device);
 //}


 //void fooFilter(const ImageContainer<short> imageIn, ImageContainer<short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 //{
	// cFooFilter(imageIn, imageOut, kernel, numIterations, device);
 //}


 //void fooFilter(const ImageContainer<unsigned short> imageIn, ImageContainer<unsigned short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 //{
	// cFooFilter(imageIn, imageOut, kernel, numIterations, device);
 //}


 //void fooFilter(const ImageContainer<int> imageIn, ImageContainer<int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 //{
	// cFooFilter(imageIn, imageOut, kernel, numIterations, device);
 //}


 //void fooFilter(const ImageContainer<unsigned int> imageIn, ImageContainer<unsigned int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 //{
	// cFooFilter(imageIn, imageOut, kernel, numIterations, device);
 //}


 //void fooFilter(const ImageContainer<float> imageIn, ImageContainer<float>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 //{
	// cFooFilter(imageIn, imageOut, kernel, numIterations, device);
 //}

 //void fooFilter(const ImageContainer<double> imageIn, ImageContainer<double>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 //{
 // cFooFilter(imageIn, imageOut, kernel, numIterations, device);
 //}

 void closure(const ImageContainer<bool> imageIn, ImageContainer<bool>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cClosure(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void closure(const ImageContainer<char> imageIn, ImageContainer<char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cClosure(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void closure(const ImageContainer<unsigned char> imageIn, ImageContainer<unsigned char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cClosure(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void closure(const ImageContainer<short> imageIn, ImageContainer<short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cClosure(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void closure(const ImageContainer<unsigned short> imageIn, ImageContainer<unsigned short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cClosure(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void closure(const ImageContainer<int> imageIn, ImageContainer<int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cClosure(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void closure(const ImageContainer<unsigned int> imageIn, ImageContainer<unsigned int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cClosure(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void closure(const ImageContainer<float> imageIn, ImageContainer<float>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cClosure(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void closure(const ImageContainer<double> imageIn, ImageContainer<double>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cClosure(imageIn, imageOut, kernel, numIterations, device);
 }

 
 void elementWiseDifference(const ImageContainer<bool> image1In, ImageContainer<bool> image2In, ImageContainer<bool>& imageOut, int device /*= -1*/)
 {
	 cElementWiseDifference(image1In, image2In, imageOut, device);
 }

 void elementWiseDifference(const ImageContainer<char> image1In, ImageContainer<char> image2In, ImageContainer<char>& imageOut, int device /*= -1*/)
 {
	 cElementWiseDifference(image1In, image2In, imageOut, device);
 }

 void elementWiseDifference(const ImageContainer<unsigned char> image1In, ImageContainer<unsigned char> image2In, ImageContainer<unsigned char>& imageOut, int device /*= -1*/)
 {
	 cElementWiseDifference(image1In, image2In, imageOut, device);
 }

 void elementWiseDifference(const ImageContainer<short> image1In, ImageContainer<short> image2In, ImageContainer<short>& imageOut, int device /*= -1*/)
 {
	 cElementWiseDifference(image1In, image2In, imageOut, device);
 }

 void elementWiseDifference(const ImageContainer<unsigned short> image1In, ImageContainer<unsigned short> image2In, ImageContainer<unsigned short>& imageOut, int device /*= -1*/)
 {
	 cElementWiseDifference(image1In, image2In, imageOut, device);
 }

 void elementWiseDifference(const ImageContainer<int> image1In, ImageContainer<int> image2In, ImageContainer<int>& imageOut, int device /*= -1*/)
 {
	 cElementWiseDifference(image1In, image2In, imageOut, device);
 }

 void elementWiseDifference(const ImageContainer<unsigned int> image1In, ImageContainer<unsigned int> image2In, ImageContainer<unsigned int>& imageOut, int device /*= -1*/)
 {
	 cElementWiseDifference(image1In, image2In, imageOut, device);
 }

 void elementWiseDifference(const ImageContainer<float> image1In, ImageContainer<float> image2In, ImageContainer<float>& imageOut, int device /*= -1*/)
 {
	 cElementWiseDifference(image1In, image2In, imageOut, device);
 }

 void elementWiseDifference(const ImageContainer<double> image1In, ImageContainer<double> image2In, ImageContainer<double>& imageOut, int device /*= -1*/)
 {
	 cElementWiseDifference(image1In, image2In, imageOut, device);
 }


void gaussian(const ImageContainer<bool> imageIn, ImageContainer<bool>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cGaussian(imageIn, imageOut, sigmas, numIterations, device);
 }
 
 void gaussian(const ImageContainer<char> imageIn, ImageContainer<char>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cGaussian(imageIn, imageOut, sigmas, numIterations, device);
 }
 
 void gaussian(const ImageContainer<unsigned char> imageIn, ImageContainer<unsigned char>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cGaussian(imageIn, imageOut, sigmas, numIterations, device);
 }
 
 void gaussian(const ImageContainer<short> imageIn, ImageContainer<short>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cGaussian(imageIn, imageOut, sigmas, numIterations, device);
 }
 
 void gaussian(const ImageContainer<unsigned short> imageIn, ImageContainer<unsigned short>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cGaussian(imageIn, imageOut, sigmas, numIterations, device);
 }
 
 void gaussian(const ImageContainer<int> imageIn, ImageContainer<int>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cGaussian(imageIn, imageOut, sigmas, numIterations, device);
 }
 
 void gaussian(const ImageContainer<unsigned int> imageIn, ImageContainer<unsigned int>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cGaussian(imageIn, imageOut, sigmas, numIterations, device);
 }

 void gaussian(const ImageContainer<float> imageIn, ImageContainer<float>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cGaussian(imageIn, imageOut, sigmas, numIterations, device);
 }

 void gaussian(const ImageContainer<double> imageIn, ImageContainer<double>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cGaussian(imageIn, imageOut, sigmas, numIterations, device);
 }


 void getMinMax(const bool* imageIn, size_t numElements, bool& minVal, bool& maxVal, int device /*= 0*/)
 {
	 cGetMinMax(imageIn, numElements, minVal, maxVal, device);
 }

 void getMinMax(const char* imageIn, size_t numElements, char& minVal, char& maxVal, int device /*= 0*/)
 {
	 cGetMinMax(imageIn, numElements, minVal, maxVal, device);
 }

 void getMinMax(const unsigned char* imageIn, size_t numElements, unsigned char& minVal, unsigned char& maxVal, int device /*= 0*/)
 {
	 cGetMinMax(imageIn, numElements, minVal, maxVal, device);
 }

 void getMinMax(const short* imageIn, size_t numElements, short& minVal, short& maxVal, int device /*= 0*/)
 {
	 cGetMinMax(imageIn, numElements, minVal, maxVal, device);
 }

 void getMinMax(const unsigned short* imageIn, size_t numElements, unsigned short& minVal, unsigned short& maxVal, int device /*= 0*/)
 {
	 cGetMinMax(imageIn, numElements, minVal, maxVal, device);
 }

 void getMinMax(const int* imageIn, size_t numElements, int& minVal, int& maxVal, int device /*= 0*/)
 {
	 cGetMinMax(imageIn, numElements, minVal, maxVal, device);
 }

 void getMinMax(const unsigned int* imageIn, size_t numElements, unsigned int& minVal, unsigned int& maxVal, int device /*= 0*/)
 {
	 cGetMinMax(imageIn, numElements, minVal, maxVal, device);
 }

 void getMinMax(const float* imageIn, size_t numElements, float& minVal, float& maxVal, int device /*= 0*/)
 {
	 cGetMinMax(imageIn, numElements, minVal, maxVal, device);
 }

 void getMinMax(const double* imageIn, size_t numElements, double& minVal, double& maxVal, int device /*= 0*/)
 {
	 cGetMinMax(imageIn, numElements, minVal, maxVal, device);
 }


 void highPassFilter(const ImageContainer<bool> imageIn, ImageContainer<bool>& imageOut, Vec<double> sigmas, int device /*= -1*/)
 {
	 cHighPassFilter(imageIn, imageOut, sigmas, device);
 }
 
 void highPassFilter(const ImageContainer<char> imageIn, ImageContainer<char>& imageOut, Vec<double> sigmas, int device /*= -1*/)
 {
	 cHighPassFilter(imageIn, imageOut, sigmas, device);
 }
 
 void highPassFilter(const ImageContainer<unsigned char> imageIn, ImageContainer<unsigned char>& imageOut, Vec<double> sigmas, int device /*= -1*/)
 {
	 cHighPassFilter(imageIn, imageOut, sigmas, device);
 }
 
 void highPassFilter(const ImageContainer<short> imageIn, ImageContainer<short>& imageOut, Vec<double> sigmas, int device /*= -1*/)
 {
	 cHighPassFilter(imageIn, imageOut, sigmas, device);
 }
 
 void highPassFilter(const ImageContainer<unsigned short> imageIn, ImageContainer<unsigned short>& imageOut, Vec<double> sigmas, int device /*= -1*/)
 {
	 cHighPassFilter(imageIn, imageOut, sigmas, device);
 }
 
 void highPassFilter(const ImageContainer<int> imageIn, ImageContainer<int>& imageOut, Vec<double> sigmas, int device /*= -1*/)
 {
	 cHighPassFilter(imageIn, imageOut, sigmas, device);
 }
 
 void highPassFilter(const ImageContainer<unsigned int> imageIn, ImageContainer<unsigned int>& imageOut, Vec<double> sigmas, int device /*= -1*/)
 {
	 cHighPassFilter(imageIn, imageOut, sigmas, device);
 }
 
 void highPassFilter(const ImageContainer<float> imageIn, ImageContainer<float>& imageOut, Vec<double> sigmas, int device /*= -1*/)
 {
	 cHighPassFilter(imageIn, imageOut, sigmas, device);
 }
 
 void highPassFilter(const ImageContainer<double> imageIn, ImageContainer<double>& imageOut, Vec<double> sigmas, int device /*= -1*/)
 {
	 cHighPassFilter(imageIn, imageOut, sigmas, device);
 }


 void LoG(const ImageContainer<bool> imageIn, ImageContainer<float>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cLoG(imageIn, imageOut, sigmas, numIterations, device);
 }

 void LoG(const ImageContainer<char> imageIn, ImageContainer<float>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cLoG(imageIn, imageOut, sigmas, numIterations, device);
 }

 void LoG(const ImageContainer<unsigned char> imageIn, ImageContainer<float>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cLoG(imageIn, imageOut, sigmas, numIterations, device);
 }

 void LoG(const ImageContainer<short> imageIn, ImageContainer<float>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cLoG(imageIn, imageOut, sigmas, numIterations, device);
 }

 void LoG(const ImageContainer<unsigned short> imageIn, ImageContainer<float>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cLoG(imageIn, imageOut, sigmas, numIterations, device);
 }

 void LoG(const ImageContainer<int> imageIn, ImageContainer<float>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cLoG(imageIn, imageOut, sigmas, numIterations, device);
 }

 void LoG(const ImageContainer<unsigned int> imageIn, ImageContainer<float>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cLoG(imageIn, imageOut, sigmas, numIterations, device);
 }

 void LoG(const ImageContainer<float> imageIn, ImageContainer<float>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cLoG(imageIn, imageOut, sigmas, numIterations, device);
 }

 void LoG(const ImageContainer<double> imageIn, ImageContainer<float>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cLoG(imageIn, imageOut, sigmas, numIterations, device);
 }


 void maxFilter(const ImageContainer<bool> imageIn, ImageContainer<bool>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void maxFilter(const ImageContainer<char> imageIn, ImageContainer<char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void maxFilter(const ImageContainer<unsigned char> imageIn, ImageContainer<unsigned char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void maxFilter(const ImageContainer<short> imageIn, ImageContainer<short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void maxFilter(const ImageContainer<unsigned short> imageIn, ImageContainer<unsigned short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void maxFilter(const ImageContainer<int> imageIn, ImageContainer<int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void maxFilter(const ImageContainer<unsigned int> imageIn, ImageContainer<unsigned int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void maxFilter(const ImageContainer<float> imageIn, ImageContainer<float>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void maxFilter(const ImageContainer<double> imageIn, ImageContainer<double>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 

 void meanFilter(const ImageContainer<bool> imageIn, ImageContainer<bool>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMeanFilter(imageIn, imageOut, kernel, numIterations, device);
 }

 void meanFilter(const ImageContainer<char> imageIn, ImageContainer<char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMeanFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void meanFilter(const ImageContainer<unsigned char> imageIn, ImageContainer<unsigned char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMeanFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void meanFilter(const ImageContainer<short> imageIn, ImageContainer<short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMeanFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void meanFilter(const ImageContainer<unsigned short> imageIn, ImageContainer<unsigned short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMeanFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void meanFilter(const ImageContainer<int> imageIn, ImageContainer<int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMeanFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void meanFilter(const ImageContainer<unsigned int> imageIn, ImageContainer<unsigned int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMeanFilter(imageIn, imageOut, kernel, numIterations, device);
 }

 void meanFilter(const ImageContainer<float> imageIn, ImageContainer<float>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMeanFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void meanFilter(const ImageContainer<double> imageIn, ImageContainer<double>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMeanFilter(imageIn, imageOut, kernel, numIterations, device);
 }


 void medianFilter(const ImageContainer<bool> imageIn, ImageContainer<bool>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMedianFilter(imageIn, imageOut, kernel, numIterations, device);
 }

 void medianFilter(const ImageContainer<char> imageIn, ImageContainer<char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMedianFilter(imageIn, imageOut, kernel, numIterations, device);
 }

 void medianFilter(const ImageContainer<unsigned char> imageIn, ImageContainer<unsigned char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMedianFilter(imageIn, imageOut, kernel, numIterations, device);
 }

 void medianFilter(const ImageContainer<short> imageIn, ImageContainer<short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMedianFilter(imageIn, imageOut, kernel, numIterations, device);
 }

 void medianFilter(const ImageContainer<unsigned short> imageIn, ImageContainer<unsigned short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMedianFilter(imageIn, imageOut, kernel, numIterations, device);
 }

 void medianFilter(const ImageContainer<int> imageIn, ImageContainer<int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMedianFilter(imageIn, imageOut, kernel, numIterations, device);
 }

 void medianFilter(const ImageContainer<unsigned int> imageIn, ImageContainer<unsigned int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMedianFilter(imageIn, imageOut, kernel, numIterations, device);
 }

 void medianFilter(const ImageContainer<float> imageIn, ImageContainer<float>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMedianFilter(imageIn, imageOut, kernel, numIterations, device);
 }

 void medianFilter(const ImageContainer<double> imageIn, ImageContainer<double>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMedianFilter(imageIn, imageOut, kernel, numIterations, device);
 }


 void minFilter(const ImageContainer<bool> imageIn, ImageContainer<bool>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMinFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void minFilter(const ImageContainer<char> imageIn, ImageContainer<char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMinFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void minFilter(const ImageContainer<unsigned char> imageIn, ImageContainer<unsigned char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMinFilter(imageIn, imageOut, kernel, numIterations, device);
 }

 void minFilter(const ImageContainer<short> imageIn, ImageContainer<short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMinFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void minFilter(const ImageContainer<unsigned short> imageIn, ImageContainer<unsigned short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMinFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void minFilter(const ImageContainer<int> imageIn, ImageContainer<int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMinFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void minFilter(const ImageContainer<unsigned int> imageIn, ImageContainer<unsigned int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMinFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void minFilter(const ImageContainer<float> imageIn, ImageContainer<float>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMinFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void minFilter(const ImageContainer<double> imageIn, ImageContainer<double>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMinFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 

 void multiplySum(const ImageContainer<bool> imageIn, ImageContainer<bool>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMultiplySum(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void multiplySum(const ImageContainer<char> imageIn, ImageContainer<char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMultiplySum(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void multiplySum(const ImageContainer<unsigned char> imageIn, ImageContainer<unsigned char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMultiplySum(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void multiplySum(const ImageContainer<short> imageIn, ImageContainer<short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMultiplySum(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void multiplySum(const ImageContainer<unsigned short> imageIn, ImageContainer<unsigned short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMultiplySum(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void multiplySum(const ImageContainer<int> imageIn, ImageContainer<int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMultiplySum(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void multiplySum(const ImageContainer<unsigned int> imageIn, ImageContainer<unsigned int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMultiplySum(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void multiplySum(const ImageContainer<float> imageIn, ImageContainer<float>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMultiplySum(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void multiplySum(const ImageContainer<double> imageIn, ImageContainer<double>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMultiplySum(imageIn, imageOut, kernel, numIterations, device);
 }
 

 void opener(const ImageContainer<bool> imageIn, ImageContainer<bool>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cOpener(imageIn, imageOut, kernel, numIterations, device);
 }

 void opener(const ImageContainer<char> imageIn, ImageContainer<char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cOpener(imageIn, imageOut, kernel, numIterations, device);
 }

 void opener(const ImageContainer<unsigned char> imageIn, ImageContainer<unsigned char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cOpener(imageIn, imageOut, kernel, numIterations, device);
 }

 void opener(const ImageContainer<short> imageIn, ImageContainer<short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cOpener(imageIn, imageOut, kernel, numIterations, device);
 }

 void opener(const ImageContainer<unsigned short> imageIn, ImageContainer<unsigned short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cOpener(imageIn, imageOut, kernel, numIterations, device);
 }

 void opener(const ImageContainer<int> imageIn, ImageContainer<int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cOpener(imageIn, imageOut, kernel, numIterations, device);
 }

 void opener(const ImageContainer<unsigned int> imageIn, ImageContainer<unsigned int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cOpener(imageIn, imageOut, kernel, numIterations, device);
 }

 void opener(const ImageContainer<float> imageIn, ImageContainer<float>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cOpener(imageIn, imageOut, kernel, numIterations, device);
 }

 void opener(const ImageContainer<double> imageIn, ImageContainer<double>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cOpener(imageIn, imageOut, kernel, numIterations, device);
 }


 void stdFilter(const ImageContainer<bool> imageIn, ImageContainer<bool>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cStdFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void stdFilter(const ImageContainer<char> imageIn, ImageContainer<char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cStdFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void stdFilter(const ImageContainer<unsigned char> imageIn, ImageContainer<unsigned char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cStdFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void stdFilter(const ImageContainer<short> imageIn, ImageContainer<short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cStdFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void stdFilter(const ImageContainer<unsigned short> imageIn, ImageContainer<unsigned short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cStdFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void stdFilter(const ImageContainer<int> imageIn, ImageContainer<int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cStdFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void stdFilter(const ImageContainer<unsigned int> imageIn, ImageContainer<unsigned int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cStdFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void stdFilter(const ImageContainer<float> imageIn, ImageContainer<float>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cStdFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void stdFilter(const ImageContainer<double> imageIn, ImageContainer<double>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cStdFilter(imageIn, imageOut, kernel, numIterations, device);
 }
