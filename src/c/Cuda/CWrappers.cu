#include "CWrappers.h"
#include "CudaDeviceCount.cuh"
#include "CudaDeviceStats.h"
#include "CudaMemoryStats.cuh"

#include "CudaMaxFilter.cuh"


void clearDevice()
{
	hipDeviceReset();
}

 int deviceCount()
 {
	 return cDeviceCount();
 }

 int deviceStats(DevStats** stats)
 {
	 return cDeviceStats(stats);
 }

 int memoryStats(size_t** stats)
 {
	 return cMemoryStats(stats);
 }


 void maxFilter(const ImageContainer<bool> imageIn, ImageContainer<bool>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }


 void maxFilter(const ImageContainer<char> imageIn, ImageContainer<char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }


 void maxFilter(const ImageContainer<unsigned char> imageIn, ImageContainer<unsigned char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }


 void maxFilter(const ImageContainer<short> imageIn, ImageContainer<short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }


 void maxFilter(const ImageContainer<unsigned short> imageIn, ImageContainer<unsigned short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }


 void maxFilter(const ImageContainer<int> imageIn, ImageContainer<int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }


 void maxFilter(const ImageContainer<unsigned int> imageIn, ImageContainer<unsigned int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }


 void maxFilter(const ImageContainer<float> imageIn, ImageContainer<float>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }


 void maxFilter(const ImageContainer<double> imageIn, ImageContainer<double>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }
