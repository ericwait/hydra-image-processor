#include "CWrappers.h"
#include "CudaDeviceCount.cuh"
#include "CudaDeviceStats.h"
#include "CudaMemoryStats.cuh"

#include "CudaGaussian.cuh"
#include "CudaMaxFilter.cuh"
#include "CudaMinFilter.cuh"
#include "CudaMultiplySum.cuh"


void clearDevice()
{
	hipDeviceReset();
}

 int deviceCount()
 {
	 return cDeviceCount();
 }

 int deviceStats(DevStats** stats)
 {
	 return cDeviceStats(stats);
 }

 int memoryStats(size_t** stats)
 {
	 return cMemoryStats(stats);
 }

 /// Example wrapper code
 //void fooFilter(const ImageContainer<bool> imageIn, ImageContainer<bool>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 //{
	// cFooFilter(imageIn, imageOut, kernel, numIterations, device);
 //}


 //void fooFilter(const ImageContainer<char> imageIn, ImageContainer<char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 //{
	// cFooFilter(imageIn, imageOut, kernel, numIterations, device);
 //}


 //void fooFilter(const ImageContainer<unsigned char> imageIn, ImageContainer<unsigned char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 //{
	// cFooFilter(imageIn, imageOut, kernel, numIterations, device);
 //}


 //void fooFilter(const ImageContainer<short> imageIn, ImageContainer<short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 //{
	// cFooFilter(imageIn, imageOut, kernel, numIterations, device);
 //}


 //void fooFilter(const ImageContainer<unsigned short> imageIn, ImageContainer<unsigned short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 //{
	// cFooFilter(imageIn, imageOut, kernel, numIterations, device);
 //}


 //void fooFilter(const ImageContainer<int> imageIn, ImageContainer<int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 //{
	// cFooFilter(imageIn, imageOut, kernel, numIterations, device);
 //}


 //void fooFilter(const ImageContainer<unsigned int> imageIn, ImageContainer<unsigned int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 //{
	// cFooFilter(imageIn, imageOut, kernel, numIterations, device);
 //}


 //void fooFilter(const ImageContainer<float> imageIn, ImageContainer<float>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 //{
	// cFooFilter(imageIn, imageOut, kernel, numIterations, device);
 //}


 //void fooFilter(const ImageContainer<double> imageIn, ImageContainer<double>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 //{
	// cFooFilter(imageIn, imageOut, kernel, numIterations, device);
 //}



 void maxFilter(const ImageContainer<bool> imageIn, ImageContainer<bool>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }


 void maxFilter(const ImageContainer<char> imageIn, ImageContainer<char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }


 void maxFilter(const ImageContainer<unsigned char> imageIn, ImageContainer<unsigned char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }


 void maxFilter(const ImageContainer<short> imageIn, ImageContainer<short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }


 void maxFilter(const ImageContainer<unsigned short> imageIn, ImageContainer<unsigned short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }


 void maxFilter(const ImageContainer<int> imageIn, ImageContainer<int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }


 void maxFilter(const ImageContainer<unsigned int> imageIn, ImageContainer<unsigned int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }


 void maxFilter(const ImageContainer<float> imageIn, ImageContainer<float>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }


 void maxFilter(const ImageContainer<double> imageIn, ImageContainer<double>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 void gaussian(const ImageContainer<bool> imageIn, ImageContainer<bool>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
	 cGaussian(imageIn, imageOut, sigmas, numIterations, device);
 }
 
 void gaussian(const ImageContainer<char> imageIn, ImageContainer<char>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cGaussian(imageIn, imageOut, sigmas, numIterations, device);
 }
 
 void gaussian(const ImageContainer<unsigned char> imageIn, ImageContainer<unsigned char>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cGaussian(imageIn, imageOut, sigmas, numIterations, device);
 }
 
 void gaussian(const ImageContainer<short> imageIn, ImageContainer<short>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cGaussian(imageIn, imageOut, sigmas, numIterations, device);
 }
 
 void gaussian(const ImageContainer<unsigned short> imageIn, ImageContainer<unsigned short>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cGaussian(imageIn, imageOut, sigmas, numIterations, device);
 }
 
 void gaussian(const ImageContainer<int> imageIn, ImageContainer<int>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cGaussian(imageIn, imageOut, sigmas, numIterations, device);
 }
 
 void gaussian(const ImageContainer<unsigned int> imageIn, ImageContainer<unsigned int>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cGaussian(imageIn, imageOut, sigmas, numIterations, device);
 }

 void gaussian(const ImageContainer<float> imageIn, ImageContainer<float>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cGaussian(imageIn, imageOut, sigmas, numIterations, device);
 }

 void gaussian(const ImageContainer<double> imageIn, ImageContainer<double>& imageOut, Vec<double> sigmas, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cGaussian(imageIn, imageOut, sigmas, numIterations, device);
 }


 void maxFilter(const ImageContainer<bool> imageIn, ImageContainer<bool>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void maxFilter(const ImageContainer<char> imageIn, ImageContainer<char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void maxFilter(const ImageContainer<unsigned char> imageIn, ImageContainer<unsigned char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void maxFilter(const ImageContainer<short> imageIn, ImageContainer<short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void maxFilter(const ImageContainer<unsigned short> imageIn, ImageContainer<unsigned short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void maxFilter(const ImageContainer<int> imageIn, ImageContainer<int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void maxFilter(const ImageContainer<unsigned int> imageIn, ImageContainer<unsigned int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void maxFilter(const ImageContainer<float> imageIn, ImageContainer<float>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void maxFilter(const ImageContainer<double> imageIn, ImageContainer<double>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMaxFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void minFilter(const ImageContainer<bool> imageIn, ImageContainer<bool>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMinFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void minFilter(const ImageContainer<char> imageIn, ImageContainer<char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMinFilter(imageIn, imageOut, kernel, numIterations, device);
 }
 
 void minFilter(const ImageContainer<unsigned char> imageIn, ImageContainer<unsigned char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMinFilter(imageIn, imageOut, kernel, numIterations, device);
 }


 void minFilter(const ImageContainer<short> imageIn, ImageContainer<short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMinFilter(imageIn, imageOut, kernel, numIterations, device);
 }


 void minFilter(const ImageContainer<unsigned short> imageIn, ImageContainer<unsigned short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMinFilter(imageIn, imageOut, kernel, numIterations, device);
 }


 void minFilter(const ImageContainer<int> imageIn, ImageContainer<int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMinFilter(imageIn, imageOut, kernel, numIterations, device);
 }


 void minFilter(const ImageContainer<unsigned int> imageIn, ImageContainer<unsigned int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMinFilter(imageIn, imageOut, kernel, numIterations, device);
 }


 void minFilter(const ImageContainer<float> imageIn, ImageContainer<float>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMinFilter(imageIn, imageOut, kernel, numIterations, device);
 }


 void minFilter(const ImageContainer<double> imageIn, ImageContainer<double>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMinFilter(imageIn, imageOut, kernel, numIterations, device);
 }



 IMAGE_PROCESSOR_API void multiplySum(const ImageContainer<bool> imageIn, ImageContainer<bool>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMultiplySum(imageIn, imageOut, kernel, numIterations, device);
 }


 IMAGE_PROCESSOR_API void multiplySum(const ImageContainer<char> imageIn, ImageContainer<char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMultiplySum(imageIn, imageOut, kernel, numIterations, device);
 }


 IMAGE_PROCESSOR_API void multiplySum(const ImageContainer<unsigned char> imageIn, ImageContainer<unsigned char>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMultiplySum(imageIn, imageOut, kernel, numIterations, device);
 }


 IMAGE_PROCESSOR_API void multiplySum(const ImageContainer<short> imageIn, ImageContainer<short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMultiplySum(imageIn, imageOut, kernel, numIterations, device);
 }


 IMAGE_PROCESSOR_API void multiplySum(const ImageContainer<unsigned short> imageIn, ImageContainer<unsigned short>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMultiplySum(imageIn, imageOut, kernel, numIterations, device);
 }


 IMAGE_PROCESSOR_API void multiplySum(const ImageContainer<int> imageIn, ImageContainer<int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMultiplySum(imageIn, imageOut, kernel, numIterations, device);
 }


 IMAGE_PROCESSOR_API void multiplySum(const ImageContainer<unsigned int> imageIn, ImageContainer<unsigned int>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMultiplySum(imageIn, imageOut, kernel, numIterations, device);
 }


 IMAGE_PROCESSOR_API void multiplySum(const ImageContainer<float> imageIn, ImageContainer<float>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMultiplySum(imageIn, imageOut, kernel, numIterations, device);
 }


 IMAGE_PROCESSOR_API void multiplySum(const ImageContainer<double> imageIn, ImageContainer<double>& imageOut, ImageContainer<float> kernel, int numIterations /*= 1*/, int device /*= -1*/)
 {
	 cMultiplySum(imageIn, imageOut, kernel, numIterations, device);
 }

