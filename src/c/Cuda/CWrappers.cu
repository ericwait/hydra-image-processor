#include "CWrappers.h"
#include "ImageContainer.h"
#include "CudaDeviceCount.cuh"
#include "CudaDeviceStats.h"
#include "CudaMemoryStats.cuh"


void clearDevice()
{
	hipDeviceReset();
}

 int deviceCount()
 {
	 return cDeviceCount();
 }

 int deviceStats(DevStats** stats)
 {
	 return cDeviceStats(stats);
 }

 int memoryStats(size_t** stats)
 {
	 return cMemoryStats(stats);
 }
